////////////////////////////////////////////////////////////////////
//
// Dean Andrew Hidas <dhidas@bnl.gov>
//
// Created on: Tue May 24 11:33:19 EDT 2016
//
////////////////////////////////////////////////////////////////////

#include <hip/hip_complex.h>

#include "OSCARSTH_Cuda.h"

#include "OSCARSTH.h"

#include <cmath>
#include <fstream>
#include <sstream>



#define NTHREADS_PER_BLOCK 512






extern "C" int OSCARSTH_Cuda_GetDeviceCount ()
{
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);

  return ngpu;
}





