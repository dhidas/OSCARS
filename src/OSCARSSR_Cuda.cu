#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////
//
// Dean Andrew Hidas <dhidas@bnl.gov>
//
// Created on: Tue May 24 11:33:19 EDT 2016
//
////////////////////////////////////////////////////////////////////

#include <hip/hip_complex.h>

#include "OSCARSSR_Cuda.h"

#include "OSCARSSR.h"

#include <cmath>
#include <fstream>
#include <sstream>
#include <algorithm>

#include "TVector3DC.h"
#include "TSpectrumContainer.h"


#define NTHREADS_PER_BLOCK 512






extern "C" int OSCARSSR_Cuda_GetDeviceCount ()
{
  static int ngpu = 0;
  hipGetDeviceCount(&ngpu);

  return ngpu;
}





std::string OSCARSSR_Cuda_GetDeviceProperties (int const i)
{
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);

  char buf[300];

  if (i >= ngpu) {
    sprintf(buf, "ERROR: GPU %i Not available", i);
    return std::string(buf);
  }

  std::string ret = "";

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, i);

  sprintf(buf, "  Device name: %s\n", prop.name);
  ret += std::string(buf);
  sprintf(buf, "  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
  ret += std::string(buf);
  sprintf(buf, "  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
  ret += std::string(buf);
  sprintf(buf, "  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  ret += std::string(buf);

  return ret;
}





__device__ static __inline__ void Orthogonal(double *a, double *b)
{
  // Return a vector which is orthogonal vector a
  double xx = a[0] < 0.0 ? -a[0] : a[0];
  double yy = a[1] < 0.0 ? -a[1] : a[1];
  double zz = a[2] < 0.0 ? -a[2] : a[2];
  if (xx < yy) {
    if (xx < zz) {
      b[0] = 0;
      b[1] = a[2];
      b[2] = -a[1];
    } else {
      b[0] = a[1];
      b[1] = -a[0];
      b[2] = 0;
    }
  } else {
    if (yy < zz) {
      b[0] = -a[2];
      b[1] = 0;
      b[2] = a[0];
    } else {
      b[0] = a[1];
      b[1] = -a[0];
      b[2] = 0;
    }
  }
  return;
}



__host__ __device__ static __inline__ hipDoubleComplex cuCexp(hipDoubleComplex x)
{
  double factor = exp(x.x);
  return make_hipDoubleComplex(factor * cos(x.y), factor * sin(x.y));
}




__device__ static __inline__ void GetInterpolatingIMinIMax(double *fx, int* nx, double* x, int *imin, int* imax)
{
  // get the klo and khi for interpolation

  int klo=0;
  int khi = *nx - 1;
  int k;
  while (khi - klo > 1) {
    k = (khi + klo) >> 1;
    if (fx[k] > *x) {
      khi = k;
    } else {
      klo = k;
    }
  }

  *imin = klo;
  *imax = khi;

  return;
}








__device__ static __inline__ double Interpolate (double *fx, double* fy, double *fypp, double* x, int* klo, int* khi)
{
  // Return the Y-value according to spline


  // Distance between points, check that it isn't zero!
  double const h = fx[*khi] - fx[*klo];
  if (h == 0) {
    // UPDATE: supposed to throw CUDA
  }

  // Fractional distance to the points on either side
  double const a = (fx[*khi] - *x) / h;
  double const b = (*x - fx[*klo]) / h;

  // Return the value of Y
  return a * fy[*klo] + b * fy[*khi] + ((a * a * a - a) * fypp[*klo] + (b * b * b - b) * fypp[*khi]) * (h * h) / 6.;
}











__device__ static __inline__ double Interpolate (double *fx, int* nx, double* fy, double *fypp, double* x)
{
  // Return the Y-value according to spline

  int klo=0;
  int khi = *nx - 1;
  int k;
  while (khi - klo > 1) {
    k = (khi + klo) >> 1;
    if (fx[k] > *x) {
      khi = k;
    } else {
      klo = k;
    }
  }

  // Distance between points, check that it isn't zero!
  double const h = fx[khi] - fx[klo];
  if (h == 0) {
    // UPDATE: supposed to throw CUDA
  }

  // Fractional distance to the points on either side
  double const a = (fx[khi] - *x) / h;
  double const b = (*x - fx[klo]) / h;

  // Return the value of Y
  return a * fy[klo] + b * fy[khi] + ((a * a * a - a) * fypp[klo] + (b * b * b - b) * fypp[khi]) * (h * h) / 6.;
}





















__global__ void OSCARSSR_Cuda_FluxGPUMulti (double *x, double *y, double *z, double *bx, double *by, double *bz, double *sx, double *sy, double *sz, double *dt, int *nt, int *ns, double *C0, double *C2, double *C, double *Omega, int *ifirst, double *flux)
{
  // Check that this is within the number of spectrum points requested
  int const ith = threadIdx.x + blockIdx.x * blockDim.x;
  int const is = ith + *ifirst;
  if (is >= *ns) {
    return;
  }


  // Complex i
  hipDoubleComplex I = make_hipDoubleComplex(0, 1);

  hipDoubleComplex ICoverOmega = make_hipDoubleComplex(0, (*C) / (*Omega));

  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);


  // Loop over all points in trajectory
  for (int i = 0; i < *nt; ++i) {

    // Distance to observer
    double const D = sqrt( pow( (ox) - x[i], 2) + pow( (oy) - y[i], 2) + pow((oz) - z[i], 2) );

    // Normal in direction of observer
    double const NX = ((ox) - x[i]) / D;
    double const NY = ((oy) - y[i]) / D;
    double const NZ = ((oz) - z[i]) / D;

    // Exponent for fourier transformed field
    hipDoubleComplex Exponent = make_hipDoubleComplex(0, (*Omega) * ((*dt) * i + D / (*C)));

    hipDoubleComplex X1 = make_hipDoubleComplex((bx[i] - NX) / D, -(*C) * NX / ((*Omega) * D * D));
    hipDoubleComplex Y1 = make_hipDoubleComplex((by[i] - NY) / D, -(*C) * NY / ((*Omega) * D * D));
    hipDoubleComplex Z1 = make_hipDoubleComplex((bz[i] - NZ) / D, -(*C) * NZ / ((*Omega) * D * D));

    hipDoubleComplex MyEXP = cuCexp(Exponent);
    //hipDoubleComplex MyEXP = make_hipDoubleComplex( exp(Exponent.x) * cos(Exponent.y), exp(Exponent.x) * sin(Exponent.y));

    hipDoubleComplex X2 = hipCmul(X1, MyEXP);
    hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
    hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


    SumEX = hipCadd(SumEX, X2);
    SumEY = hipCadd(SumEY, Y2);
    SumEZ = hipCadd(SumEZ, Z2);

    // Sum in fourier transformed field (integral)
    //SumEX += (TVector3DC(B) - (N *     (One + (ICoverOmega / (D)))     )) / D * std::exp(Exponent);
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEZ);


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;

  // Multiply field by Constant C1 and time step
  //SumE *= C1 * DeltaT;

  // Set the flux for this frequency / energy point
  //Spectrum.AddToFlux(i, C2 *  SumE.Dot( SumE.CC() ).real() * Weight);

  flux[ith] = (*C2) * (EX + EY + EZ);

  return;
}








__global__ void OSCARSSR_Cuda_FluxGPUMultiWithAInterpolated (double *t,
                                                             double  *x,   double *y,   double *z,
                                                             double  *xp,  double *yp,  double *zp,
                                                             double  *bx,  double *by,  double *bz,
                                                             double  *bxp, double *byp, double *bzp,
                                                             double  *ax,  double *ay,  double *az,
                                                             double  *axp, double *ayp, double *azp,
                                                             double  *sx,  double *sy,  double *sz,
                                                             double  *tstart, double *tstop,
                                                             int *nt,
                                                             int *ns,
                                                             double *C0,   double *C2,  double *C,
                                                             double *Omega,
                                                             int *ifirst,
                                                             int *ml,
                                                             double *prec,
                                                             double *flux)
{
  // Check that this is within the number of spectrum points requested
  int const ith = threadIdx.x + blockIdx.x * blockDim.x;
  int const is = ith + *ifirst;


  double const ox = is >= *ns ? 0 : sx[is];
  double const oy = is >= *ns ? 0 : sy[is];
  double const oz = is >= *ns ? 0 : sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);



  __shared__ double _t[NTHREADS_PER_BLOCK];
  __shared__ double _x[NTHREADS_PER_BLOCK];
  __shared__ double _y[NTHREADS_PER_BLOCK];
  __shared__ double _z[NTHREADS_PER_BLOCK];
  __shared__ double _bx[NTHREADS_PER_BLOCK];
  __shared__ double _by[NTHREADS_PER_BLOCK];
  __shared__ double _bz[NTHREADS_PER_BLOCK];
  __shared__ double _ax[NTHREADS_PER_BLOCK];
  __shared__ double _ay[NTHREADS_PER_BLOCK];
  __shared__ double _az[NTHREADS_PER_BLOCK];

  __shared__ bool   _done[NTHREADS_PER_BLOCK];
  __shared__ bool   _all_done;


  if (threadIdx.x == 1) {
    _all_done = false;
  }

  // initialize _done.  If not a surface point, we're going to use
  // the thread to do trajectory calculation anyways
  _done[threadIdx.x] = is >= *ns ? true : false;
  
  __syncthreads();

  int this_nt = 1;

  double this_result = 0;
  double last_result = 1;

  double result = -1;
  double dt_total = 0;
  for (int ilevel = 0; !_all_done && (ilevel <= *ml); ++ilevel) {

    // DeltaT inclusive up to this level
    dt_total = (*tstop - *tstart) / pow(2., ilevel+1);//(*tstop - *tstart) / (2 * this_nt);

    // deltaT this level and Time start this level
    double const dt = (*tstop - *tstart) / pow(2., ilevel);//(*tstop - *tstart) / this_nt;
    double const ts = *tstart + (*tstop - *tstart) / pow(2., ilevel + 1);//*tstart + (*tstop - *tstart) / (2. * this_nt);

    int const NTrajectoryBlocks = this_nt / blockDim.x + (this_nt % blockDim.x == 0 ? 0 : 1);

    for (int itb = 0; itb < NTrajectoryBlocks; ++itb) {
      _t[threadIdx.x] = dt * (itb * blockDim.x + threadIdx.x) + ts;

      if (_t[threadIdx.x] < *tstop) {
        int imin, imax;
        GetInterpolatingIMinIMax(t, nt, &(_t[threadIdx.x]), &imin, &imax);

        _x[threadIdx.x]  = Interpolate(t,  x,  xp, &(_t[threadIdx.x]), &imin, &imax);
        _y[threadIdx.x]  = Interpolate(t,  y,  yp, &(_t[threadIdx.x]), &imin, &imax);
        _z[threadIdx.x]  = Interpolate(t,  z,  zp, &(_t[threadIdx.x]), &imin, &imax);
        _bx[threadIdx.x] = Interpolate(t, bx, bxp, &(_t[threadIdx.x]), &imin, &imax);
        _by[threadIdx.x] = Interpolate(t, by, byp, &(_t[threadIdx.x]), &imin, &imax);
        _bz[threadIdx.x] = Interpolate(t, bz, bzp, &(_t[threadIdx.x]), &imin, &imax);
        _ax[threadIdx.x] = Interpolate(t, ax, axp, &(_t[threadIdx.x]), &imin, &imax);
        _ay[threadIdx.x] = Interpolate(t, ay, ayp, &(_t[threadIdx.x]), &imin, &imax);
        _az[threadIdx.x] = Interpolate(t, az, azp, &(_t[threadIdx.x]), &imin, &imax);
      }

      __syncthreads();

      if (!_done[threadIdx.x]) {
      for (int i = 0; i < blockDim.x; ++i) {

        // Check if we are over the limit of trajectory points
        if (is < *ns && (_t[i] < *tstop)) {

        // DO MATH HERE
        // Distance to observer
        double const D = sqrt( pow( (ox) - _x[i], 2) + pow( (oy) - _y[i], 2) + pow((oz) - _z[i], 2) );

        // Normal in direction of observer
        double const NX = ((ox) - _x[i]) / D;
        double const NY = ((oy) - _y[i]) / D;
        double const NZ = ((oz) - _z[i]) / D;

        // Magnitude of Beta squared
        double const One_Minus_BMag2 = 1. -  (_bx[i] * _bx[i] + _by[i] * _by[i] + _bz[i] * _bz[i]);

        // N dot Beta
        double const NDotBeta = NX * _bx[i] + NY * _by[i] + NZ * _bz[i];

        double const FarFieldDenominator =  D * (pow(1. - NDotBeta, 2));
        double const NearFieldDenominator = D * FarFieldDenominator;
        double const NearField_X = One_Minus_BMag2 * (NX - _bx[i]) / NearFieldDenominator;
        double const NearField_Y = One_Minus_BMag2 * (NY - _by[i]) / NearFieldDenominator;
        double const NearField_Z = One_Minus_BMag2 * (NZ - _bz[i]) / NearFieldDenominator;

        double const FFX = (NY - _by[i]) * _az[i] - (NZ - _bz[i]) * _ay[i];
        double const FFY = (NZ - _bz[i]) * _ax[i] - (NX - _bx[i]) * _az[i];
        double const FFZ = (NX - _bx[i]) * _ay[i] - (NY - _by[i]) * _ax[i];

        double const FarField_X = (NY * FFZ - NZ * FFY) / FarFieldDenominator;
        double const FarField_Y = (NZ * FFX - NX * FFZ) / FarFieldDenominator;
        double const FarField_Z = (NX * FFY - NY * FFX) / FarFieldDenominator;


        // Exponent for fourier transformed field
        hipDoubleComplex Exponent = make_hipDoubleComplex(0, -(*Omega) * (_t[i] + D / (*C)));

        hipDoubleComplex X1 = make_hipDoubleComplex(NearField_X + FarField_X, 0);
        hipDoubleComplex Y1 = make_hipDoubleComplex(NearField_Y + FarField_Y, 0);
        hipDoubleComplex Z1 = make_hipDoubleComplex(NearField_Z + FarField_Z, 0);

        hipDoubleComplex MyEXP = cuCexp(Exponent);

        hipDoubleComplex X2 = hipCmul(X1, MyEXP);
        hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
        hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);

        SumEX = hipCadd(SumEX, X2);
        SumEY = hipCadd(SumEY, Y2);
        SumEZ = hipCadd(SumEZ, Z2);

        }
      }
      }

    }

    if (!_done[threadIdx.x]) {
    hipDoubleComplex TSumEX = hipCmul(make_hipDoubleComplex((*C0) * (dt_total), 0), SumEX);
    hipDoubleComplex TSumEY = hipCmul(make_hipDoubleComplex((*C0) * (dt_total), 0), SumEY);
    hipDoubleComplex TSumEZ = hipCmul(make_hipDoubleComplex((*C0) * (dt_total), 0), SumEZ);

    double const EX = (TSumEX.x * TSumEX.x + TSumEX.y * TSumEX.y);
    double const EY = (TSumEY.x * TSumEY.x + TSumEY.y * TSumEY.y);
    double const EZ = (TSumEZ.x * TSumEZ.x + TSumEZ.y * TSumEZ.y);

    this_result = fabs((*C2) * (EX + EY + EZ));

    if (!_done[threadIdx.x] && (ilevel > 8) && (fabs((this_result - last_result) / last_result) < *prec) ) {
      _done[threadIdx.x] = true;
      result = this_result;
    }

    last_result = this_result;
    }


    __syncthreads();
    if (threadIdx.x == 1) {
      for (int ith = 0; ith < NTHREADS_PER_BLOCK; ++ith) {
        _all_done = true;
        if (!_done[ith]) {
          _all_done = false;
        }
      }
    }

    this_nt *= 2;

    __syncthreads();
  }

  if (is >= *ns) {
    return;
  }

  flux[ith] = (double) is; //result;

  return;
}








__global__ void OSCARSSR_Cuda_FluxGPUMultiWithA (double  *x, double  *y, double  *z,
                                                 double *bx, double *by, double *bz,
                                                 double *ax, double *ay, double *az,
                                                 double *sx, double *sy, double *sz,
                                                 double *dt,
                                                 int *nt,
                                                 int *ns,
                                                 double *C0, double *C2, double *C,
                                                 double *Omega,
                                                 int *ifirst,
                                                 double *flux)
{
  // Check that this is within the number of spectrum points requested
  int const ith = threadIdx.x + blockIdx.x * blockDim.x;
  int const is = ith + *ifirst;
  if (is >= *ns) {
    return;
  }


  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);


  // Loop over all points in trajectory
  for (int i = 0; i < *nt; ++i) {

    // Distance to observer
    double const D = sqrt( pow( (ox) - x[i], 2) + pow( (oy) - y[i], 2) + pow((oz) - z[i], 2) );

    // Normal in direction of observer
    double const NX = ((ox) - x[i]) / D;
    double const NY = ((oy) - y[i]) / D;
    double const NZ = ((oz) - z[i]) / D;

    // Magnitude of Beta squared
    double const One_Minus_BMag2 = 1. -  (bx[i] * bx[i] + by[i] * by[i] + bz[i] * bz[i]);

    // N dot Beta
    double const NDotBeta = NX * bx[i] + NY * by[i] + NZ * bz[i];

    double const FarFieldDenominator =  D * (pow(1. - NDotBeta, 2));
    double const NearFieldDenominator = D * FarFieldDenominator;
    double const NearField_X = One_Minus_BMag2 * (NX - bx[i]) / NearFieldDenominator;
    double const NearField_Y = One_Minus_BMag2 * (NY - by[i]) / NearFieldDenominator;
    double const NearField_Z = One_Minus_BMag2 * (NZ - bz[i]) / NearFieldDenominator;

    double const FFX = (NY - by[i]) * az[i] - (NZ - bz[i]) * ay[i];
    double const FFY = (NZ - bz[i]) * ax[i] - (NX - bx[i]) * az[i];
    double const FFZ = (NX - bx[i]) * ay[i] - (NY - by[i]) * ax[i];

    double const FarField_X = (NY * FFZ - NZ * FFY) / FarFieldDenominator;
    double const FarField_Y = (NZ * FFX - NX * FFZ) / FarFieldDenominator;
    double const FarField_Z = (NX * FFY - NY * FFX) / FarFieldDenominator;
    

    // Exponent for fourier transformed field
    hipDoubleComplex Exponent = make_hipDoubleComplex(0, -(*Omega) * ((*dt) * ((double) i) + D / (*C)));

    hipDoubleComplex X1 = make_hipDoubleComplex(NearField_X + FarField_X, 0);
    hipDoubleComplex Y1 = make_hipDoubleComplex(NearField_Y + FarField_Y, 0);
    hipDoubleComplex Z1 = make_hipDoubleComplex(NearField_Z + FarField_Z, 0);




    hipDoubleComplex MyEXP = cuCexp(Exponent);

    hipDoubleComplex X2 = hipCmul(X1, MyEXP);
    hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
    hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


    SumEX = hipCadd(SumEX, X2);
    SumEY = hipCadd(SumEY, Y2);
    SumEZ = hipCadd(SumEZ, Z2);

  }

  SumEX = hipCmul(make_hipDoubleComplex((*C0) * (*dt), 0), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex((*C0) * (*dt), 0), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex((*C0) * (*dt), 0), SumEZ);


  double const EX = (SumEX.x * SumEX.x + SumEX.y * SumEX.y);
  double const EY = (SumEY.x * SumEY.x + SumEY.y * SumEY.y);
  double const EZ = (SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y);

  flux[ith] = (*C2) * (EX + EY + EZ);

  return;
}








extern "C" void OSCARSSR_Cuda_CalculateFluxGPU (OSCARSSR& OSR,
                                                TSurfacePoints const& Surface,
                                                double const Energy_eV,
                                                T3DScalarContainer& FluxContainer,
                                                std::string const& Polarization,
                                                double const Angle,
                                                TVector3D const& HorizontalDirection,
                                                TVector3D const& PropogationDirection,
                                                int const NParticles,
                                                std::vector<int> const& GPUVector)
{
  // Calculate the flux for NParticles using the GPUs given in GPUVector.  Each particle's
  // trajectory will be sent to all GPUs for processing, meanwhile a new trajectory will
  // be calculated

  // Number of available GPUs
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Make sure that a gpu listed is within the range and not a duplicate
  std::vector<int> GPUsToUse;
  for (std::vector<int>::const_iterator it = GPUVector.begin(); it != GPUVector.end(); ++it) {
    if ( !(std::find(GPUsToUse.begin(), GPUsToUse.end(), *it) != GPUsToUse.end() && (*it < ngpu)) ) {
      GPUsToUse.push_back(*it);
    }
  }

  // Make sure we have at least one
  if (GPUsToUse.size() == 0) {
    throw std::invalid_argument("GPUs selected do not match hardware");
  }
  int const NGPUsToUse = (int) GPUsToUse.size();

  // Do we calculate for the current particle?
  bool const ThisParticleOnly = NParticles == 0 ? true : false;
  int  const NParticlesReally = ThisParticleOnly ? 1 : NParticles;
  // Type check, new particle if no type


  int *h_nt, *h_nt_max, *h_ns;
  double *h_dt;
  hipHostAlloc((void**) &h_nt_max, sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nt,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ns,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_dt,     sizeof(double), hipHostMallocWriteCombined | hipHostMallocMapped);

  // First one, set particle and trajectory
  if (!ThisParticleOnly) {
    OSR.SetNewParticle();
  }
  if (OSR.GetTrajectory().GetNPoints() == 0) {
    OSR.CalculateTrajectory();
  }

  // Needed number of points in the track and time step
  *h_nt_max = (int) OSR.GetNPointsTrajectory();
  *h_nt     = (int) OSR.GetTrajectory().GetNPoints();
  *h_ns     = (int) Surface.GetNPoints();
  *h_dt     = (double) OSR.GetTrajectory().GetDeltaT();


  int const NThreads = *h_ns;
  int const NThreadsPerBlock = NTHREADS_PER_BLOCK;
  int const NThreadsRemainder = NThreads % NThreadsPerBlock;
  int const NBlocksTotal = (NThreads - 1) / NThreadsPerBlock + 1;
  int const NBlocksPerGPU = NBlocksTotal / NGPUsToUse;
  int const NRemainderBlocks = NBlocksTotal % NGPUsToUse;
  // UPDATE: To be modified
  int const NFlux = NThreadsPerBlock * (NBlocksPerGPU + (NRemainderBlocks > 0 ? 1 : 0));

  std::vector<int> NBlocksThisGPU(NGPUsToUse, NBlocksPerGPU);
  for (int i = 0; i < NRemainderBlocks; ++i) {
    ++NBlocksThisGPU[i];
  }

  // Memory allocation for Host
  double  *h_x,  *h_y,  *h_z,  *h_bx,  *h_by,  *h_bz,  *h_sx,  *h_sy,  *h_sz,   *h_c0,  *h_c2,  *h_c,  *h_omega;
  int     *h_ifirst;
  double **h_flux;
  hipHostAlloc((void**) &h_x,       *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_y,       *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_z,       *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bx,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_by,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bz,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sx,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sy,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sz,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c0,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c2,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c,                   sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_omega,               sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ifirst, NGPUsToUse * sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_flux,   NGPUsToUse * sizeof(double*), hipHostMallocWriteCombined | hipHostMallocMapped);
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    hipHostAlloc((void**) &(h_flux[i]), NFlux * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  }

  // First surface point for each gpu
  int NBlocksUsed = 0;
  for (int i = 0; i < NGPUsToUse; ++i) {
    h_ifirst[i] = NBlocksUsed * NThreadsPerBlock;
    NBlocksUsed += NBlocksThisGPU[i];
  }


  // Memor allocations for GPU
  int    **d_nt;
  int    **d_ns;
  double **d_dt;
  double **d_x;
  double **d_y;
  double **d_z;
  double **d_bx;
  double **d_by;
  double **d_bz;
  double **d_sx;
  double **d_sy;
  double **d_sz;
  double **d_c0;
  double **d_c2;
  double **d_c;
  double **d_omega;
  int    **d_ifirst;
  double **d_flux;

  hipHostAlloc((void **) &d_nt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ns,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_dt,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_x,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_y,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_z,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_by,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sy,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c0,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c2,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_omega,  NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ifirst, NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_flux,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipMalloc((void **) &d_nt[i],                 sizeof(int));
    hipMalloc((void **) &d_ns[i],                 sizeof(int));
    hipMalloc((void **) &d_dt[i],                 sizeof(double));
    hipMalloc((void **) &d_x[i],      *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_y[i],      *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_z[i],      *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_bx[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_by[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_bz[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_sx[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sy[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sz[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_c0[i],                 sizeof(double));
    hipMalloc((void **) &d_c2[i],                 sizeof(double));
    hipMalloc((void **) &d_c[i],                  sizeof(double));
    hipMalloc((void **) &d_omega[i],              sizeof(double));
    hipMalloc((void **) &d_ifirst[i],             sizeof(int));
    hipMalloc((void **) &d_flux[i],       NFlux * sizeof(double));

    // Copy device number to device
    hipMemcpyAsync(d_ifirst[i], &(h_ifirst[i]), sizeof(int), hipMemcpyHostToDevice);
  }

  // Compute known host values
  *h_c0    = OSR.GetCurrentParticle().GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());
  *h_c2    = TOSCARSSR::FourPi() * OSR.GetCurrentParticle().GetCurrent() / (TOSCARSSR::H() * fabs(OSR.GetCurrentParticle().GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;
  *h_c     = TOSCARSSR::C();
  *h_omega = TOSCARSSR::EvToAngularFrequency(Energy_eV);
  for (size_t i = 0; i < *h_ns; ++i) {
    h_sx[i] = Surface.GetPoint(i).GetX();
    h_sy[i] = Surface.GetPoint(i).GetY();
    h_sz[i] = Surface.GetPoint(i).GetZ();
  }

  // Copy constants to first device (async)
  int const d0 = GPUsToUse[0];
  hipSetDevice(d0);
  hipMemcpyAsync(d_ns[0],    h_ns,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_dt[0],    h_dt,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c0[0],    h_c0,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c2[0],    h_c2,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c[0],     h_c,           sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_omega[0], h_omega,       sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sx[0],    h_sx,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sy[0],    h_sy,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sz[0],    h_sz,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_ns[i+1],     d1, d_ns[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_dt[i+1],     d1, d_dt[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c0[i+1],     d1, d_c0[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c2[i+1],     d1, d_c2[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c[i+1],      d1, d_c[i],      d, sizeof(double));
    hipMemcpyPeerAsync( d_omega[i+1],  d1, d_omega[i],  d, sizeof(double));
    hipMemcpyPeerAsync( d_sx[i+1],     d1, d_sx[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sy[i+1],     d1, d_sy[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sz[i+1],     d1, d_sz[i],     d, *h_ns * sizeof(double));
  }

  // Set first trajectory
  TParticleTrajectoryPoints const& T = OSR.GetTrajectory();
  int const NPointsThisTrajectory = T.GetNPoints();
  *h_nt = 0;
  for (size_t i = 0; i < NPointsThisTrajectory; ++i) {
    if (T.GetA(i).Mag() < 1e-100) {
      continue;
    }
    h_x[*h_nt]  = T.GetX(i).GetX();
    h_y[*h_nt]  = T.GetX(i).GetY();
    h_z[*h_nt]  = T.GetX(i).GetZ();
    h_bx[*h_nt] = T.GetB(i).GetX();
    h_by[*h_nt] = T.GetB(i).GetY();
    h_bz[*h_nt] = T.GetB(i).GetZ();
    ++(*h_nt);
  }
  hipSetDevice(d0);
  hipMemcpyAsync(d_nt[0],    h_nt,          sizeof(int),    hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_nt[i+1],     d1, d_nt[i],     d, sizeof(int));
  }


  // Set the surface points
  // GPU events
  hipEvent_t *event_fluxcopy = new hipEvent_t[NGPUsToUse];
  for (int ig = 0; ig < NGPUsToUse; ++ig) {
    int const d = GPUsToUse[ig];
    hipSetDevice(d);
    hipEventCreate(&(event_fluxcopy[ig]));
  }

  // Enable peer (direct gpu-gpu) writes
  for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
    // Device number
    int const d  = GPUsToUse[ig];
    int const d1 = GPUsToUse[ig+1];
    int access;
    hipDeviceCanAccessPeer(&access, d, d1);
    if (access == 1) {
      hipSetDevice(d);
      hipDeviceEnablePeerAccess(d1, 0);
    }
  }

  // Loop over number of particles
  for (int ip = 0; ip < NParticlesReally; ++ip) {

    // Copy trajectory to first GPU, then internal async transfers (where possible)
    hipSetDevice(d0);
    hipMemcpyAsync(d_nt[0], h_nt,         sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpyAsync(d_x[0],  h_x,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_y[0],  h_y,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_z[0],  h_z,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bx[0], h_bx, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_by[0], h_by, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bz[0], h_bz, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
      // Device number
      int const d  = GPUsToUse[ig];
      int const d1 = GPUsToUse[ig+1];
      hipSetDevice(d);
      hipMemcpyPeerAsync(d_nt[ig+1], d1, d_nt[ig], d,         sizeof(int));
      hipMemcpyPeerAsync(d_x[ig+1],  d1, d_x[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_y[ig+1],  d1, d_y[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_z[ig+1],  d1, d_z[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bx[ig+1], d1, d_bx[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_by[ig+1], d1, d_by[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bz[ig+1], d1, d_bz[ig], d, *h_nt * sizeof(double));

    }

    // Wait for previous copy, start next one
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d = GPUsToUse[ig];
      hipSetDevice(d);
      hipEventSynchronize(event_fluxcopy[ig]);
      OSCARSSR_Cuda_FluxGPUMulti<<<NBlocksThisGPU[ig], NThreadsPerBlock>>>(d_x[ig], d_y[ig], d_z[ig], d_bx[ig], d_by[ig], d_bz[ig], d_sx[ig], d_sy[ig], d_sz[ig], d_dt[ig], d_nt[ig], d_ns[ig], d_c0[ig], d_c2[ig], d_c[ig], d_omega[ig], d_ifirst[ig], d_flux[ig]);
    }


    // Add result to flux container (from **previous**)
    if (ip > 0) {
      int NBlocksUsed = 0;
      for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
        for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
          if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
            break;
          }
          int iss = ith + NThreadsPerBlock * NBlocksUsed;
          FluxContainer.AddToPoint(iss, h_flux[ig][ith]);
        }
        NBlocksUsed += NBlocksThisGPU[ig];
      }
    }

    // Add copy back to streams
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d  = GPUsToUse[ig];
      hipSetDevice(d);
      hipMemcpyAsync(h_flux[ig],  d_flux[ig],  NFlux * sizeof(double), hipMemcpyDeviceToHost);
      hipEventRecord(event_fluxcopy[ig]);
    }

    // If it's not the last one, calculate a new trajectory
    if (ip < NParticlesReally - 1) {
      OSR.SetNewParticle();
      OSR.CalculateTrajectory();
      TParticleTrajectoryPoints const& T = OSR.GetTrajectory();
      int const NPointsThisTrajectory = T.GetNPoints();

      *h_nt = 0;
      for (size_t it = 0; it < NPointsThisTrajectory; ++it) {
        if (T.GetA(it).Mag() < 1e-100) {
          continue;
        }
        h_x[*h_nt]  = T.GetX(it).GetX();
        h_y[*h_nt]  = T.GetX(it).GetY();
        h_z[*h_nt]  = T.GetX(it).GetZ();
        h_bx[*h_nt] = T.GetB(it).GetX();
        h_by[*h_nt] = T.GetB(it).GetY();
        h_bz[*h_nt] = T.GetB(it).GetZ();
        ++(*h_nt);
      }
    }
  }

  // Wait for last copy
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    hipEventSynchronize(event_fluxcopy[ig]);
  }

  // Add result to flux container (from **previous**)
  NBlocksUsed = 0;
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
      if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
        break;
      }
      int iss = ith + NThreadsPerBlock * NBlocksUsed;
      FluxContainer.AddToPoint(iss, h_flux[ig][ith]);
    }
    NBlocksUsed += NBlocksThisGPU[ig];
  }

  // Weighting for multi-particle
  double const Weight = 1.0 / (double) NParticlesReally;
  FluxContainer.WeightAll(Weight);

  // Free host memory
  hipHostFree(h_nt_max);
  hipHostFree(h_nt);
  hipHostFree(h_ns);
  hipHostFree(h_dt);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_bx);
  hipHostFree(h_by);
  hipHostFree(h_bz);
  hipHostFree(h_sx);
  hipHostFree(h_sy);
  hipHostFree(h_sz);
  hipHostFree(h_c0);
  hipHostFree(h_c2);
  hipHostFree(h_c);
  hipHostFree(h_omega);
  hipHostFree(h_ifirst);
  // Free host and GPU memory
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {

    hipHostFree(h_flux[i]);

    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipFree(d_nt[i]);
    hipFree(d_ns[i]);
    hipFree(d_dt[i]);
    hipFree(d_x[i]);
    hipFree(d_y[i]);
    hipFree(d_z[i]);
    hipFree(d_bx[i]);
    hipFree(d_by[i]);
    hipFree(d_bz[i]);
    hipFree(d_sx[i]);
    hipFree(d_sy[i]);
    hipFree(d_sz[i]);
    hipFree(d_c0[i]);
    hipFree(d_c2[i]);
    hipFree(d_c[i]);
    hipFree(d_omega[i]);
    hipFree(d_ifirst[i]);
    hipFree(d_flux[i]);
  }
  hipFree(h_flux);

  hipFree(d_nt);
  hipFree(d_ns);
  hipFree(d_dt);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);
  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);
  hipFree(d_c0);
  hipFree(d_c2);
  hipFree(d_c);
  hipFree(d_omega);
  hipFree(h_ifirst);
  hipFree(d_flux);

  // Delete host gpu pointer arrays
  delete [] event_fluxcopy;


  return;
}

















extern "C" void OSCARSSR_Cuda_CalculateFluxGPUWithA (OSCARSSR& OSR,
                                                TSurfacePoints const& Surface,
                                                double const Energy_eV,
                                                T3DScalarContainer& FluxContainer,
                                                std::string const& Polarization,
                                                double const Angle,
                                                TVector3D const& HorizontalDirection,
                                                TVector3D const& PropogationDirection,
                                                int const NParticles,
                                                std::vector<int> const& GPUVector)
{
  // Calculate the flux for NParticles using the GPUs given in GPUVector.  Each particle's
  // trajectory will be sent to all GPUs for processing, meanwhile a new trajectory will
  // be calculated

  // Number of available GPUs
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Make sure that a gpu listed is within the range and not a duplicate
  std::vector<int> GPUsToUse;
  for (std::vector<int>::const_iterator it = GPUVector.begin(); it != GPUVector.end(); ++it) {
    if ( !(std::find(GPUsToUse.begin(), GPUsToUse.end(), *it) != GPUsToUse.end() && (*it < ngpu)) ) {
      GPUsToUse.push_back(*it);
    }
  }

  // Make sure we have at least one
  if (GPUsToUse.size() == 0) {
    throw std::invalid_argument("GPUs selected do not match hardware");
  }
  int const NGPUsToUse = (int) GPUsToUse.size();

  // Do we calculate for the current particle?
  bool const ThisParticleOnly = NParticles == 0 ? true : false;
  int  const NParticlesReally = ThisParticleOnly ? 1 : NParticles;
  // Type check, new particle if no type


  int *h_nt, *h_nt_max, *h_ns;
  double *h_dt;
  hipHostAlloc((void**) &h_nt_max, sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nt,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ns,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_dt,     sizeof(double), hipHostMallocWriteCombined | hipHostMallocMapped);

  // First one, set particle and trajectory
  if (!ThisParticleOnly) {
    OSR.SetNewParticle();
  }
  if (OSR.GetTrajectory().GetNPoints() == 0) {
    OSR.CalculateTrajectory();
  }

  // Needed number of points in the track and time step
  *h_nt_max = (int) OSR.GetNPointsTrajectory();
  *h_nt     = (int) OSR.GetTrajectory().GetNPoints();
  *h_ns     = (int) Surface.GetNPoints();
  *h_dt     = (double) OSR.GetTrajectory().GetDeltaT();


  int const NThreads = *h_ns;
  int const NThreadsPerBlock = NTHREADS_PER_BLOCK;
  int const NThreadsRemainder = NThreads % NThreadsPerBlock;
  int const NBlocksTotal = (NThreads - 1) / NThreadsPerBlock + 1;
  int const NBlocksPerGPU = NBlocksTotal / NGPUsToUse;
  int const NRemainderBlocks = NBlocksTotal % NGPUsToUse;
  // UPDATE: To be modified
  int const NFlux = NThreadsPerBlock * (NBlocksPerGPU + (NRemainderBlocks > 0 ? 1 : 0));

  std::vector<int> NBlocksThisGPU(NGPUsToUse, NBlocksPerGPU);
  for (int i = 0; i < NRemainderBlocks; ++i) {
    ++NBlocksThisGPU[i];
  }

  // Memory allocation for Host
  double  *h_x,  *h_y,  *h_z,  *h_bx,  *h_by,  *h_bz,  *h_ax, *h_ay, *h_az, *h_sx,  *h_sy,  *h_sz,   *h_c0,  *h_c2,  *h_c,  *h_omega;
  int     *h_ifirst;
  double **h_flux;
  hipHostAlloc((void**) &h_x,       *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_y,       *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_z,       *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bx,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_by,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bz,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ax,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ay,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_az,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sx,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sy,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sz,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c0,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c2,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c,                   sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_omega,               sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ifirst, NGPUsToUse * sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_flux,   NGPUsToUse * sizeof(double*), hipHostMallocWriteCombined | hipHostMallocMapped);
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    hipHostAlloc((void**) &(h_flux[i]), NFlux * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  }

  // First surface point for each gpu
  int NBlocksUsed = 0;
  for (int i = 0; i < NGPUsToUse; ++i) {
    h_ifirst[i] = NBlocksUsed * NThreadsPerBlock;
    NBlocksUsed += NBlocksThisGPU[i];
  }


  // Memor allocations for GPU
  int    **d_nt;
  int    **d_ns;
  double **d_dt;
  double **d_x;
  double **d_y;
  double **d_z;
  double **d_bx;
  double **d_by;
  double **d_bz;
  double **d_ax;
  double **d_ay;
  double **d_az;
  double **d_sx;
  double **d_sy;
  double **d_sz;
  double **d_c0;
  double **d_c2;
  double **d_c;
  double **d_omega;
  int    **d_ifirst;
  double **d_flux;

  hipHostAlloc((void **) &d_nt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ns,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_dt,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_x,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_y,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_z,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_by,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ax,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ay,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_az,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sy,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c0,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c2,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_omega,  NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ifirst, NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_flux,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipMalloc((void **) &d_nt[i],                 sizeof(int));
    hipMalloc((void **) &d_ns[i],                 sizeof(int));
    hipMalloc((void **) &d_dt[i],                 sizeof(double));
    hipMalloc((void **) &d_x[i],      *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_y[i],      *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_z[i],      *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_bx[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_by[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_bz[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_ax[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_ay[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_az[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_sx[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sy[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sz[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_c0[i],                 sizeof(double));
    hipMalloc((void **) &d_c2[i],                 sizeof(double));
    hipMalloc((void **) &d_c[i],                  sizeof(double));
    hipMalloc((void **) &d_omega[i],              sizeof(double));
    hipMalloc((void **) &d_ifirst[i],             sizeof(int));
    hipMalloc((void **) &d_flux[i],       NFlux * sizeof(double));

    // Copy device number to device
    hipMemcpyAsync(d_ifirst[i], &(h_ifirst[i]), sizeof(int), hipMemcpyHostToDevice);
  }

  // Compute known host values
  *h_c0    = OSR.GetCurrentParticle().GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());
  *h_c2    = TOSCARSSR::FourPi() * OSR.GetCurrentParticle().GetCurrent() / (TOSCARSSR::H() * fabs(OSR.GetCurrentParticle().GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;
  *h_c     = TOSCARSSR::C();
  *h_omega = TOSCARSSR::EvToAngularFrequency(Energy_eV);
  for (size_t i = 0; i < *h_ns; ++i) {
    h_sx[i] = Surface.GetPoint(i).GetX();
    h_sy[i] = Surface.GetPoint(i).GetY();
    h_sz[i] = Surface.GetPoint(i).GetZ();
  }

  // Copy constants to first device (async)
  int const d0 = GPUsToUse[0];
  hipSetDevice(d0);
  hipMemcpyAsync(d_ns[0],    h_ns,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_dt[0],    h_dt,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c0[0],    h_c0,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c2[0],    h_c2,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c[0],     h_c,           sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_omega[0], h_omega,       sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sx[0],    h_sx,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sy[0],    h_sy,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sz[0],    h_sz,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_ns[i+1],     d1, d_ns[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_dt[i+1],     d1, d_dt[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c0[i+1],     d1, d_c0[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c2[i+1],     d1, d_c2[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c[i+1],      d1, d_c[i],      d, sizeof(double));
    hipMemcpyPeerAsync( d_omega[i+1],  d1, d_omega[i],  d, sizeof(double));
    hipMemcpyPeerAsync( d_sx[i+1],     d1, d_sx[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sy[i+1],     d1, d_sy[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sz[i+1],     d1, d_sz[i],     d, *h_ns * sizeof(double));
  }

  // Set first trajectory
  TParticleTrajectoryPoints const& T = OSR.GetTrajectory();
  int const NPointsThisTrajectory = T.GetNPoints();
  *h_nt = 0;
  for (size_t i = 0; i < NPointsThisTrajectory; ++i) {
    h_x[*h_nt]  = T.GetX(i).GetX();
    h_y[*h_nt]  = T.GetX(i).GetY();
    h_z[*h_nt]  = T.GetX(i).GetZ();
    h_bx[*h_nt] = T.GetB(i).GetX();
    h_by[*h_nt] = T.GetB(i).GetY();
    h_bz[*h_nt] = T.GetB(i).GetZ();
    h_ax[*h_nt] = T.GetAoverC(i).GetX();
    h_ay[*h_nt] = T.GetAoverC(i).GetY();
    h_az[*h_nt] = T.GetAoverC(i).GetZ();
    ++(*h_nt);
  }
  hipSetDevice(d0);
  hipMemcpyAsync(d_nt[0],    h_nt,          sizeof(int),    hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_nt[i+1],     d1, d_nt[i],     d, sizeof(int));
  }


  // Set the surface points
  // GPU events
  hipEvent_t *event_fluxcopy = new hipEvent_t[NGPUsToUse];
  for (int ig = 0; ig < NGPUsToUse; ++ig) {
    int const d = GPUsToUse[ig];
    hipSetDevice(d);
    hipEventCreate(&(event_fluxcopy[ig]));
  }

  // Enable peer (direct gpu-gpu) writes
  for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
    // Device number
    int const d  = GPUsToUse[ig];
    int const d1 = GPUsToUse[ig+1];
    int access;
    hipDeviceCanAccessPeer(&access, d, d1);
    if (access == 1) {
      hipSetDevice(d);
      hipDeviceEnablePeerAccess(d1, 0);
    }
  }

  // Loop over number of particles
  for (int ip = 0; ip < NParticlesReally; ++ip) {

    // Copy trajectory to first GPU, then internal async transfers (where possible)
    hipSetDevice(d0);
    hipMemcpyAsync(d_nt[0], h_nt,         sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpyAsync(d_x[0],  h_x,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_y[0],  h_y,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_z[0],  h_z,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bx[0], h_bx, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_by[0], h_by, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bz[0], h_bz, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ax[0], h_ax, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ay[0], h_ay, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_az[0], h_az, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
      // Device number
      int const d  = GPUsToUse[ig];
      int const d1 = GPUsToUse[ig+1];
      hipSetDevice(d);
      hipMemcpyPeerAsync(d_nt[ig+1], d1, d_nt[ig], d,         sizeof(int));
      hipMemcpyPeerAsync(d_x[ig+1],  d1, d_x[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_y[ig+1],  d1, d_y[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_z[ig+1],  d1, d_z[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bx[ig+1], d1, d_bx[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_by[ig+1], d1, d_by[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bz[ig+1], d1, d_bz[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ax[ig+1], d1, d_ax[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ay[ig+1], d1, d_ay[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_az[ig+1], d1, d_az[ig], d, *h_nt * sizeof(double));

    }

    // Wait for previous copy, start next one
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d = GPUsToUse[ig];
      hipSetDevice(d);
      hipEventSynchronize(event_fluxcopy[ig]);
      OSCARSSR_Cuda_FluxGPUMultiWithA<<<NBlocksThisGPU[ig], NThreadsPerBlock>>>( d_x[ig],  d_y[ig],  d_z[ig],
                                                                           d_bx[ig], d_by[ig], d_bz[ig],
                                                                           d_ax[ig], d_ay[ig], d_az[ig],
                                                                           d_sx[ig], d_sy[ig], d_sz[ig],
                                                                           d_dt[ig],
                                                                           d_nt[ig],
                                                                           d_ns[ig],
                                                                           d_c0[ig], d_c2[ig], d_c[ig],
                                                                           d_omega[ig],
                                                                           d_ifirst[ig],
                                                                           d_flux[ig]);
    }


    // Add result to flux container (from **previous**)
    if (ip > 0) {
      int NBlocksUsed = 0;
      for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
        for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
          if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
            break;
          }
          int iss = ith + NThreadsPerBlock * NBlocksUsed;
          FluxContainer.AddToPoint(iss, h_flux[ig][ith]);
        }
        NBlocksUsed += NBlocksThisGPU[ig];
      }
    }

    // Add copy back to streams
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d  = GPUsToUse[ig];
      hipSetDevice(d);
      hipMemcpyAsync(h_flux[ig],  d_flux[ig],  NFlux * sizeof(double), hipMemcpyDeviceToHost);
      hipEventRecord(event_fluxcopy[ig]);
    }

    // If it's not the last one, calculate a new trajectory
    if (ip < NParticlesReally - 1) {
      OSR.SetNewParticle();
      OSR.CalculateTrajectory();
      TParticleTrajectoryPoints const& T = OSR.GetTrajectory();
      int const NPointsThisTrajectory = T.GetNPoints();

      *h_nt = 0;
      for (size_t it = 0; it < NPointsThisTrajectory; ++it) {
        h_x[*h_nt]  = T.GetX(it).GetX();
        h_y[*h_nt]  = T.GetX(it).GetY();
        h_z[*h_nt]  = T.GetX(it).GetZ();
        h_bx[*h_nt] = T.GetB(it).GetX();
        h_by[*h_nt] = T.GetB(it).GetY();
        h_bz[*h_nt] = T.GetB(it).GetZ();
        h_ax[*h_nt] = T.GetAoverC(it).GetX();
        h_ay[*h_nt] = T.GetAoverC(it).GetY();
        h_az[*h_nt] = T.GetAoverC(it).GetZ();
        ++(*h_nt);
      }
    }
  }

  // Wait for last copy
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    hipEventSynchronize(event_fluxcopy[ig]);
  }

  // Add result to flux container (from **previous**)
  NBlocksUsed = 0;
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
      if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
        break;
      }
      int iss = ith + NThreadsPerBlock * NBlocksUsed;
      FluxContainer.AddToPoint(iss, h_flux[ig][ith]);
    }
    NBlocksUsed += NBlocksThisGPU[ig];
  }

  // Weighting for multi-particle
  double const Weight = 1.0 / (double) NParticlesReally;
  FluxContainer.WeightAll(Weight);

  // Free host memory
  hipHostFree(h_nt_max);
  hipHostFree(h_nt);
  hipHostFree(h_ns);
  hipHostFree(h_dt);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_bx);
  hipHostFree(h_by);
  hipHostFree(h_bz);
  hipHostFree(h_ax);
  hipHostFree(h_ay);
  hipHostFree(h_az);
  hipHostFree(h_sx);
  hipHostFree(h_sy);
  hipHostFree(h_sz);
  hipHostFree(h_c0);
  hipHostFree(h_c2);
  hipHostFree(h_c);
  hipHostFree(h_omega);
  hipHostFree(h_ifirst);
  // Free host and GPU memory
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {

    hipHostFree(h_flux[i]);

    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipFree(d_nt[i]);
    hipFree(d_ns[i]);
    hipFree(d_dt[i]);
    hipFree(d_x[i]);
    hipFree(d_y[i]);
    hipFree(d_z[i]);
    hipFree(d_bx[i]);
    hipFree(d_by[i]);
    hipFree(d_bz[i]);
    hipFree(d_ax[i]);
    hipFree(d_ay[i]);
    hipFree(d_az[i]);
    hipFree(d_sx[i]);
    hipFree(d_sy[i]);
    hipFree(d_sz[i]);
    hipFree(d_c0[i]);
    hipFree(d_c2[i]);
    hipFree(d_c[i]);
    hipFree(d_omega[i]);
    hipFree(d_ifirst[i]);
    hipFree(d_flux[i]);
  }
  hipFree(h_flux);

  hipFree(d_nt);
  hipFree(d_ns);
  hipFree(d_dt);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);
  hipFree(d_ax);
  hipFree(d_ay);
  hipFree(d_az);
  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);
  hipFree(d_c0);
  hipFree(d_c2);
  hipFree(d_c);
  hipFree(d_omega);
  hipFree(h_ifirst);
  hipFree(d_flux);

  // Delete host gpu pointer arrays
  delete [] event_fluxcopy;


  return;
}
























extern "C" void OSCARSSR_Cuda_CalculateFluxGPUWithAInterpolated (OSCARSSR& OSR,
                                                TSurfacePoints const& Surface,
                                                double const Energy_eV,
                                                T3DScalarContainer& FluxContainer,
                                                std::string const& Polarization,
                                                double const Angle,
                                                TVector3D const& HorizontalDirection,
                                                TVector3D const& PropogationDirection,
                                                int const NParticles,
                                                std::vector<int> const& GPUVector,
                                                double const Precision,
                                                int const MaxLevel)
{
  // Calculate the flux for NParticles using the GPUs given in GPUVector.  Each particle's
  // trajectory will be sent to all GPUs for processing, meanwhile a new trajectory will
  // be calculated

  // Number of available GPUs
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Make sure that a gpu listed is within the range and not a duplicate
  std::vector<int> GPUsToUse;
  for (std::vector<int>::const_iterator it = GPUVector.begin(); it != GPUVector.end(); ++it) {
    if ( !(std::find(GPUsToUse.begin(), GPUsToUse.end(), *it) != GPUsToUse.end() && (*it < ngpu)) ) {
      GPUsToUse.push_back(*it);
    }
  }

  // Make sure we have at least one
  if (GPUsToUse.size() == 0) {
    throw std::invalid_argument("GPUs selected do not match hardware");
  }
  int const NGPUsToUse = (int) GPUsToUse.size();

  // Do we calculate for the current particle?
  bool const ThisParticleOnly = NParticles == 0 ? true : false;
  int  const NParticlesReally = ThisParticleOnly ? 1 : NParticles;
  // Type check, new particle if no type

  // nt - number of interpolated track points
  // t  - array of timestamps in trajectory
  // x, y, z; xp, yp, zp - position and derivs
  // beta x, y, z; xp, yp, zp
  // aOc x, y, z; xp, yp, zp
  // max_level_extended or max_level
  // surface points
  // nthreads_per_block

  int *h_ns;
  int *h_nt;
  hipHostAlloc((void**) &h_ns,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nt,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);

  // First one, set particle and trajectory
  if (!ThisParticleOnly) {
    OSR.SetNewParticle();
  }
  if (OSR.GetTrajectory().GetNPoints() == 0) {
    OSR.CalculateTrajectory();
  }

  // Needed number of points in the track and time step
  *h_nt     = (int) OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetNPoints();
  *h_ns     = (int) Surface.GetNPoints();


  int const NThreads = *h_ns;
  int const NThreadsPerBlock = NTHREADS_PER_BLOCK;
  int const NThreadsRemainder = NThreads % NThreadsPerBlock;
  int const NBlocksTotal = (NThreads - 1) / NThreadsPerBlock + 1;
  int const NBlocksPerGPU = NBlocksTotal / NGPUsToUse;
  int const NRemainderBlocks = NBlocksTotal % NGPUsToUse;
  // UPDATE: To be modified
  int const NFlux = NThreadsPerBlock * (NBlocksPerGPU + (NRemainderBlocks > 0 ? 1 : 0));

  std::vector<int> NBlocksThisGPU(NGPUsToUse, NBlocksPerGPU);
  for (int i = 0; i < NRemainderBlocks; ++i) {
    ++NBlocksThisGPU[i];
  }

  // Interpolating structure
  double  *h_t;
  double  *h_tstart, *h_tstop;
  double  *h_x,   *h_y,   *h_z;
  double  *h_xp,  *h_yp,  *h_zp;
  double  *h_bx,  *h_by,  *h_bz;
  double  *h_bxp, *h_byp, *h_bzp;
  double  *h_ax,  *h_ay,  *h_az;
  double  *h_axp, *h_ayp, *h_azp;

  // Surface points
  double  *h_sx,  *h_sy,  *h_sz;

  // Constants and photon frequency
  double *h_c0,  *h_c2,  *h_c,  *h_omega;

  // first point for each thread, max level
  int     *h_ifirst;
  int     *h_ml;

  double *h_prec;

  // Results
  double **h_flux;

  // Allocate host memory
  hipHostAlloc((void**) &h_t,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_tstart,          sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_tstop,           sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_x,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_y,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_z,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_xp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_yp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_zp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bx,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_by,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bz,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bxp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_byp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bzp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ax,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ay,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_az,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_axp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ayp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_azp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_sx,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sy,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sz,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_c0,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c2,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c,                   sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_omega,               sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ifirst, NGPUsToUse * sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ml,                  sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_prec,                sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_flux,   NGPUsToUse * sizeof(double*), hipHostMallocWriteCombined | hipHostMallocMapped);
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    hipHostAlloc((void**) &(h_flux[i]), NFlux * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  }

  // First surface point for each gpu
  int NBlocksUsed = 0;
  for (int i = 0; i < NGPUsToUse; ++i) {
    h_ifirst[i] = NBlocksUsed * NThreadsPerBlock;
    NBlocksUsed += NBlocksThisGPU[i];
  }

  *h_ml = MaxLevel; //UPDATE: max level should be an input

  // Precision
  *h_prec = Precision;

  // Memor allocations for GPU
  int    **d_nt;
  double **d_tstart;
  double **d_tstop;
  int    **d_ns;

  double **d_t;

  double **d_x;
  double **d_y;
  double **d_z;
  double **d_xp;
  double **d_yp;
  double **d_zp;

  double **d_bx;
  double **d_by;
  double **d_bz;
  double **d_bxp;
  double **d_byp;
  double **d_bzp;

  double **d_ax;
  double **d_ay;
  double **d_az;
  double **d_axp;
  double **d_ayp;
  double **d_azp;

  double **d_sx;
  double **d_sy;
  double **d_sz;

  double **d_c0;
  double **d_c2;
  double **d_c;
  double **d_omega;

  int    **d_ifirst;
  int    **d_ml;
  double **d_prec;
  double **d_flux;

  hipHostAlloc((void **) &d_nt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_tstart, NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_tstop,  NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ns,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_t,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_x,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_y,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_z,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_xp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_yp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_zp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_bx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_by,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bxp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_byp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bzp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_ax,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ay,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_az,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_axp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ayp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_azp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_sx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sy,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_c0,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c2,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_omega,  NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_ifirst, NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ml,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_prec,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_flux,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipMalloc((void **) &d_nt[i],                 sizeof(int));
    hipMalloc((void **) &d_tstart[i],             sizeof(double));
    hipMalloc((void **) &d_tstop[i],              sizeof(double));
    hipMalloc((void **) &d_ns[i],                 sizeof(int));

    hipMalloc((void **) &d_t[i],          *h_nt * sizeof(double));

    hipMalloc((void **) &d_x[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_y[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_z[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_xp[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_yp[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_zp[i],         *h_nt * sizeof(double));

    hipMalloc((void **) &d_bx[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_by[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_bz[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_bxp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_byp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_bzp[i],        *h_nt * sizeof(double));

    hipMalloc((void **) &d_ax[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_ay[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_az[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_axp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_ayp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_azp[i],        *h_nt * sizeof(double));

    hipMalloc((void **) &d_sx[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sy[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sz[i],         *h_ns * sizeof(double));

    hipMalloc((void **) &d_c0[i],                 sizeof(double));
    hipMalloc((void **) &d_c2[i],                 sizeof(double));
    hipMalloc((void **) &d_c[i],                  sizeof(double));
    hipMalloc((void **) &d_omega[i],              sizeof(double));

    hipMalloc((void **) &d_ifirst[i],             sizeof(int));
    hipMalloc((void **) &d_ml[i],                 sizeof(int));
    hipMalloc((void **) &d_prec[i],               sizeof(double));
    hipMalloc((void **) &d_flux[i],       NFlux * sizeof(double));

    // Copy device number to device
    hipMemcpyAsync(d_ifirst[i], &(h_ifirst[i]), sizeof(int), hipMemcpyHostToDevice);
  }

  // Compute known host values
  *h_c0    = OSR.GetCurrentParticle().GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());
  *h_c2    = TOSCARSSR::FourPi() * OSR.GetCurrentParticle().GetCurrent() / (TOSCARSSR::H() * fabs(OSR.GetCurrentParticle().GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;
  *h_c     = TOSCARSSR::C();
  *h_omega = TOSCARSSR::EvToAngularFrequency(Energy_eV);
  for (size_t i = 0; i < *h_ns; ++i) {
    h_sx[i] = Surface.GetPoint(i).GetX();
    h_sy[i] = Surface.GetPoint(i).GetY();
    h_sz[i] = Surface.GetPoint(i).GetZ();
  }

  // Copy constants to first device (async)
  int const d0 = GPUsToUse[0];
  hipSetDevice(d0);
  hipMemcpyAsync(d_ns[0],    h_ns,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c0[0],    h_c0,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c2[0],    h_c2,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c[0],     h_c,           sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_omega[0], h_omega,       sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sx[0],    h_sx,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sy[0],    h_sy,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sz[0],    h_sz,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_ml[0],    h_ml,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_prec[0],  h_prec,        sizeof(double), hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_ns[i+1],     d1, d_ns[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_c0[i+1],     d1, d_c0[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c2[i+1],     d1, d_c2[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c[i+1],      d1, d_c[i],      d, sizeof(double));
    hipMemcpyPeerAsync( d_omega[i+1],  d1, d_omega[i],  d, sizeof(double));
    hipMemcpyPeerAsync( d_sx[i+1],     d1, d_sx[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sy[i+1],     d1, d_sy[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sz[i+1],     d1, d_sz[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_ml[i+1],     d1, d_ml[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_prec[i+1],   d1, d_prec[i],   d, sizeof(double));
  }

  // Set first trajectory
  TOMATH::TSpline1D3<TParticleTrajectoryPoint> const& T = OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetSpline();
  int const NPointsThisTrajectory = T.GetNPoints();
  *h_nt = NPointsThisTrajectory;
  *h_tstart = T.GetXStart();
  *h_tstop  = T.GetXStop();
  for (size_t i = 0; i < NPointsThisTrajectory; ++i) {
    TParticleTrajectoryPoint const& P  = T.GetY(i);
    TParticleTrajectoryPoint const& PP = T.GetYPP(i);

    h_t[i]   = T.GetX(i);

    h_x[i]   =  P.GetX().GetX();
    h_y[i]   =  P.GetX().GetY();
    h_z[i]   =  P.GetX().GetZ();
    h_xp[i]  = PP.GetX().GetX();
    h_yp[i]  = PP.GetX().GetY();
    h_zp[i]  = PP.GetX().GetZ();

    h_bx[i]  =  P.GetB().GetX();
    h_by[i]  =  P.GetB().GetY();
    h_bz[i]  =  P.GetB().GetZ();
    h_bxp[i] = PP.GetB().GetX();
    h_byp[i] = PP.GetB().GetY();
    h_bzp[i] = PP.GetB().GetZ();

    h_ax[i]  =  P.GetAoverC().GetX();
    h_ay[i]  =  P.GetAoverC().GetY();
    h_az[i]  =  P.GetAoverC().GetZ();
    h_axp[i] = PP.GetAoverC().GetX();
    h_ayp[i] = PP.GetAoverC().GetY();
    h_azp[i] = PP.GetAoverC().GetZ();
  }


  // Set the surface points
  // GPU events
  hipEvent_t *event_fluxcopy = new hipEvent_t[NGPUsToUse];
  for (int ig = 0; ig < NGPUsToUse; ++ig) {
    int const d = GPUsToUse[ig];
    hipSetDevice(d);
    hipEventCreate(&(event_fluxcopy[ig]));
  }

  // Enable peer (direct gpu-gpu) writes
  for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
    // Device number
    int const d  = GPUsToUse[ig];
    int const d1 = GPUsToUse[ig+1];
    int access;
    hipDeviceCanAccessPeer(&access, d, d1);
    if (access == 1) {
      hipSetDevice(d);
      hipDeviceEnablePeerAccess(d1, 0);
    }
  }

  // Loop over number of particles
  for (int ip = 0; ip < NParticlesReally; ++ip) {

    // Copy trajectory to first GPU, then internal async transfers (where possible)
    hipSetDevice(d0);
    hipMemcpyAsync(d_nt[0],  h_nt,          sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpyAsync(d_t[0],   h_t,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_tstart[0], h_tstart,   sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_tstop[0],  h_tstop,    sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_x[0],   h_x,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_y[0],   h_y,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_z[0],   h_z,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_xp[0],  h_xp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_yp[0],  h_yp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_zp[0],  h_zp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bx[0],  h_bx,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_by[0],  h_by,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bz[0],  h_bz,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bxp[0], h_bxp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_byp[0], h_byp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bzp[0], h_bzp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ax[0],  h_ax,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ay[0],  h_ay,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_az[0],  h_az,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_axp[0], h_axp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ayp[0], h_ayp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_azp[0], h_azp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
      // Device number
      int const d  = GPUsToUse[ig];
      int const d1 = GPUsToUse[ig+1];
      hipSetDevice(d);
      hipMemcpyPeerAsync(d_nt[ig+1],  d1, d_nt[ig],  d,         sizeof(int));
      hipMemcpyPeerAsync(d_t[ig+1],   d1, d_t[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_tstart[ig+1], d1, d_tstart[ig], d,   sizeof(double));
      hipMemcpyPeerAsync(d_tstop[ig+1],  d1, d_tstop[ig],  d,    sizeof(double));
      hipMemcpyPeerAsync(d_x[ig+1],   d1, d_x[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_y[ig+1],   d1, d_y[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_z[ig+1],   d1, d_z[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_xp[ig+1],  d1, d_xp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_yp[ig+1],  d1, d_yp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_zp[ig+1],  d1, d_zp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bx[ig+1],  d1, d_bx[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_by[ig+1],  d1, d_by[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bz[ig+1],  d1, d_bz[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bxp[ig+1], d1, d_bxp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_byp[ig+1], d1, d_byp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bzp[ig+1], d1, d_bzp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ax[ig+1],  d1, d_ax[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ay[ig+1],  d1, d_ay[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_az[ig+1],  d1, d_az[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_axp[ig+1], d1, d_axp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ayp[ig+1], d1, d_ayp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_azp[ig+1], d1, d_azp[ig], d, *h_nt * sizeof(double));

    }

    // Wait for previous copy, start next one
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d = GPUsToUse[ig];
      hipSetDevice(d);
      hipEventSynchronize(event_fluxcopy[ig]);
      OSCARSSR_Cuda_FluxGPUMultiWithAInterpolated<<<NBlocksThisGPU[ig], NThreadsPerBlock>>>(
                                                                           d_t[ig],
                                                                           d_x[ig],   d_y[ig],   d_z[ig],
                                                                           d_xp[ig],  d_yp[ig],  d_zp[ig],
                                                                           d_bx[ig],  d_by[ig],  d_bz[ig],
                                                                           d_bxp[ig], d_byp[ig], d_bzp[ig],
                                                                           d_ax[ig],  d_ay[ig],  d_az[ig],
                                                                           d_axp[ig], d_ayp[ig], d_azp[ig],
                                                                           d_sx[ig],  d_sy[ig],  d_sz[ig],
                                                                           d_tstart[ig], d_tstop[ig],
                                                                           d_nt[ig],
                                                                           d_ns[ig],
                                                                           d_c0[ig], d_c2[ig], d_c[ig],
                                                                           d_omega[ig],
                                                                           d_ifirst[ig],
                                                                           d_ml[ig],
                                                                           d_prec[ig],
                                                                           d_flux[ig]);
    }


    // Add result to flux container (from **previous**)
    if (ip > 0) {
      int NBlocksUsed = 0;
      for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
        for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
          if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
            break;
          }
          int iss = ith + NThreadsPerBlock * NBlocksUsed;
          FluxContainer.AddToPoint(iss, h_flux[ig][ith]);
        }
        NBlocksUsed += NBlocksThisGPU[ig];
      }
    }

    // Add copy back to streams
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d  = GPUsToUse[ig];
      hipSetDevice(d);
      hipMemcpyAsync(h_flux[ig],  d_flux[ig],  NFlux * sizeof(double), hipMemcpyDeviceToHost);
      hipEventRecord(event_fluxcopy[ig]);
    }

    // If it's not the last one, calculate a new trajectory
    if (ip < NParticlesReally - 1) {
      OSR.SetNewParticle();
      OSR.CalculateTrajectory();

      TOMATH::TSpline1D3<TParticleTrajectoryPoint> const& T = OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetSpline();
      int const NPointsThisTrajectory = T.GetNPoints();
      *h_nt = NPointsThisTrajectory;
      for (size_t it = 0; it < NPointsThisTrajectory; ++it) {
        TParticleTrajectoryPoint const& P  = T.GetY(it);
        TParticleTrajectoryPoint const& PP = T.GetYPP(it);

        h_t[it]   = T.GetX(it);

        h_x[it]   =  P.GetX().GetX();
        h_y[it]   =  P.GetX().GetY();
        h_z[it]   =  P.GetX().GetZ();
        h_xp[it]  = PP.GetX().GetX();
        h_yp[it]  = PP.GetX().GetY();
        h_zp[it]  = PP.GetX().GetZ();

        h_bx[it]  =  P.GetB().GetX();
        h_by[it]  =  P.GetB().GetY();
        h_bz[it]  =  P.GetB().GetZ();
        h_bxp[it] = PP.GetB().GetX();
        h_byp[it] = PP.GetB().GetY();
        h_bzp[it] = PP.GetB().GetZ();

        h_ax[it]  =  P.GetAoverC().GetX();
        h_ay[it]  =  P.GetAoverC().GetY();
        h_az[it]  =  P.GetAoverC().GetZ();
        h_axp[it] = PP.GetAoverC().GetX();
        h_ayp[it] = PP.GetAoverC().GetY();
        h_azp[it] = PP.GetAoverC().GetZ();
      }
    }
  }

  // Wait for last copy
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    hipEventSynchronize(event_fluxcopy[ig]);
  }

  // Add result to flux container (from **previous**)
  NBlocksUsed = 0;
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
      if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
        break;
      }
      int iss = ith + NThreadsPerBlock * NBlocksUsed;
      FluxContainer.AddToPoint(iss, h_flux[ig][ith]);
    }
    NBlocksUsed += NBlocksThisGPU[ig];
  }

  // Weighting for multi-particle
  double const Weight = 1.0 / (double) NParticlesReally;
  FluxContainer.WeightAll(Weight);

  // Free host memory
  hipHostFree(h_nt);
  hipHostFree(h_t);
  hipHostFree(h_ns);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_xp);
  hipHostFree(h_yp);
  hipHostFree(h_zp);
  hipHostFree(h_bx);
  hipHostFree(h_by);
  hipHostFree(h_bz);
  hipHostFree(h_bxp);
  hipHostFree(h_byp);
  hipHostFree(h_bzp);
  hipHostFree(h_ax);
  hipHostFree(h_ay);
  hipHostFree(h_az);
  hipHostFree(h_axp);
  hipHostFree(h_ayp);
  hipHostFree(h_azp);
  hipHostFree(h_sx);
  hipHostFree(h_sy);
  hipHostFree(h_sz);
  hipHostFree(h_c0);
  hipHostFree(h_c2);
  hipHostFree(h_c);
  hipHostFree(h_omega);
  hipHostFree(h_ifirst);
  hipHostFree(h_ml);
  hipHostFree(h_prec);
  // Free host and GPU memory
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {

    hipHostFree(h_flux[i]);

    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipFree(d_nt[i]);
    hipFree(d_t[i]);
    hipFree(d_ns[i]);
    hipFree(d_x[i]);
    hipFree(d_y[i]);
    hipFree(d_z[i]);
    hipFree(d_xp[i]);
    hipFree(d_yp[i]);
    hipFree(d_zp[i]);
    hipFree(d_bx[i]);
    hipFree(d_by[i]);
    hipFree(d_bz[i]);
    hipFree(d_bxp[i]);
    hipFree(d_byp[i]);
    hipFree(d_bzp[i]);
    hipFree(d_ax[i]);
    hipFree(d_ay[i]);
    hipFree(d_az[i]);
    hipFree(d_axp[i]);
    hipFree(d_ayp[i]);
    hipFree(d_azp[i]);
    hipFree(d_sx[i]);
    hipFree(d_sy[i]);
    hipFree(d_sz[i]);
    hipFree(d_c0[i]);
    hipFree(d_c2[i]);
    hipFree(d_c[i]);
    hipFree(d_omega[i]);
    hipFree(d_ifirst[i]);
    hipFree(d_ml[i]);
    hipFree(d_prec[i]);
    hipFree(d_flux[i]);
  }
  hipFree(h_flux);

  hipFree(d_nt);
  hipFree(d_t);
  hipFree(d_ns);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_xp);
  hipFree(d_yp);
  hipFree(d_zp);
  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);
  hipFree(d_bxp);
  hipFree(d_byp);
  hipFree(d_bzp);
  hipFree(d_ax);
  hipFree(d_ay);
  hipFree(d_az);
  hipFree(d_axp);
  hipFree(d_ayp);
  hipFree(d_azp);
  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);
  hipFree(d_c0);
  hipFree(d_c2);
  hipFree(d_c);
  hipFree(d_omega);
  hipFree(h_ifirst);
  hipFree(d_ml);
  hipFree(d_prec);
  hipFree(d_flux);

  // Delete host gpu pointer arrays
  delete [] event_fluxcopy;


  return;
}




























































__global__ void OSCARSSR_Cuda_SpectrumGPUMulti (double *x, double *y, double *z, double *bx, double *by, double *bz, double *obs, double *dt, int *nt, int *ns, double *C0, double *C2, double *EvToOmega, double *C, double *se, double *sf, hipDoubleComplex* pol, int *pol_state, int const *ifirst)
{
  // Check that this is within the number of spectrum points requested
  int const ith = threadIdx.x + blockIdx.x * blockDim.x;
  int const is = ith + *ifirst;
  if (is >= *ns) {
    return;
  }

  // Complex i
  hipDoubleComplex I = make_hipDoubleComplex(0, 1);

  double const Omega = *EvToOmega * se[is];
  hipDoubleComplex ICoverOmega = make_hipDoubleComplex(0, (*C) / Omega);


  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);


  // Loop over all points in trajectory
  for (int i = 0; i < *nt; ++i) {

    // Distance to observer
    double const D = sqrt( pow( obs[0] - x[i], 2) + pow( obs[1] - y[i], 2) + pow(obs[2] - z[i], 2) );

    // Normal in direction of observer
    double const NX = (obs[0] - x[i]) / D;
    double const NY = (obs[1] - y[i]) / D;
    double const NZ = (obs[2] - z[i]) / D;

    // Exponent for fourier transformed field
    hipDoubleComplex Exponent = make_hipDoubleComplex(0, Omega * ((*dt) * i + D / (*C)));

    hipDoubleComplex X1 = make_hipDoubleComplex((bx[i] - NX) / D, -(*C) * NX / (Omega * D * D));
    hipDoubleComplex Y1 = make_hipDoubleComplex((by[i] - NY) / D, -(*C) * NY / (Omega * D * D));
    hipDoubleComplex Z1 = make_hipDoubleComplex((bz[i] - NZ) / D, -(*C) * NZ / (Omega * D * D));

    hipDoubleComplex MyEXP = cuCexp(Exponent);
    //hipDoubleComplex MyEXP = make_hipDoubleComplex( exp(Exponent.x) * cos(Exponent.y), exp(Exponent.x) * sin(Exponent.y));

    hipDoubleComplex X2 = hipCmul(X1, MyEXP);
    hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
    hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


    SumEX = hipCadd(SumEX, X2);
    SumEY = hipCadd(SumEY, Y2);
    SumEZ = hipCadd(SumEZ, Z2);

    // Sum in fourier transformed field (integral)
    //SumEX += (TVector3DC(B) - (N *     (One + (ICoverOmega / (D)))     )) / D * std::exp(Exponent);
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * Omega * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * Omega * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * Omega * (*dt)), SumEZ);


  // Check for polarization state
  if (*pol_state == 0) {
    // Do nothing
  } else if (*pol_state == 1) {
    // Linear, just dot with vector and put in direction of vector
    hipDoubleComplex Magnitude = hipCadd(hipCadd(hipCmul(SumEX, pol[0]), hipCmul(SumEY, pol[1])),  hipCmul(SumEZ, pol[2]));
    SumEX = hipCmul(Magnitude, pol[0]);
    SumEY = hipCmul(Magnitude, pol[1]);
    SumEZ = hipCmul(Magnitude, pol[2]);
  } else if (*pol_state == 2) {
    hipDoubleComplex Magnitude = hipCadd(hipCadd(hipCmul(SumEX, hipConj(pol[0])), hipCmul(SumEY, hipConj(pol[1]))),  hipCmul(SumEZ, hipConj(pol[2])));
    SumEX = hipCmul(Magnitude, pol[0]);
    SumEY = hipCmul(Magnitude, pol[1]);
    SumEZ = hipCmul(Magnitude, pol[2]);
  } else {
    // UPDATE: Serious problem
  }


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;

  // Multiply field by Constant C1 and time step
  //SumE *= C1 * DeltaT;

  // Set the flux for this frequency / energy point
  //Spectrum.AddToFlux(i, C2 *  SumE.Dot( SumE.CC() ).real() * Weight);

  sf[ith] = (*C2) * (EX + EY + EZ);

  return;
}








extern "C" void OSCARSSR_Cuda_CalculateSpectrumGPU (OSCARSSR& OSR,
                                                    TParticleA& Particle,
                                                    TVector3D const& ObservationPoint,
                                                    TSpectrumContainer& Spectrum,
                                                    std::string const& Polarization,
                                                    double const Angle,
                                                    TVector3D const& HorizontalDirection,
                                                    TVector3D const& PropogationDirection,
                                                    int const NParticles,
                                                    std::vector<int> const& GPUVector)
{
  // Calculate the spectrum for NParticles using the GPUs given in GPUVector.  Each particle's
  // trajectory will be sent to all GPUs for processing, meanwhile a new trajectory will
  // be calculated

  // Number of available GPUs
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Make sure that a gpu listed is within the range and not a duplicate
  std::vector<int> GPUsToUse;
  for (std::vector<int>::const_iterator it = GPUVector.begin(); it != GPUVector.end(); ++it) {
    if ( !(std::find(GPUsToUse.begin(), GPUsToUse.end(), *it) != GPUsToUse.end() && (*it < ngpu)) ) {
      GPUsToUse.push_back(*it);
    }
  }

  // Make sure we have at least one
  if (GPUsToUse.size() == 0) {
    throw std::invalid_argument("GPUs selected do not match hardware");
  }
  int const NGPUsToUse = (int) GPUsToUse.size();

  // Do we calculate for the current particle?
  bool const ThisParticleOnly = NParticles == 0 ? true : false;
  int  const NParticlesReally = ThisParticleOnly ? 1 : NParticles;
  // Type check, new particle if no type


  // Imaginary "i" and complxe 1+0i
  std::complex<double> const I(0, 1);
  std::complex<double> const One(1, 0);

  // Photon vertical direction and positive and negative helicity
  TVector3D const VerticalDirection = PropogationDirection.Cross(HorizontalDirection).UnitVector();
  TVector3DC const Positive = 1. / sqrt(2) * (TVector3DC(HorizontalDirection) + VerticalDirection * I );
  TVector3DC const Negative = 1. / sqrt(2) * (TVector3DC(HorizontalDirection) - VerticalDirection * I );

  // For polarization input to the gpu
  hipDoubleComplex *pol = new hipDoubleComplex[3];

  // State of polarization: 0 for all, 1 for linear, 2 for circular
  // (requires different threatment of vector pol interally)
  int pol_state = 1;

  if (Polarization == "all") {
    // Do nothing, it is already ALL
    pol_state = 0;
  } else if (Polarization == "linear-horizontal") {
    pol[0] = make_hipDoubleComplex(HorizontalDirection.GetX(), 0);
    pol[1] = make_hipDoubleComplex(HorizontalDirection.GetY(), 0);
    pol[2] = make_hipDoubleComplex(HorizontalDirection.GetZ(), 0);
  } else if (Polarization == "linear-vertical") {
    pol[0] = make_hipDoubleComplex(VerticalDirection.GetX(), 0);
    pol[1] = make_hipDoubleComplex(VerticalDirection.GetY(), 0);
    pol[2] = make_hipDoubleComplex(VerticalDirection.GetZ(), 0);
  } else if (Polarization == "linear") {
    TVector3D PolarizationAngle = HorizontalDirection;
    PolarizationAngle.RotateSelf(Angle, PropogationDirection);
    pol[0] = make_hipDoubleComplex(PolarizationAngle.GetX(), 0);
    pol[1] = make_hipDoubleComplex(PolarizationAngle.GetY(), 0);
    pol[2] = make_hipDoubleComplex(PolarizationAngle.GetZ(), 0);
  } else if (Polarization == "circular-left") {
    //SumE = SumE.Dot(Positive.CC()) * Positive;
    pol_state = 2;
    pol[0] = make_hipDoubleComplex(Positive.CC().GetX().real(), Positive.CC().GetX().imag());
    pol[1] = make_hipDoubleComplex(Positive.CC().GetY().real(), Positive.CC().GetY().imag());
    pol[2] = make_hipDoubleComplex(Positive.CC().GetZ().real(), Positive.CC().GetZ().imag());
  } else if (Polarization == "circular-right") {
    //SumE = SumE.Dot(Negative.CC()) * Negative;
    pol_state = 2;
    pol[0] = make_hipDoubleComplex(Negative.CC().GetX().real(), Negative.CC().GetX().imag());
    pol[1] = make_hipDoubleComplex(Negative.CC().GetY().real(), Negative.CC().GetY().imag());
    pol[2] = make_hipDoubleComplex(Negative.CC().GetZ().real(), Negative.CC().GetZ().imag());
  } else {
    // Throw invalid argument if polarization is not recognized
    //throw std::invalid_argument("Polarization requested not recognized");
  }

  int *h_nt, *h_nt_max, *h_ns;
  double *h_dt;
  hipHostAlloc((void**) &h_nt_max, sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nt,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ns,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_dt,     sizeof(double), hipHostMallocWriteCombined | hipHostMallocMapped);

  // First one, set particle and trajectory
  if (!ThisParticleOnly) {
    OSR.SetNewParticle();
  }
  if (OSR.GetTrajectory().GetNPoints() == 0) {
    OSR.CalculateTrajectory();
  }

  // Needed number of points in the track and time step
  *h_nt_max = (int) OSR.GetNPointsTrajectory();
  *h_nt     = (int) OSR.GetTrajectory().GetNPoints();
  *h_ns     = (int) Spectrum.GetNPoints();
  *h_dt     = (double) OSR.GetTrajectory().GetDeltaT();


  int const NThreads = *h_ns;
  int const NThreadsPerBlock = NTHREADS_PER_BLOCK;
  int const NThreadsRemainder = NThreads % NThreadsPerBlock;
  int const NBlocksTotal = (NThreads - 1) / NThreadsPerBlock + 1;
  int const NBlocksPerGPU = NBlocksTotal / NGPUsToUse;
  int const NRemainderBlocks = NBlocksTotal % NGPUsToUse;
  // UPDATE: To be modified
  int const NSpectrum = NThreadsPerBlock * (NBlocksPerGPU + (NRemainderBlocks > 0 ? 1 : 0));

  std::vector<int> NBlocksThisGPU(NGPUsToUse, NBlocksPerGPU);
  for (int i = 0; i < NRemainderBlocks; ++i) {
    ++NBlocksThisGPU[i];
  }

  // Memory allocation for Host
  double  *h_x,  *h_y,  *h_z,  *h_bx,  *h_by,  *h_bz,  *h_obs, *h_se,   *h_c0,  *h_c2,  *h_c,  *h_ev2omega;
  int     *h_ifirst, *h_pol_state;
  double **h_spectrum;
  hipDoubleComplex *h_pol;

  hipHostAlloc((void**) &h_x,       *h_nt_max * sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_y,       *h_nt_max * sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_z,       *h_nt_max * sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bx,      *h_nt_max * sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_by,      *h_nt_max * sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bz,      *h_nt_max * sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_obs,             3 * sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_se,          *h_ns * sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c0,                  sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c2,                  sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c,                   sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ev2omega,            sizeof(double),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_pol,                 sizeof(int),             hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_pol_state,       3 * sizeof(hipDoubleComplex), hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ifirst, NGPUsToUse * sizeof(int),             hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_spectrum, NGPUsToUse * sizeof(double*),       hipHostMallocWriteCombined | hipHostMallocMapped);
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    hipHostAlloc((void**) &(h_spectrum[i]), NSpectrum * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  }

  // First surface point for each gpu
  int NBlocksUsed = 0;
  for (int i = 0; i < NGPUsToUse; ++i) {
    h_ifirst[i] = NBlocksUsed * NThreadsPerBlock;
    NBlocksUsed += NBlocksThisGPU[i];
  }


  // Memor allocations for GPU
  int             **d_nt;
  int             **d_ns;
  double          **d_dt;
  double          **d_x;
  double          **d_y;
  double          **d_z;
  double          **d_bx;
  double          **d_by;
  double          **d_bz;
  double          **d_obs;
  double          **d_se;
  double          **d_c0;
  double          **d_c2;
  double          **d_c;
  double          **d_ev2omega;
  int             **d_ifirst;
  hipDoubleComplex **d_pol;
  int             **d_pol_state;
  double          **d_spectrum;

  hipHostAlloc((void **) &d_nt,        NGPUsToUse * sizeof(int*),             hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ns,        NGPUsToUse * sizeof(int*),             hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_dt,        NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_x,         NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_y,         NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_z,         NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bx,        NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_by,        NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bz,        NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_obs,       NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_se,        NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c0,        NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c2,        NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c,         NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ev2omega,  NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ifirst,    NGPUsToUse * sizeof(int*),             hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_pol,       NGPUsToUse * sizeof(hipDoubleComplex*), hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_pol_state, NGPUsToUse * sizeof(int*),             hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_spectrum,  NGPUsToUse * sizeof(double*),          hipHostMallocWriteCombined | hipHostMallocMapped);

  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipMalloc((void **) &d_nt[i],                   sizeof(int));
    hipMalloc((void **) &d_ns[i],                   sizeof(int));
    hipMalloc((void **) &d_dt[i],                   sizeof(double));
    hipMalloc((void **) &d_x[i],        *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_y[i],        *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_z[i],        *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_bx[i],       *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_by[i],       *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_bz[i],       *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_obs[i],              3 * sizeof(double));
    hipMalloc((void **) &d_se[i],           *h_ns * sizeof(double));
    hipMalloc((void **) &d_c0[i],                   sizeof(double));
    hipMalloc((void **) &d_c2[i],                   sizeof(double));
    hipMalloc((void **) &d_c[i],                    sizeof(double));
    hipMalloc((void **) &d_ev2omega[i],             sizeof(double));
    hipMalloc((void **) &d_ifirst[i],               sizeof(int));
    hipMalloc((void **) &d_pol[i],              3 * sizeof(hipDoubleComplex));
    hipMalloc((void **) &d_pol_state[i],            sizeof(int));
    hipMalloc((void **) &d_spectrum[i], NSpectrum * sizeof(double));

    // Copy device number to device
    hipMemcpyAsync(d_ifirst[i], &(h_ifirst[i]), sizeof(int), hipMemcpyHostToDevice);
  }

  // Compute known host values
  h_obs[0]     = ObservationPoint[0];
  h_obs[1]     = ObservationPoint[1];
  h_obs[2]     = ObservationPoint[2];
  *h_c0        = OSR.GetCurrentParticle().GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());
  *h_c2        = TOSCARSSR::FourPi() * OSR.GetCurrentParticle().GetCurrent() / (TOSCARSSR::H() * fabs(OSR.GetCurrentParticle().GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;
  *h_c         = TOSCARSSR::C();
  *h_ev2omega  = TOSCARSSR::EvToAngularFrequency(1);
  h_pol[0]     = pol[0];
  h_pol[1]     = pol[1];
  h_pol[2]     = pol[2];
  *h_pol_state = pol_state;
  for (size_t i = 0; i < *h_ns; ++i) {
    h_se[i] = Spectrum.GetEnergy(i);
  }

  // Copy constants to first device (async)
  int const d0 = GPUsToUse[0];
  hipSetDevice(d0);
  hipMemcpyAsync(d_nt[0],       h_nt,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_ns[0],       h_ns,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_dt[0],       h_dt,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_obs[0],      h_obs,     3 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c0[0],       h_c0,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c2[0],       h_c2,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c[0],        h_c,           sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_ev2omega[0], h_ev2omega,    sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_se[0],       h_se,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_pol[0],      h_pol,     3 * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_pol_state[0],h_pol_state,   sizeof(int),    hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_nt[i+1],        d1, d_nt[i],       d,         sizeof(int));
    hipMemcpyPeerAsync( d_ns[i+1],        d1, d_ns[i],       d,         sizeof(int));
    hipMemcpyPeerAsync( d_dt[i+1],        d1, d_dt[i],       d,         sizeof(double));
    hipMemcpyPeerAsync( d_obs[i+1],       d1, d_obs[i],      d,     3 * sizeof(double));
    hipMemcpyPeerAsync( d_c0[i+1],        d1, d_c0[i],       d,         sizeof(double));
    hipMemcpyPeerAsync( d_c2[i+1],        d1, d_c2[i],       d,         sizeof(double));
    hipMemcpyPeerAsync( d_c[i+1],         d1, d_c[i],        d,         sizeof(double));
    hipMemcpyPeerAsync( d_ev2omega[i+1],  d1, d_ev2omega[i], d,         sizeof(double));
    hipMemcpyPeerAsync( d_se[i+1],        d1, d_se[i],       d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_pol[i+1],       d1, d_pol[i],      d,     3 * sizeof(hipDoubleComplex));
    hipMemcpyPeerAsync( d_pol_state[i+1], d1, d_pol_state[i],d,         sizeof(int));
  }

  // Set first trajectory
  TParticleTrajectoryPoints const& T = OSR.GetTrajectory();
  for (size_t i = 0; i < *h_nt; ++i) {
    h_x[i]  = T.GetX(i).GetX();
    h_y[i]  = T.GetX(i).GetY();
    h_z[i]  = T.GetX(i).GetZ();
    h_bx[i] = T.GetB(i).GetX();
    h_by[i] = T.GetB(i).GetY();
    h_bz[i] = T.GetB(i).GetZ();
  }

  // Set the surface points
  // GPU events
  hipEvent_t *event_spectrumcopy = new hipEvent_t[NGPUsToUse];
  for (int ig = 0; ig < NGPUsToUse; ++ig) {
    int const d = GPUsToUse[ig];
    hipSetDevice(d);
    hipEventCreate(&(event_spectrumcopy[ig]));
  }

  // Enable peer (direct gpu-gpu) writes
  for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
    // Device number
    int const d  = GPUsToUse[ig];
    int const d1 = GPUsToUse[ig+1];
    int access;
    hipDeviceCanAccessPeer(&access, d, d1);
    if (access == 1) {
      hipSetDevice(d);
      hipDeviceEnablePeerAccess(d1, 0);
    }
  }

  // Loop over number of particles
  for (int ip = 0; ip < NParticlesReally; ++ip) {

    // Copy trajectory to first GPU, then internal async transfers (where possible)
    hipSetDevice(d0);
    hipMemcpyAsync(d_nt[0], h_nt,         sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpyAsync(d_x[0],  h_x,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_y[0],  h_y,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_z[0],  h_z,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bx[0], h_bx, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_by[0], h_by, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bz[0], h_bz, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
      // Device number
      int const d  = GPUsToUse[ig];
      int const d1 = GPUsToUse[ig+1];
      hipSetDevice(d);
      hipMemcpyPeerAsync(d_nt[ig+1], d1, d_nt[ig], d,         sizeof(int));
      hipMemcpyPeerAsync(d_x[ig+1],  d1, d_x[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_y[ig+1],  d1, d_y[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_z[ig+1],  d1, d_z[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bx[ig+1], d1, d_bx[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_by[ig+1], d1, d_by[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bz[ig+1], d1, d_bz[ig], d, *h_nt * sizeof(double));

    }

    // Wait for previous copy, start next one
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d = GPUsToUse[ig];
      hipSetDevice(d);
      hipEventSynchronize(event_spectrumcopy[ig]);
      OSCARSSR_Cuda_SpectrumGPUMulti<<<NBlocksThisGPU[ig], NThreadsPerBlock>>>(d_x[ig], d_y[ig], d_z[ig], d_bx[ig], d_by[ig], d_bz[ig], d_obs[ig], d_dt[ig], d_nt[ig], d_ns[ig], d_c0[ig], d_c2[ig], d_ev2omega[ig], d_c[ig], d_se[ig], d_spectrum[ig], d_pol[ig], d_pol_state[ig], d_ifirst[ig]);
    }


    // Add result to spectrum container (from **previous**)
    if (ip > 0) {
      int NBlocksUsed = 0;
      for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
        for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
          if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
            break;
          }
          int iss = ith + NThreadsPerBlock * NBlocksUsed;
          Spectrum.AddToFlux(iss, h_spectrum[ig][ith]);
        }
        NBlocksUsed += NBlocksThisGPU[ig];
      }
    }

    // Add copy back to streams
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d  = GPUsToUse[ig];
      hipSetDevice(d);
      hipMemcpyAsync(h_spectrum[ig],  d_spectrum[ig],  NSpectrum * sizeof(double), hipMemcpyDeviceToHost);
      hipEventRecord(event_spectrumcopy[ig]);
    }

    // If it's not the last one, calculate a new trajectory
    if (ip < NParticlesReally - 1) {
      OSR.SetNewParticle();
      OSR.CalculateTrajectory();
      TParticleTrajectoryPoints const& T = OSR.GetTrajectory();
      *h_nt = T.GetNPoints();

      for (size_t it = 0; it < *h_nt; ++it) {
        h_x[it]  = T.GetX(it).GetX();
        h_y[it]  = T.GetX(it).GetY();
        h_z[it]  = T.GetX(it).GetZ();
        h_bx[it] = T.GetB(it).GetX();
        h_by[it] = T.GetB(it).GetY();
        h_bz[it] = T.GetB(it).GetZ();
      }
    }
  }

  // Wait for last copy
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    hipEventSynchronize(event_spectrumcopy[ig]);
  }

  // Add result to spectrum container (from **previous**)
  NBlocksUsed = 0;
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
      if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
        break;
      }
      int iss = ith + NThreadsPerBlock * NBlocksUsed;
      Spectrum.AddToFlux(iss, h_spectrum[ig][ith]);
    }
    NBlocksUsed += NBlocksThisGPU[ig];
  }

  // Weighting for multi-particle
  double const Weight = 1.0 / (double) NParticlesReally;
  Spectrum.Scale(Weight);

  // Free host memory
  hipHostFree(h_nt_max);
  hipHostFree(h_nt);
  hipHostFree(h_ns);
  hipHostFree(h_dt);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_bx);
  hipHostFree(h_by);
  hipHostFree(h_bz);
  hipHostFree(h_obs);
  hipHostFree(h_se);
  hipHostFree(h_c0);
  hipHostFree(h_c2);
  hipHostFree(h_c);
  hipHostFree(h_ev2omega);
  hipHostFree(h_ifirst);
  hipHostFree(h_pol);
  hipHostFree(h_pol_state);
  // Free host and GPU memory
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {

    hipHostFree(h_spectrum[i]);

    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipFree(d_nt[i]);
    hipFree(d_ns[i]);
    hipFree(d_dt[i]);
    hipFree(d_x[i]);
    hipFree(d_y[i]);
    hipFree(d_z[i]);
    hipFree(d_bx[i]);
    hipFree(d_by[i]);
    hipFree(d_bz[i]);
    hipFree(d_obs[i]);
    hipFree(d_se[i]);
    hipFree(d_c0[i]);
    hipFree(d_c2[i]);
    hipFree(d_c[i]);
    hipFree(d_ev2omega[i]);
    hipFree(d_ifirst[i]);
    hipFree(d_pol[i]);
    hipFree(d_pol_state[i]);
    hipFree(d_spectrum[i]);
  }
  hipFree(h_spectrum);

  hipFree(d_nt);
  hipFree(d_ns);
  hipFree(d_dt);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);
  hipFree(d_obs);
  hipFree(d_se);
  hipFree(d_c0);
  hipFree(d_c2);
  hipFree(d_c);
  hipFree(d_ev2omega);
  hipFree(h_ifirst);
  hipFree(h_pol);
  hipFree(h_pol_state);
  hipFree(d_spectrum);

  // Delete host gpu pointer arrays
  delete [] event_spectrumcopy;


  return;
}








__global__ void OSCARSSR_Cuda_PowerDensityGPUMultiWithA (double  *x, double  *y, double  *z,  // position
                                                         double *bx, double *by, double *bz,  // beta
                                                         double *ax, double *ay, double *az,  // a / c
                                                         double *sx, double *sy, double *sz,  // surface coordinates
                                                         double *nx, double *ny, double *nz,  // surface normal vectors
                                                         double *dt,                          // DeltaT
                                                         int *nt,                             // number of trajectory points
                                                         int *ns,                             // number of surface elements
                                                         int *shn,                            // use normal
                                                         int *ifirst,                         // first index for this thread
                                                         double *power_density)
{
  // Check that this is within the number of spectrum points requested
  int const ith = threadIdx.x + blockIdx.x * blockDim.x;
  int const is = ith + *ifirst;
  if (is >= *ns) {
    return;
  }


  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // Normal vector from input
  double const NormalX = nx[is];
  double const NormalY = ny[is];
  double const NormalZ = nz[is];

  double Sum = 0;

  for (int i = 0; i < *nt; ++i) {

    // Normal vector in direction of observation point
    double const R1 = sqrt( pow(ox - x[i], 2) + pow(oy - y[i], 2) + pow(oz - z[i], 2) );
    double const N1X = (ox - x[i]) / R1;
    double const N1Y = (oy - y[i]) / R1;
    double const N1Z = (oz - z[i]) / R1;

    // Surface normal dot with vector normal
    double const N1DotNormal = *shn == 1 ? N1X * NormalX + N1Y * NormalY + N1Z * NormalZ : 1;

    // Orthogonal vector 2 & 3
    double N2X;
    double N2Y;
    double N2Z;

    double const xx = N1X < 0.0 ? -N1X : N1X;
    double const yy = N1Y < 0.0 ? -N1Y : N1Y;
    double const zz = N1Z < 0.0 ? -N1Z : N1Z;
    if (xx < yy) {
      if (xx < zz) {
        N2X = 0;
        N2Y = N1Z;
        N2Z = -N1Y;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    } else {
      if (yy < zz) {
        N2X = -N1Z;
        N2Y = 0;
        N2Z = N1X;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    }
    double const R2 = sqrt(N2X * N2X + N2Y * N2Y + N2Z * N2Z);
    N2X /= R2;
    N2Y /= R2;
    N2Z /= R2;

    // Ortohgonal vector N3
    double const N3X = N1Y * N2Z - N1Z * N2Y;
    double const N3Y = N1Z * N2X - N1X * N2Z;
    double const N3Z = N1X * N2Y - N1Y * N2X;

    double const x1 = N1X - bx[i];
    double const y1 = N1Y - by[i];
    double const z1 = N1Z - bz[i];

    double const x2 = y1 * az[i] - z1 * ay[i];
    double const y2 = z1 * ax[i] - x1 * az[i];
    double const z2 = x1 * ay[i] - y1 * ax[i];

    // Numerator = N1.Cross( ( (N1 - B).Cross((AoverC)) ) );
    double const x3 = N1Y * z2 - N1Z * y2;
    double const y3 = N1Z * x2 - N1X * z2;
    double const z3 = N1X * y2 - N1Y * x2;

    double const BdotN1 = bx[i] * N1X + by[i] * N1Y + bz[i] * N1Z;
    double const Denominator = pow(1. - BdotN1, 5);

    Sum += pow( x3 * N2X + y3 * N2Y + z3 * N2Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
    Sum += pow( x3 * N3X + y3 * N3Y + z3 * N3Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
  }

  power_density[ith] = Sum * (*dt);

  return;
}








__global__ void OSCARSSR_Cuda_PowerDensityGPU (double  *x, double  *y, double  *z,
                                               double *bx, double *by, double *bz,
                                               double *aocx, double *aocy, double *aocz,
                                               double *sx, double *sy, double *sz,
                                               double *snx, double *sny, double *snz,
                                               double *dt,
                                               int *nt,
                                               int *ns,
                                               int *shn,
                                               double *power_density)
{
  // Get surface id from block and thread number
  int is = threadIdx.x + blockIdx.x * blockDim.x;

  if (is >= *ns) {
    return;
  }




  // If you could copy int ultra-fast memory, something like this:
  //__shared__ double temp[6144];
  //if (threadIdx.x == 0) {
  //  for (int i = 0; i < *nt; ++i) {
  //    if (i <= 6144) {
  //      break;
  //    }
  //    temp[i] = x[i];
  //  }
  //}
  // __syncthreads();



  // Observation point
  double const OX = sx[is];
  double const OY = sy[is];
  double const OZ = sz[is];

  // Normal vector from input
  double const NormalX = snx[is];
  double const NormalY = sny[is];
  double const NormalZ = snz[is];

  double Sum = 0;

  for (int i = 0; i < *nt; ++i) {

    // Normal vector in direction of observation point
    double const R1 = sqrt( pow(OX - x[i], 2) + pow(OY - y[i], 2) + pow(OZ - z[i], 2) );
    double const N1X = (OX - x[i]) / R1;
    double const N1Y = (OY - y[i]) / R1;
    double const N1Z = (OZ - z[i]) / R1;

    // Surface normal dot with vector normal
    double const N1DotNormal = *shn == 1 ? N1X * NormalX + N1Y * NormalY + N1Z * NormalZ : 1;

    // Orthogonal vector 2 & 3
    double N2X;
    double N2Y;
    double N2Z;

    double const xx = N1X < 0.0 ? -N1X : N1X;
    double const yy = N1Y < 0.0 ? -N1Y : N1Y;
    double const zz = N1Z < 0.0 ? -N1Z : N1Z;
    if (xx < yy) {
      if (xx < zz) {
        N2X = 0;
        N2Y = N1Z;
        N2Z = -N1Y;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    } else {
      if (yy < zz) {
        N2X = -N1Z;
        N2Y = 0;
        N2Z = N1X;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    }
    double const R2 = sqrt(N2X * N2X + N2Y * N2Y + N2Z * N2Z);
    N2X /= R2;
    N2Y /= R2;
    N2Z /= R2;

    // Ortohgonal vector N3
    double const N3X = N1Y * N2Z - N1Z * N2Y;
    double const N3Y = N1Z * N2X - N1X * N2Z;
    double const N3Z = N1X * N2Y - N1Y * N2X;





    double const x1 = N1X - bx[i];
    double const y1 = N1Y - by[i];
    double const z1 = N1Z - bz[i];

    double const x2 = y1 * aocz[i] - z1 * aocy[i];
    double const y2 = z1 * aocx[i] - x1 * aocz[i];
    double const z2 = x1 * aocy[i] - y1 * aocx[i];

    // Numerator = N1.Cross( ( (N1 - B).Cross((AoverC)) ) );
    double const x3 = N1Y * z2 - N1Z * y2;
    double const y3 = N1Z * x2 - N1X * z2;
    double const z3 = N1X * y2 - N1Y * x2;

    double const BdotN1 = bx[i] * N1X + by[i] * N1Y + bz[i] * N1Z;
    double const Denominator = pow(1. - BdotN1, 5);

    Sum += pow( x3 * N2X + y3 * N2Y + z3 * N2Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
    Sum += pow( x3 * N3X + y3 * N3Y + z3 * N3Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
  }

  power_density[is] = Sum * (*dt);

  return;
}



extern "C" void OSCARSSR_Cuda_CalculatePowerDensityGPUWithA (OSCARSSR& OSR,
                                                             TSurfacePoints const& Surface,
                                                             T3DScalarContainer& PowerDensityContainer,
                                                             int const NParticles,
                                                             std::vector<int> const& GPUVector)
{
  // Calculate the pd for NParticles using the GPUs given in GPUVector.  Each particle's
  // trajectory will be sent to all GPUs for processing, meanwhile a new trajectory will
  // be calculated

  // Number of available GPUs
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Make sure that a gpu listed is within the range and not a duplicate
  std::vector<int> GPUsToUse;
  for (std::vector<int>::const_iterator it = GPUVector.begin(); it != GPUVector.end(); ++it) {
    if ( !(std::find(GPUsToUse.begin(), GPUsToUse.end(), *it) != GPUsToUse.end() && (*it < ngpu)) ) {
      GPUsToUse.push_back(*it);
    }
  }

  // Make sure we have at least one
  if (GPUsToUse.size() == 0) {
    throw std::invalid_argument("GPUs selected do not match hardware");
  }
  int const NGPUsToUse = (int) GPUsToUse.size();

  // Do we calculate for the current particle?
  bool const ThisParticleOnly = NParticles == 0 ? true : false;
  int  const NParticlesReally = ThisParticleOnly ? 1 : NParticles;
  // Type check, new particle if no type


  int *h_nt, *h_nt_max, *h_ns;
  double *h_dt;
  hipHostAlloc((void**) &h_nt_max, sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nt,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ns,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_dt,     sizeof(double), hipHostMallocWriteCombined | hipHostMallocMapped);

  // First one, set particle and trajectory
  if (!ThisParticleOnly) {
    OSR.SetNewParticle();
  }
  if (OSR.GetTrajectory().GetNPoints() == 0) {
    OSR.CalculateTrajectory();
  }

  // Needed number of points in the track and time step
  *h_nt_max = (int) OSR.GetNPointsTrajectory();
  *h_nt     = (int) OSR.GetTrajectory().GetNPoints();
  *h_ns     = (int) Surface.GetNPoints();
  *h_dt     = (double) OSR.GetTrajectory().GetDeltaT();


  int const NThreads = *h_ns;
  int const NThreadsPerBlock = NTHREADS_PER_BLOCK;
  int const NThreadsRemainder = NThreads % NThreadsPerBlock;
  int const NBlocksTotal = (NThreads - 1) / NThreadsPerBlock + 1;
  int const NBlocksPerGPU = NBlocksTotal / NGPUsToUse;
  int const NRemainderBlocks = NBlocksTotal % NGPUsToUse;
  // UPDATE: To be modified
  int const NPowerDensity = NThreadsPerBlock * (NBlocksPerGPU + (NRemainderBlocks > 0 ? 1 : 0));

  std::vector<int> NBlocksThisGPU(NGPUsToUse, NBlocksPerGPU);
  for (int i = 0; i < NRemainderBlocks; ++i) {
    ++NBlocksThisGPU[i];
  }

  // Memory allocation for Host
  double  *h_x,  *h_y,  *h_z,  *h_bx,  *h_by,  *h_bz,  *h_ax, *h_ay, *h_az, *h_sx,  *h_sy,  *h_sz, *h_nx, *h_ny, *h_nz;
  int     *h_ifirst, *h_shn;
  double **h_pd;
  hipHostAlloc((void**) &h_x,       *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_y,       *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_z,       *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bx,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_by,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bz,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ax,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ay,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_az,      *h_nt_max * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sx,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sy,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sz,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nx,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ny,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nz,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ifirst, NGPUsToUse * sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_shn,                 sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_pd,     NGPUsToUse * sizeof(double*), hipHostMallocWriteCombined | hipHostMallocMapped);
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    hipHostAlloc((void**) &(h_pd[i]), NPowerDensity* sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  }

  // First surface point for each gpu
  int NBlocksUsed = 0;
  for (int i = 0; i < NGPUsToUse; ++i) {
    h_ifirst[i] = NBlocksUsed * NThreadsPerBlock;
    NBlocksUsed += NBlocksThisGPU[i];
  }


  // Memor allocations for GPU
  int    **d_nt;
  int    **d_ns;
  double **d_dt;
  double **d_x;
  double **d_y;
  double **d_z;
  double **d_bx;
  double **d_by;
  double **d_bz;
  double **d_ax;
  double **d_ay;
  double **d_az;
  double **d_sx;
  double **d_sy;
  double **d_sz;
  double **d_nx;
  double **d_ny;
  double **d_nz;
  int    **d_ifirst;
  int    **d_shn;
  double **d_pd;

  hipHostAlloc((void **) &d_nt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ns,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_dt,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_x,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_y,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_z,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_by,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ax,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ay,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_az,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sy,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_nx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ny,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_nz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ifirst, NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_shn,    NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_pd,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipMalloc((void **) &d_nt[i],                 sizeof(int));
    hipMalloc((void **) &d_ns[i],                 sizeof(int));
    hipMalloc((void **) &d_dt[i],                 sizeof(double));
    hipMalloc((void **) &d_x[i],      *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_y[i],      *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_z[i],      *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_bx[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_by[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_bz[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_ax[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_ay[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_az[i],     *h_nt_max * sizeof(double));
    hipMalloc((void **) &d_sx[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sy[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sz[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_nx[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_ny[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_nz[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_ifirst[i],             sizeof(int));
    hipMalloc((void **) &d_shn[i],                sizeof(int));
    hipMalloc((void **) &d_pd[i], NPowerDensity * sizeof(double));

    // Copy device number to device
    hipMemcpyAsync(d_ifirst[i], &(h_ifirst[i]), sizeof(int), hipMemcpyHostToDevice);
  }

  // Compute known host values
  *h_shn   = Surface.HasNormal() ? 1 : 0;
  for (size_t i = 0; i < *h_ns; ++i) {
    h_sx[i] = Surface.GetPoint(i).GetX();
    h_sy[i] = Surface.GetPoint(i).GetY();
    h_sz[i] = Surface.GetPoint(i).GetZ();

    h_nx[i] = Surface.GetPoint(i).GetNormalX();
    h_ny[i] = Surface.GetPoint(i).GetNormalY();
    h_nz[i] = Surface.GetPoint(i).GetNormalZ();
  }

  // Copy constants to first device (async)
  int const d0 = GPUsToUse[0];
  hipSetDevice(d0);
  hipMemcpyAsync(d_ns[0],    h_ns,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_dt[0],    h_dt,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_shn[0],   h_shn,         sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sx[0],    h_sx,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sy[0],    h_sy,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sz[0],    h_sz,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_nx[0],    h_nx,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_ny[0],    h_ny,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_nz[0],    h_nz,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_ns[i+1],     d1, d_ns[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_dt[i+1],     d1, d_dt[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_shn[i+1],    d1, d_shn[i],    d, sizeof(int));
    hipMemcpyPeerAsync( d_sx[i+1],     d1, d_sx[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sy[i+1],     d1, d_sy[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sz[i+1],     d1, d_sz[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_nx[i+1],     d1, d_nx[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_ny[i+1],     d1, d_ny[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_nz[i+1],     d1, d_nz[i],     d, *h_ns * sizeof(double));
  }

  // Set first trajectory
  TParticleTrajectoryPoints const& T = OSR.GetTrajectory();
  int const NPointsThisTrajectory = T.GetNPoints();
  *h_nt = 0;
  for (size_t i = 0; i < NPointsThisTrajectory; ++i) {
    h_x[*h_nt]  = T.GetX(i).GetX();
    h_y[*h_nt]  = T.GetX(i).GetY();
    h_z[*h_nt]  = T.GetX(i).GetZ();
    h_bx[*h_nt] = T.GetB(i).GetX();
    h_by[*h_nt] = T.GetB(i).GetY();
    h_bz[*h_nt] = T.GetB(i).GetZ();
    h_ax[*h_nt] = T.GetAoverC(i).GetX();
    h_ay[*h_nt] = T.GetAoverC(i).GetY();
    h_az[*h_nt] = T.GetAoverC(i).GetZ();
    ++(*h_nt);
  }
  hipSetDevice(d0);
  hipMemcpyAsync(d_nt[0],    h_nt,          sizeof(int),    hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_nt[i+1],     d1, d_nt[i],     d, sizeof(int));
  }


  // Set the surface points
  // GPU events
  hipEvent_t *event_pdcopy = new hipEvent_t[NGPUsToUse];
  for (int ig = 0; ig < NGPUsToUse; ++ig) {
    int const d = GPUsToUse[ig];
    hipSetDevice(d);
    hipEventCreate(&(event_pdcopy[ig]));
  }

  // Enable peer (direct gpu-gpu) writes
  for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
    // Device number
    int const d  = GPUsToUse[ig];
    int const d1 = GPUsToUse[ig+1];
    int access;
    hipDeviceCanAccessPeer(&access, d, d1);
    if (access == 1) {
      hipSetDevice(d);
      hipDeviceEnablePeerAccess(d1, 0);
    }
  }

  // Loop over number of particles
  for (int ip = 0; ip < NParticlesReally; ++ip) {

    // Copy trajectory to first GPU, then internal async transfers (where possible)
    hipSetDevice(d0);
    hipMemcpyAsync(d_nt[0], h_nt,         sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpyAsync(d_x[0],  h_x,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_y[0],  h_y,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_z[0],  h_z,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bx[0], h_bx, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_by[0], h_by, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bz[0], h_bz, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ax[0], h_ax, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ay[0], h_ay, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_az[0], h_az, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
      // Device number
      int const d  = GPUsToUse[ig];
      int const d1 = GPUsToUse[ig+1];
      hipSetDevice(d);
      hipMemcpyPeerAsync(d_nt[ig+1], d1, d_nt[ig], d,         sizeof(int));
      hipMemcpyPeerAsync(d_x[ig+1],  d1, d_x[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_y[ig+1],  d1, d_y[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_z[ig+1],  d1, d_z[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bx[ig+1], d1, d_bx[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_by[ig+1], d1, d_by[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bz[ig+1], d1, d_bz[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ax[ig+1], d1, d_ax[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ay[ig+1], d1, d_ay[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_az[ig+1], d1, d_az[ig], d, *h_nt * sizeof(double));

    }

    // Wait for previous copy, start next one
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d = GPUsToUse[ig];
      hipSetDevice(d);
      hipEventSynchronize(event_pdcopy[ig]);
      OSCARSSR_Cuda_PowerDensityGPUMultiWithA<<<NBlocksThisGPU[ig], NThreadsPerBlock>>>( d_x[ig],  d_y[ig],  d_z[ig],
                                                                           d_bx[ig], d_by[ig], d_bz[ig],
                                                                           d_ax[ig], d_ay[ig], d_az[ig],
                                                                           d_sx[ig], d_sy[ig], d_sz[ig],
                                                                           d_nx[ig], d_ny[ig], d_nz[ig],
                                                                           d_dt[ig],
                                                                           d_nt[ig],
                                                                           d_ns[ig],
                                                                           d_shn[ig],
                                                                           d_ifirst[ig],
                                                                           d_pd[ig]);
    }


    // Add result to pd container (from **previous**)
    if (ip > 0) {
      int NBlocksUsed = 0;
      for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
        for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
          if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
            break;
          }
          int iss = ith + NThreadsPerBlock * NBlocksUsed;
          PowerDensityContainer.AddToPoint(iss, h_pd[ig][ith] * fabs(OSR.GetCurrentParticle().GetQ() * OSR.GetCurrentParticle().GetCurrent()) / (16 * TOSCARSSR::Pi2() * TOSCARSSR::Epsilon0() * TOSCARSSR::C()) / 1e6);
        }
        NBlocksUsed += NBlocksThisGPU[ig];
      }
    }

    // Add copy back to streams
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d  = GPUsToUse[ig];
      hipSetDevice(d);
      hipMemcpyAsync(h_pd[ig],  d_pd[ig],  NPowerDensity * sizeof(double), hipMemcpyDeviceToHost);
      hipEventRecord(event_pdcopy[ig]);
    }

    // If it's not the last one, calculate a new trajectory
    if (ip < NParticlesReally - 1) {
      OSR.SetNewParticle();
      OSR.CalculateTrajectory();
      TParticleTrajectoryPoints const& T = OSR.GetTrajectory();
      int const NPointsThisTrajectory = T.GetNPoints();

      *h_nt = 0;
      for (size_t it = 0; it < NPointsThisTrajectory; ++it) {
        h_x[*h_nt]  = T.GetX(it).GetX();
        h_y[*h_nt]  = T.GetX(it).GetY();
        h_z[*h_nt]  = T.GetX(it).GetZ();
        h_bx[*h_nt] = T.GetB(it).GetX();
        h_by[*h_nt] = T.GetB(it).GetY();
        h_bz[*h_nt] = T.GetB(it).GetZ();
        h_ax[*h_nt] = T.GetAoverC(it).GetX();
        h_ay[*h_nt] = T.GetAoverC(it).GetY();
        h_az[*h_nt] = T.GetAoverC(it).GetZ();
        ++(*h_nt);
      }
    }
  }

  // Wait for last copy
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    hipEventSynchronize(event_pdcopy[ig]);
  }

  // Add result to pd container (from **previous**)
  NBlocksUsed = 0;
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
      if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
        break;
      }
      int iss = ith + NThreadsPerBlock * NBlocksUsed;
      PowerDensityContainer.AddToPoint(iss, h_pd[ig][ith] * fabs(OSR.GetCurrentParticle().GetQ() * OSR.GetCurrentParticle().GetCurrent()) / (16 * TOSCARSSR::Pi2() * TOSCARSSR::Epsilon0() * TOSCARSSR::C()) / 1e6);
    }
    NBlocksUsed += NBlocksThisGPU[ig];
  }

  // Weighting for multi-particle
  double const Weight = 1.0 / (double) NParticlesReally;
  PowerDensityContainer.WeightAll(Weight);

  // Free host memory
  hipHostFree(h_nt_max);
  hipHostFree(h_nt);
  hipHostFree(h_ns);
  hipHostFree(h_dt);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_bx);
  hipHostFree(h_by);
  hipHostFree(h_bz);
  hipHostFree(h_ax);
  hipHostFree(h_ay);
  hipHostFree(h_az);
  hipHostFree(h_sx);
  hipHostFree(h_sy);
  hipHostFree(h_sz);
  hipHostFree(h_nx);
  hipHostFree(h_ny);
  hipHostFree(h_nz);
  hipHostFree(h_shn);
  hipHostFree(h_ifirst);
  // Free host and GPU memory
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {

    hipHostFree(h_pd[i]);

    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipFree(d_nt[i]);
    hipFree(d_ns[i]);
    hipFree(d_dt[i]);
    hipFree(d_x[i]);
    hipFree(d_y[i]);
    hipFree(d_z[i]);
    hipFree(d_bx[i]);
    hipFree(d_by[i]);
    hipFree(d_bz[i]);
    hipFree(d_ax[i]);
    hipFree(d_ay[i]);
    hipFree(d_az[i]);
    hipFree(d_sx[i]);
    hipFree(d_sy[i]);
    hipFree(d_sz[i]);
    hipFree(d_nx[i]);
    hipFree(d_ny[i]);
    hipFree(d_nz[i]);
    hipFree(d_shn[i]);
    hipFree(d_ifirst[i]);
    hipFree(d_pd[i]);
  }
  hipFree(h_pd);

  hipFree(d_nt);
  hipFree(d_ns);
  hipFree(d_dt);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);
  hipFree(d_ax);
  hipFree(d_ay);
  hipFree(d_az);
  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);
  hipFree(d_nx);
  hipFree(d_ny);
  hipFree(d_nz);
  hipFree(d_shn);
  hipFree(h_ifirst);
  hipFree(d_pd);

  // Delete host gpu pointer arrays
  delete [] event_pdcopy;


  return;
}
extern "C" void OSCARSSR_Cuda_CalculatePowerDensityGPU (TParticleA& Particle,
                                                        TSurfacePoints const& Surface,
                                                        T3DScalarContainer& PowerDensityContainer,
                                                        bool const Directional,
                                                        double const Weight)
{

  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Grab the Trajectory
  TParticleTrajectoryPoints& T = Particle.GetTrajectory();

  // Number of points in Trajectory
  int const NTPoints = (int) T.GetNPoints();

  // Timestep from trajectory
  double const DeltaT = T.GetDeltaT();

  double *x     = new double[NTPoints];
  double *y     = new double[NTPoints];
  double *z     = new double[NTPoints];
  double *bx    = new double[NTPoints];
  double *by    = new double[NTPoints];
  double *bz    = new double[NTPoints];
  double *aocx  = new double[NTPoints];
  double *aocy  = new double[NTPoints];
  double *aocz  = new double[NTPoints];

  int const NSPoints = (int) Surface.GetNPoints();

  double *sx     = new double[NSPoints];
  double *sy     = new double[NSPoints];
  double *sz     = new double[NSPoints];

  int     shn    = Surface.HasNormal() ? 1 : 0;
  double *snx    = new double[NSPoints];
  double *sny    = new double[NSPoints];
  double *snz    = new double[NSPoints];

  double *power_density = new double[NSPoints];


  for (size_t i = 0; i < NTPoints; ++i) {
    x[i] = T.GetX(i).GetX();
    y[i] = T.GetX(i).GetY();
    z[i] = T.GetX(i).GetZ();

    bx[i] = T.GetB(i).GetX();
    by[i] = T.GetB(i).GetY();
    bz[i] = T.GetB(i).GetZ();

    aocx[i] = T.GetAoverC(i).GetX();
    aocy[i] = T.GetAoverC(i).GetY();
    aocz[i] = T.GetAoverC(i).GetZ();
  }



  for (size_t i = 0; i < NSPoints; ++i) {
    sx[i] = Surface.GetPoint(i).GetX();
    sy[i] = Surface.GetPoint(i).GetY();
    sz[i] = Surface.GetPoint(i).GetZ();

    snx[i] = Surface.GetPoint(i).GetNormalX();
    sny[i] = Surface.GetPoint(i).GetNormalY();
    snz[i] = Surface.GetPoint(i).GetNormalZ();
  }



  double *d_x, *d_y, *d_z;
  double *d_bx, *d_by, *d_bz;
  double *d_aocx, *d_aocy, *d_aocz;
  double *d_sx, *d_sy, *d_sz;
  double *d_snx, *d_sny, *d_snz;
  double *d_power_density;
  double *d_dt;
  int    *d_nt, *d_ns, *d_shn;

  int const size_x = NTPoints * sizeof(double);
  int const size_s = NSPoints * sizeof(double);

  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_y, size_x);
  hipMalloc((void **) &d_z, size_x);

  hipMalloc((void **) &d_bx, size_x);
  hipMalloc((void **) &d_by, size_x);
  hipMalloc((void **) &d_bz, size_x);

  hipMalloc((void **) &d_aocx, size_x);
  hipMalloc((void **) &d_aocy, size_x);
  hipMalloc((void **) &d_aocz, size_x);

  hipMalloc((void **) &d_sx, size_s);
  hipMalloc((void **) &d_sy, size_s);
  hipMalloc((void **) &d_sz, size_s);

  hipMalloc((void **) &d_snx, size_s);
  hipMalloc((void **) &d_sny, size_s);
  hipMalloc((void **) &d_snz, size_s);

  hipMalloc((void **) &d_power_density, size_s);

  hipMalloc((void **) &d_dt, sizeof(double));

  hipMalloc((void **) &d_nt, sizeof(int));
  hipMalloc((void **) &d_ns, sizeof(int));
  hipMalloc((void **) &d_shn, sizeof(int));


  hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_bx, bx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_by, by, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_bz, bz, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_aocx, aocx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_aocy, aocy, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_aocz, aocz, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_sx, sx, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sy, sy, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sz, sz, size_s, hipMemcpyHostToDevice);

  hipMemcpy(d_snx, snx, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sny, sny, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_snz, snz, size_s, hipMemcpyHostToDevice);

  hipMemcpy(d_dt, &DeltaT, sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_nt, &NTPoints, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_ns, &NSPoints, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_shn, &shn, sizeof(int), hipMemcpyHostToDevice);


  // Send computation to gpu
  int const NBlocks = NSPoints / NTHREADS_PER_BLOCK + 1;
  OSCARSSR_Cuda_PowerDensityGPU<<<NBlocks, NTHREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_bx, d_by, d_bz, d_aocx, d_aocy, d_aocz, d_sx, d_sy, d_sz, d_snx, d_sny, d_snz, d_dt, d_nt, d_ns, d_shn, d_power_density);

  // Copy result back from GPU
  hipMemcpy(power_density, d_power_density, size_s, hipMemcpyDeviceToHost);



  // Add result to power density container
  for (size_t i = 0; i < NSPoints; ++i) {
    PowerDensityContainer.AddToPoint(i, power_density[i] * fabs(Particle.GetQ() * Particle.GetCurrent()) / (16 * TOSCARSSR::Pi2() * TOSCARSSR::Epsilon0() * TOSCARSSR::C()) / 1e6 * Weight);
  }


  // Free all gpu memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);

  hipFree(d_aocx);
  hipFree(d_aocy);
  hipFree(d_aocz);

  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);

  hipFree(d_snx);
  hipFree(d_sny);
  hipFree(d_snz);

  hipFree(d_dt);
  hipFree(d_nt);
  hipFree(d_ns);

  hipFree(d_power_density);





  // Free all heap memory
  delete [] x;
  delete [] y;
  delete [] z;

  delete [] bx;
  delete [] by;
  delete [] bz;

  delete [] aocx;
  delete [] aocy;
  delete [] aocz;

  delete [] sx;
  delete [] sy;
  delete [] sz;

  delete [] snx;
  delete [] sny;
  delete [] snz;

  delete [] power_density;

  return;
}





