#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////
//
// Dean Andrew Hidas <dhidas@bnl.gov>
//
// Created on: Tue May 24 11:33:19 EDT 2016
//
////////////////////////////////////////////////////////////////////

#include <hip/hip_complex.h>

#include "OSCARSSR_Cuda.h"

#include "OSCARSSR.h"

#include <cmath>
#include <fstream>
#include <sstream>

#include "TVector3DC.h"
#include "TSpectrumContainer.h"


#define NTHREADS_PER_BLOCK 512






extern "C" int OSCARSSR_Cuda_GetDeviceCount ()
{
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);

  return ngpu;
}





extern "C" std::string OSCARSSR_Cuda_GetDeviceProperties (int const i)
{
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);

  char buf[300];

  if (i >= ngpu) {
    sprintf(buf, "ERROR: GPU %i Not available", i);
    return std::string(buf);
  }

  std::string ret = "";

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, i);

  sprintf(buf, "  Device name: %s\n", prop.name);
  ret += std::string(buf);
  sprintf(buf, "  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
  ret += std::string(buf);
  sprintf(buf, "  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
  ret += std::string(buf);
  sprintf(buf, "  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  ret += std::string(buf);

  return ret;
}





__device__ static __inline__ void Orthogonal(double *a, double *b)
{
  // Return a vector which is orthogonal vector a
  double xx = a[0] < 0.0 ? -a[0] : a[0];
  double yy = a[1] < 0.0 ? -a[1] : a[1];
  double zz = a[2] < 0.0 ? -a[2] : a[2];
  if (xx < yy) {
    if (xx < zz) {
      b[0] = 0;
      b[1] = a[2];
      b[2] = -a[1];
    } else {
      b[0] = a[1];
      b[1] = -a[0];
      b[2] = 0;
    }
  } else {
    if (yy < zz) {
      b[0] = -a[2];
      b[1] = 0;
      b[2] = a[0];
    } else {
      b[0] = a[1];
      b[1] = -a[0];
      b[2] = 0;
    }
  }
  return;
}



__host__ __device__ static __inline__ hipDoubleComplex cuCexp(hipDoubleComplex x)
{
  double factor = exp(x.x);
  return make_hipDoubleComplex(factor * cos(x.y), factor * sin(x.y));
}










__global__ void OSCARSSR_Cuda_FluxGPU4 (double *x, double *y, double *z, double *bx, double *by, double *bz, double *sx, double *sy, double *sz, double *dt, int *nt, int *ns, double *C0, double *C2, double *C, double *Omega, double *flux)
{
  // Check that this is within the number of spectrum points requested
  int is = threadIdx.x + blockIdx.x * blockDim.x;
  // Max number for shared memory
  int const NSHAREDMAX = 1000;

  // Number for each thread to copy from global to shared memory
  int const NToCopyPerThread = (int) NSHAREDMAX / NTHREADS_PER_BLOCK;

  // Actual number of elements in shared memory to use
  int const NSHARED = NToCopyPerThread * NTHREADS_PER_BLOCK;

  // Define the shared memory
  __shared__ double sh_x[NSHARED];
  __shared__ double sh_y[NSHARED];
  __shared__ double sh_z[NSHARED];
  __shared__ double sh_bx[NSHARED];
  __shared__ double sh_by[NSHARED];
  __shared__ double sh_bz[NSHARED];


  // observer
  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);



  // Total number of copies
  int const NTotalCopies = *nt / NSHARED + 1;

  // Local offset for this thread in shared memory
  int const ThreadOffset = NToCopyPerThread * threadIdx.x;

  // icp is Copy Number
  for (int icp = 0; icp < NTotalCopies; ++icp) {

    // Offset instarting point in global array
    int const GlobalOffset = icp * NSHARED;


    __syncthreads();

    // icpth is the copy number in a thread
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_x[ThisThreadSharedIndex]  = x[GlobalIndex];
    }
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_y[ThisThreadSharedIndex]  = y[GlobalIndex];
    }
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_z[ThisThreadSharedIndex]  = z[GlobalIndex];
    }
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_bx[ThisThreadSharedIndex] = bx[GlobalIndex];
    }
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_by[ThisThreadSharedIndex] = by[GlobalIndex];
    }
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_bz[ThisThreadSharedIndex] = bz[GlobalIndex];
    }


    __syncthreads();

    if (is >= *ns) {
      continue;
    }

    for (int ish = 0; ish < NSHARED; ++ish) {
      int const i = GlobalOffset + ish;

      if (i >= *nt) {
        break;
      }



      // Distance to observer
      double const D = sqrt( pow( (ox) - sh_x[ish], 2) + pow( (oy) - sh_y[ish], 2) + pow((oz) - sh_z[ish], 2) );

      // Normal in direction of observer
      double const NX = ((ox) - sh_x[ish]) / D;
      double const NY = ((oy) - sh_y[ish]) / D;
      double const NZ = ((oz) - sh_z[ish]) / D;

      // Exponent for fourier transformed field
      hipDoubleComplex Exponent = make_hipDoubleComplex(0, (*Omega) * ((*dt) * i + D / (*C)));

      hipDoubleComplex X1 = make_hipDoubleComplex((sh_bx[ish] - NX) / D, -(*C) * NX / ((*Omega) * D * D));
      hipDoubleComplex Y1 = make_hipDoubleComplex((sh_by[ish] - NY) / D, -(*C) * NY / ((*Omega) * D * D));
      hipDoubleComplex Z1 = make_hipDoubleComplex((sh_bz[ish] - NZ) / D, -(*C) * NZ / ((*Omega) * D * D));

      hipDoubleComplex MyEXP = cuCexp(Exponent);

      hipDoubleComplex X2 = hipCmul(X1, MyEXP);
      hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
      hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


      SumEX = hipCadd(SumEX, X2);
      SumEY = hipCadd(SumEY, Y2);
      SumEZ = hipCadd(SumEZ, Z2);








    }
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEZ);


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;


  flux[is] = (*C2) * (EX + EY + EZ);

  return;
}


























__global__ void OSCARSSR_Cuda_FluxGPU3 (double *x, double *y, double *z, double *bx, double *by, double *bz, double *sx, double *sy, double *sz, double *dt, int *nt, int *ns, double *C0, double *C2, double *C, double *Omega, double *flux)
{
  // Check that this is within the number of spectrum points requested
  int is = threadIdx.x + blockIdx.x * blockDim.x;
  // Max number for shared memory

  int const NSHARED = 1000;

  // Define the shared memory
  __shared__ double sh_x[1000];
  __shared__ double sh_y[1000];
  __shared__ double sh_z[1000];
  __shared__ double sh_bx[1000];
  __shared__ double sh_by[1000];
  __shared__ double sh_bz[1000];


  // observer
  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);



  // Total number of copies
  int const NTotalCopies = *nt / NSHARED + 1;

  // icp is Copy Number
  for (int icp = 0; icp < NTotalCopies; ++icp) {

    // Offset instarting point in global array
    int const GlobalOffset = icp * NSHARED;

    __syncthreads();

    if (threadIdx.x == 0) {
      // icpth is the copy number in a thread
      for (int icpth = 0; icpth < NSHARED; ++icpth) {

        // index of *this* shared memory
        int const ThisThreadSharedIndex = icpth;

        // Global index of *this*
        int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

        // Check if we are within the nt bound
        if (GlobalIndex >= *nt) {
          break;
        }

        // Copy global memory to shared bank
        sh_x[ThisThreadSharedIndex]  = x[GlobalIndex];
        sh_y[ThisThreadSharedIndex]  = y[GlobalIndex];
        sh_z[ThisThreadSharedIndex]  = z[GlobalIndex];
        sh_bx[ThisThreadSharedIndex] = bx[GlobalIndex];
        sh_by[ThisThreadSharedIndex] = by[GlobalIndex];
        sh_bz[ThisThreadSharedIndex] = bz[GlobalIndex];
      }
    }
    __syncthreads();

    if (is >= *ns) {
      continue;
    }

    for (int ish = 0; ish < NSHARED; ++ish) {
      int const i = GlobalOffset + ish;

      if (i >= *nt) {
        break;
      }

      // Distance to observer
      double const D = sqrt( pow( (ox) - sh_x[ish], 2) + pow( (oy) - sh_y[ish], 2) + pow((oz) - sh_z[ish], 2) );

      // Normal in direction of observer
      double const NX = ((ox) - sh_x[ish]) / D;
      double const NY = ((oy) - sh_y[ish]) / D;
      double const NZ = ((oz) - sh_z[ish]) / D;

      // Exponent for fourier transformed field
      hipDoubleComplex Exponent = make_hipDoubleComplex(0, (*Omega) * ((*dt) * i + D / (*C)));

      hipDoubleComplex X1 = make_hipDoubleComplex((sh_bx[ish] - NX) / D, -(*C) * NX / ((*Omega) * D * D));
      hipDoubleComplex Y1 = make_hipDoubleComplex((sh_by[ish] - NY) / D, -(*C) * NY / ((*Omega) * D * D));
      hipDoubleComplex Z1 = make_hipDoubleComplex((sh_bz[ish] - NZ) / D, -(*C) * NZ / ((*Omega) * D * D));

      hipDoubleComplex MyEXP = cuCexp(Exponent);

      hipDoubleComplex X2 = hipCmul(X1, MyEXP);
      hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
      hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


      SumEX = hipCadd(SumEX, X2);
      SumEY = hipCadd(SumEY, Y2);
      SumEZ = hipCadd(SumEZ, Z2);
    }
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEZ);


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;


  flux[is] = (*C2) * (EX + EY + EZ);

  return;
}











__global__ void OSCARSSR_Cuda_FluxGPU2 (double *x, double *y, double *z, double *bx, double *by, double *bz, double *sx, double *sy, double *sz, double *dt, int *nt, int *ns, double *C0, double *C2, double *C, double *Omega, double *flux)
{
  // Check that this is within the number of spectrum points requested
  int is = threadIdx.x + blockIdx.x * blockDim.x;
  // Max number for shared memory
  int const NSHAREDMAX = 1000;

  // Number for each thread to copy from global to shared memory
  int const NToCopyPerThread = (int) NSHAREDMAX / NTHREADS_PER_BLOCK;

  // Actual number of elements in shared memory to use
  int const NSHARED = NToCopyPerThread * NTHREADS_PER_BLOCK;

  // Define the shared memory
  __shared__ double sh_x[NSHARED];
  __shared__ double sh_y[NSHARED];
  __shared__ double sh_z[NSHARED];
  __shared__ double sh_bx[NSHARED];
  __shared__ double sh_by[NSHARED];
  __shared__ double sh_bz[NSHARED];


  // observer
  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);



  // Total number of copies
  int const NTotalCopies = *nt / NSHARED + 1;

  // icp is Copy Number
  for (int icp = 0; icp < NTotalCopies; ++icp) {

    // Offset instarting point in global array
    int const GlobalOffset = icp * NSHARED;

    // Local offset for this thread in shared memory
    int const ThreadOffset = NToCopyPerThread * threadIdx.x;
    //flux[is] = shoffset; return;

    __syncthreads();

    // icpth is the copy number in a thread
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_x[ThisThreadSharedIndex]  = x[GlobalIndex];
      sh_y[ThisThreadSharedIndex]  = y[GlobalIndex];
      sh_z[ThisThreadSharedIndex]  = z[GlobalIndex];
      sh_bx[ThisThreadSharedIndex] = bx[GlobalIndex];
      sh_by[ThisThreadSharedIndex] = by[GlobalIndex];
      sh_bz[ThisThreadSharedIndex] = bz[GlobalIndex];
    }
    __syncthreads();

    if (is >= *ns) {
      continue;
    }

    for (int ish = 0; ish < NSHARED; ++ish) {
      int const i = GlobalOffset + ish;

      if (i >= *nt) {
        break;
      }



      // Distance to observer
      double const D = sqrt( pow( (ox) - sh_x[ish], 2) + pow( (oy) - sh_y[ish], 2) + pow((oz) - sh_z[ish], 2) );

      // Normal in direction of observer
      double const NX = ((ox) - sh_x[ish]) / D;
      double const NY = ((oy) - sh_y[ish]) / D;
      double const NZ = ((oz) - sh_z[ish]) / D;

      // Exponent for fourier transformed field
      hipDoubleComplex Exponent = make_hipDoubleComplex(0, (*Omega) * ((*dt) * i + D / (*C)));

      hipDoubleComplex X1 = make_hipDoubleComplex((sh_bx[ish] - NX) / D, -(*C) * NX / ((*Omega) * D * D));
      hipDoubleComplex Y1 = make_hipDoubleComplex((sh_by[ish] - NY) / D, -(*C) * NY / ((*Omega) * D * D));
      hipDoubleComplex Z1 = make_hipDoubleComplex((sh_bz[ish] - NZ) / D, -(*C) * NZ / ((*Omega) * D * D));

      hipDoubleComplex MyEXP = cuCexp(Exponent);

      hipDoubleComplex X2 = hipCmul(X1, MyEXP);
      hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
      hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


      SumEX = hipCadd(SumEX, X2);
      SumEY = hipCadd(SumEY, Y2);
      SumEZ = hipCadd(SumEZ, Z2);








    }
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEZ);


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;


  flux[is] = (*C2) * (EX + EY + EZ);

  return;
}








__global__ void OSCARSSR_Cuda_FluxGPU (double *x, double *y, double *z, double *bx, double *by, double *bz, double *sx, double *sy, double *sz, double *dt, int *nt, int *ns, double *C0, double *C2, double *C, double *Omega, double *flux)
{
  // Check that this is within the number of spectrum points requested
  int is = threadIdx.x + blockIdx.x * blockDim.x;
  if (is >= *ns) {
    return;
  }

  // Complex i
  hipDoubleComplex I = make_hipDoubleComplex(0, 1);

  hipDoubleComplex ICoverOmega = make_hipDoubleComplex(0, (*C) / (*Omega));

  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);


  // Loop over all points in trajectory
  for (int i = 0; i < *nt; ++i) {

    // Distance to observer
    double const D = sqrt( pow( (ox) - x[i], 2) + pow( (oy) - y[i], 2) + pow((oz) - z[i], 2) );

    // Normal in direction of observer
    double const NX = ((ox) - x[i]) / D;
    double const NY = ((oy) - y[i]) / D;
    double const NZ = ((oz) - z[i]) / D;

    // Exponent for fourier transformed field
    hipDoubleComplex Exponent = make_hipDoubleComplex(0, (*Omega) * ((*dt) * i + D / (*C)));

    hipDoubleComplex X1 = make_hipDoubleComplex((bx[i] - NX) / D, -(*C) * NX / ((*Omega) * D * D));
    hipDoubleComplex Y1 = make_hipDoubleComplex((by[i] - NY) / D, -(*C) * NY / ((*Omega) * D * D));
    hipDoubleComplex Z1 = make_hipDoubleComplex((bz[i] - NZ) / D, -(*C) * NZ / ((*Omega) * D * D));

    hipDoubleComplex MyEXP = cuCexp(Exponent);
    //hipDoubleComplex MyEXP = make_hipDoubleComplex( exp(Exponent.x) * cos(Exponent.y), exp(Exponent.x) * sin(Exponent.y));

    hipDoubleComplex X2 = hipCmul(X1, MyEXP);
    hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
    hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


    SumEX = hipCadd(SumEX, X2);
    SumEY = hipCadd(SumEY, Y2);
    SumEZ = hipCadd(SumEZ, Z2);

    // Sum in fourier transformed field (integral)
    //SumEX += (TVector3DC(B) - (N *     (One + (ICoverOmega / (D)))     )) / D * std::exp(Exponent);
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEZ);


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;

  // Multiply field by Constant C1 and time step
  //SumE *= C1 * DeltaT;

  // Set the flux for this frequency / energy point
  //Spectrum.AddToFlux(i, C2 *  SumE.Dot( SumE.CC() ).real() * Weight);

  flux[is] = (*C2) * (EX + EY + EZ);

  return;
}


















extern "C" void OSCARSSR_Cuda_CalculateFluxGPU (TParticleA& Particle,
                                                TSurfacePoints const& Surface,
                                                double const Energy_eV,
                                                T3DScalarContainer& FluxContainer,
                                                std::string const& Polarization,
                                                double const Angle,
                                                TVector3D const& HorizontalDirection,
                                                TVector3D const& PropogationDirection,
                                                double const Weight)
{
  // Do the setup for and call the GPU calculation of flux.  Your limitation here is only GPU memory.

  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Grab the Trajectory
  TParticleTrajectoryPoints& T = Particle.GetTrajectory();

  // Number of points in Trajectory
  int const NTPoints = (int) T.GetNPoints();

  // Timestep from trajectory
  double const DeltaT = T.GetDeltaT();

  double *x     = new double[NTPoints];
  double *y     = new double[NTPoints];
  double *z     = new double[NTPoints];
  double *bx    = new double[NTPoints];
  double *by    = new double[NTPoints];
  double *bz    = new double[NTPoints];


  int const NSPoints = (int) Surface.GetNPoints();

  // Observer
  double *sx     = new double[NSPoints];
  double *sy     = new double[NSPoints];
  double *sz     = new double[NSPoints];

  // Constants
  double const C = TOSCARSSR::C();
  double const Omega = TOSCARSSR::EvToAngularFrequency(Energy_eV);

  // Flux
  double *flux = new double[NSPoints];


  // Set trajectory
  for (size_t i = 0; i < NTPoints; ++i) {
    x[i] = T.GetX(i).GetX();
    y[i] = T.GetX(i).GetY();
    z[i] = T.GetX(i).GetZ();

    bx[i] = T.GetB(i).GetX();
    by[i] = T.GetB(i).GetY();
    bz[i] = T.GetB(i).GetZ();
  }

  // Set the surface points
  for (size_t i = 0; i < NSPoints; ++i) {
    sx[i] = Surface.GetPoint(i).GetX();
    sy[i] = Surface.GetPoint(i).GetY();
    sz[i] = Surface.GetPoint(i).GetZ();
  }




  double *d_x, *d_y, *d_z;
  double *d_bx, *d_by, *d_bz;
  double *d_sx, *d_sy, *d_sz;
  double *d_flux;
  double *d_dt;
  int    *d_nt, *d_ns;

  int const size_x = NTPoints * sizeof(double);
  int const size_s = NSPoints * sizeof(double);

  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_y, size_x);
  hipMalloc((void **) &d_z, size_x);

  hipMalloc((void **) &d_bx, size_x);
  hipMalloc((void **) &d_by, size_x);
  hipMalloc((void **) &d_bz, size_x);

  hipMalloc((void **) &d_sx, size_s);
  hipMalloc((void **) &d_sy, size_s);
  hipMalloc((void **) &d_sz, size_s);


  hipMalloc((void **) &d_dt, sizeof(double));
  hipMalloc((void **) &d_nt, sizeof(int));
  hipMalloc((void **) &d_ns, sizeof(int));

  hipMalloc((void **) &d_flux, size_s);


  hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_bx, bx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_by, by, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_bz, bz, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_sx, sx, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sy, sy, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sz, sz, size_s, hipMemcpyHostToDevice);



  hipMemcpy(d_dt, &DeltaT, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nt, &NTPoints, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_ns, &NSPoints, sizeof(int), hipMemcpyHostToDevice);



  // Constant C0 for calculation
  double const C0 = Particle.GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());

  // Constant for flux calculation at the end
  double const C2 = TOSCARSSR::FourPi() * Particle.GetCurrent() / (TOSCARSSR::H() * fabs(Particle.GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;

  // Constants to send in to GPU
  double *d_C0, *d_C2, *d_Omega, *d_C;

  // Allocate memory for constants
  hipMalloc((void **) &d_C0,        sizeof(double));
  hipMalloc((void **) &d_C2,        sizeof(double));
  hipMalloc((void **) &d_Omega,     sizeof(double));
  hipMalloc((void **) &d_C,         sizeof(double));

  // Copy constants to GPU
  hipMemcpy(d_C0,        &C0,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_C2,        &C2,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_Omega,     &Omega,     sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_C,         &C,         sizeof(double), hipMemcpyHostToDevice);


  // Send computation to gpu
  int const NBlocks = NSPoints / NTHREADS_PER_BLOCK + 1;
  OSCARSSR_Cuda_FluxGPU4<<<NBlocks, NTHREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_bx, d_by, d_bz, d_sx, d_sy, d_sz, d_dt, d_nt, d_ns, d_C0, d_C2, d_C, d_Omega, d_flux);

  // Copy result back from GPU
  hipMemcpy(flux, d_flux, size_s, hipMemcpyDeviceToHost);



  // Add result to power density container
  for (size_t i = 0; i < NSPoints; ++i) {
    FluxContainer.AddToPoint(i, flux[i] * Weight);
  }


  // Free all gpu memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);

  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);

  hipFree(d_dt);
  hipFree(d_nt);
  hipFree(d_ns);

  hipFree(d_flux);

  hipFree(d_C0);
  hipFree(d_C2);
  hipFree(d_Omega);
  hipFree(d_C);


  // Free all heap memory
  delete [] x;
  delete [] y;
  delete [] z;

  delete [] bx;
  delete [] by;
  delete [] bz;

  delete [] sx;
  delete [] sy;
  delete [] sz;


  delete [] flux;


  return;
}










































__global__ void OSCARSSR_Cuda_SpectrumGPU (double *x, double *y, double *z, double *bx, double *by, double *bz, double *ox, double *oy, double *oz, double *dt, int *nt, int *ns, double *C0, double *C2, double *EvToOmega, double *C, double *se, double *sf, hipDoubleComplex* pol, int *pol_state)
{
  // Check that this is within the number of spectrum points requested
  int is = threadIdx.x + blockIdx.x * blockDim.x;
  if (is >= *ns) {
    return;
  }

  // Complex i
  hipDoubleComplex I = make_hipDoubleComplex(0, 1);

  double const Omega = *EvToOmega * se[is];
  hipDoubleComplex ICoverOmega = make_hipDoubleComplex(0, (*C) / Omega);


  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);


  // Loop over all points in trajectory
  for (int i = 0; i < *nt; ++i) {

    // Distance to observer
    double const D = sqrt( pow( (*ox) - x[i], 2) + pow( (*oy) - y[i], 2) + pow((*oz) - z[i], 2) );

    // Normal in direction of observer
    double const NX = ((*ox) - x[i]) / D;
    double const NY = ((*oy) - y[i]) / D;
    double const NZ = ((*oz) - z[i]) / D;

    // Exponent for fourier transformed field
    hipDoubleComplex Exponent = make_hipDoubleComplex(0, Omega * ((*dt) * i + D / (*C)));

    hipDoubleComplex X1 = make_hipDoubleComplex((bx[i] - NX) / D, -(*C) * NX / (Omega * D * D));
    hipDoubleComplex Y1 = make_hipDoubleComplex((by[i] - NY) / D, -(*C) * NY / (Omega * D * D));
    hipDoubleComplex Z1 = make_hipDoubleComplex((bz[i] - NZ) / D, -(*C) * NZ / (Omega * D * D));

    hipDoubleComplex MyEXP = cuCexp(Exponent);
    //hipDoubleComplex MyEXP = make_hipDoubleComplex( exp(Exponent.x) * cos(Exponent.y), exp(Exponent.x) * sin(Exponent.y));

    hipDoubleComplex X2 = hipCmul(X1, MyEXP);
    hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
    hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


    SumEX = hipCadd(SumEX, X2);
    SumEY = hipCadd(SumEY, Y2);
    SumEZ = hipCadd(SumEZ, Z2);

    // Sum in fourier transformed field (integral)
    //SumEX += (TVector3DC(B) - (N *     (One + (ICoverOmega / (D)))     )) / D * std::exp(Exponent);
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * Omega * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * Omega * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * Omega * (*dt)), SumEZ);


  // Check for polarization state
  if (*pol_state == 0) {
    // Do nothing
  } else if (*pol_state == 1) {
    // Linear, just dot with vector and put in direction of vector
    hipDoubleComplex Magnitude = hipCadd(hipCadd(hipCmul(SumEX, pol[0]), hipCmul(SumEY, pol[1])),  hipCmul(SumEZ, pol[2]));
    SumEX = hipCmul(Magnitude, pol[0]);
    SumEY = hipCmul(Magnitude, pol[1]);
    SumEZ = hipCmul(Magnitude, pol[2]);
  } else if (*pol_state == 2) {
    hipDoubleComplex Magnitude = hipCadd(hipCadd(hipCmul(SumEX, hipConj(pol[0])), hipCmul(SumEY, hipConj(pol[1]))),  hipCmul(SumEZ, hipConj(pol[2])));
    SumEX = hipCmul(Magnitude, pol[0]);
    SumEY = hipCmul(Magnitude, pol[1]);
    SumEZ = hipCmul(Magnitude, pol[2]);
  } else {
    // UPDATE: Serious problem
  }


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;

  // Multiply field by Constant C1 and time step
  //SumE *= C1 * DeltaT;

  // Set the flux for this frequency / energy point
  //Spectrum.AddToFlux(i, C2 *  SumE.Dot( SumE.CC() ).real() * Weight);

  sf[is] = (*C2) * (EX + EY + EZ);

  return;
}








extern "C" void OSCARSSR_Cuda_CalculateSpectrumGPU (TParticleA& Particle,
                                                    TVector3D const& ObservationPoint,
                                                    TSpectrumContainer& Spectrum,
                                                    std::string const& Polarization,
                                                    double const Angle,
                                                    TVector3D const& HorizontalDirection,
                                                    TVector3D const& PropogationDirection,
                                                    double const Weight)
{

  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Grab the Trajectory
  TParticleTrajectoryPoints& T = Particle.GetTrajectory();

  // Number of points in Trajectory
  int const NTPoints = (int) T.GetNPoints();

  // Timestep from trajectory
  double const DeltaT = T.GetDeltaT();

  double *x     = new double[NTPoints];
  double *y     = new double[NTPoints];
  double *z     = new double[NTPoints];
  double *bx    = new double[NTPoints];
  double *by    = new double[NTPoints];
  double *bz    = new double[NTPoints];

  int const NSPoints = (int) Spectrum.GetNPoints();

  // Observer
  double ox = ObservationPoint.GetX();
  double oy = ObservationPoint.GetY();
  double oz = ObservationPoint.GetZ();

  // Constants
  double const C = TOSCARSSR::C();
  double const EvToOmega = TOSCARSSR::EvToAngularFrequency(1);

  // Spectrum energy and flux
  double *se     = new double[NSPoints];
  double *sf     = new double[NSPoints];



  // Imaginary "i" and complxe 1+0i
  std::complex<double> const I(0, 1);
  std::complex<double> const One(1, 0);

  // Photon vertical direction and positive and negative helicity
  TVector3D const VerticalDirection = PropogationDirection.Cross(HorizontalDirection).UnitVector();
  TVector3DC const Positive = 1. / sqrt(2) * (TVector3DC(HorizontalDirection) + VerticalDirection * I );
  TVector3DC const Negative = 1. / sqrt(2) * (TVector3DC(HorizontalDirection) - VerticalDirection * I );

  // For polarization input to the gpu
  hipDoubleComplex *pol = new hipDoubleComplex[3];

  // State of polarization: 0 for all, 1 for linear, 2 for circular
  // (requires different threatment of vector pol interally)
  int pol_state = 1;

  if (Polarization == "all") {
    // Do nothing, it is already ALL
    pol_state = 0;
  } else if (Polarization == "linear-horizontal") {
    pol[0] = make_hipDoubleComplex(HorizontalDirection.GetX(), 0);
    pol[1] = make_hipDoubleComplex(HorizontalDirection.GetY(), 0);
    pol[2] = make_hipDoubleComplex(HorizontalDirection.GetZ(), 0);
  } else if (Polarization == "linear-vertical") {
    pol[0] = make_hipDoubleComplex(VerticalDirection.GetX(), 0);
    pol[1] = make_hipDoubleComplex(VerticalDirection.GetY(), 0);
    pol[2] = make_hipDoubleComplex(VerticalDirection.GetZ(), 0);
  } else if (Polarization == "linear") {
    TVector3D PolarizationAngle = HorizontalDirection;
    PolarizationAngle.RotateSelf(Angle, PropogationDirection);
    pol[0] = make_hipDoubleComplex(PolarizationAngle.GetX(), 0);
    pol[1] = make_hipDoubleComplex(PolarizationAngle.GetY(), 0);
    pol[2] = make_hipDoubleComplex(PolarizationAngle.GetZ(), 0);
  } else if (Polarization == "circular-left") {
    //SumE = SumE.Dot(Positive.CC()) * Positive;
    pol_state = 2;
    pol[0] = make_hipDoubleComplex(Positive.CC().GetX().real(), Positive.CC().GetX().imag());
    pol[1] = make_hipDoubleComplex(Positive.CC().GetY().real(), Positive.CC().GetY().imag());
    pol[2] = make_hipDoubleComplex(Positive.CC().GetZ().real(), Positive.CC().GetZ().imag());
  } else if (Polarization == "circular-right") {
    //SumE = SumE.Dot(Negative.CC()) * Negative;
    pol_state = 2;
    pol[0] = make_hipDoubleComplex(Negative.CC().GetX().real(), Negative.CC().GetX().imag());
    pol[1] = make_hipDoubleComplex(Negative.CC().GetY().real(), Negative.CC().GetY().imag());
    pol[2] = make_hipDoubleComplex(Negative.CC().GetZ().real(), Negative.CC().GetZ().imag());
  } else {
    // Throw invalid argument if polarization is not recognized
    //throw std::invalid_argument("Polarization requested not recognized");
  }

  // Set trajectory
  for (size_t i = 0; i < NTPoints; ++i) {
    x[i] = T.GetX(i).GetX();
    y[i] = T.GetX(i).GetY();
    z[i] = T.GetX(i).GetZ();

    bx[i] = T.GetB(i).GetX();
    by[i] = T.GetB(i).GetY();
    bz[i] = T.GetB(i).GetZ();
  }



  // Set energy to value
  for (size_t i = 0; i < NSPoints; ++i) {
    se[i] = Spectrum.GetEnergy(i);
  }



  double *d_x, *d_y, *d_z;
  double *d_bx, *d_by, *d_bz;
  double *d_ox, *d_oy, *d_oz;
  double *d_se, *d_sf;
  double *d_dt;
  int    *d_nt, *d_ns;

  // For polarization
  hipDoubleComplex *d_pol;
  int *d_pol_state;


  int const size_x = NTPoints * sizeof(double);
  int const size_s = NSPoints * sizeof(double);

  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_y, size_x);
  hipMalloc((void **) &d_z, size_x);

  hipMalloc((void **) &d_bx, size_x);
  hipMalloc((void **) &d_by, size_x);
  hipMalloc((void **) &d_bz, size_x);

  hipMalloc((void **) &d_ox, sizeof(double));
  hipMalloc((void **) &d_oy, sizeof(double));
  hipMalloc((void **) &d_oz, sizeof(double));

  hipMalloc((void **) &d_dt, sizeof(double));
  hipMalloc((void **) &d_nt, sizeof(int));
  hipMalloc((void **) &d_ns, sizeof(int));

  hipMalloc((void **) &d_se, size_s);
  hipMalloc((void **) &d_sf, size_s);

  // Polarization
  hipMalloc((void **) &d_pol, 3*sizeof(hipDoubleComplex));
  hipMalloc((void **) &d_pol_state, sizeof(int));


  hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_bx, bx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_by, by, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_bz, bz, size_x, hipMemcpyHostToDevice);


  hipMemcpy(d_ox, &ox, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_oy, &oy, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_oz, &oz, sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_dt, &DeltaT, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nt, &NTPoints, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_ns, &NSPoints, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(d_se, se, size_s, hipMemcpyHostToDevice);

  hipMemcpy(d_pol, pol, 3*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  hipMemcpy(d_pol_state, &pol_state, sizeof(int), hipMemcpyHostToDevice);


  // Constant C0 for calculation
  double const C0 = Particle.GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());

  // Constant for flux calculation at the end
  double const C2 = TOSCARSSR::FourPi() * Particle.GetCurrent() / (TOSCARSSR::H() * fabs(Particle.GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;

  // Constants to send in to GPU
  double *d_C0, *d_C2, *d_EvToOmega, *d_C;

  // Allocate memory for constants
  hipMalloc((void **) &d_C0,        sizeof(double));
  hipMalloc((void **) &d_C2,        sizeof(double));
  hipMalloc((void **) &d_EvToOmega, sizeof(double));
  hipMalloc((void **) &d_C,         sizeof(double));

  // Copy constants to GPU
  hipMemcpy(d_C0,        &C0,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_C2,        &C2,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_EvToOmega, &EvToOmega, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_C,         &C,         sizeof(double), hipMemcpyHostToDevice);


  // Send computation to gpu
  int const NBlocks = NSPoints / NTHREADS_PER_BLOCK + 1;
  OSCARSSR_Cuda_SpectrumGPU<<<NBlocks, NTHREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_bx, d_by, d_bz, d_ox, d_oy, d_oz, d_dt, d_nt, d_ns, d_C0, d_C2, d_EvToOmega, d_C, d_se, d_sf, d_pol, d_pol_state);

  // Copy result back from GPU
  hipMemcpy(sf, d_sf, size_s, hipMemcpyDeviceToHost);



  // Add result to power density container
  for (size_t i = 0; i < NSPoints; ++i) {
    Spectrum.AddToFlux(i, sf[i] * Weight);
  }


  // Free all gpu memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);

  hipFree(d_ox);
  hipFree(d_oy);
  hipFree(d_oz);

  hipFree(d_dt);
  hipFree(d_nt);
  hipFree(d_ns);

  hipFree(d_se);
  hipFree(d_sf);

  hipFree(d_C0);
  hipFree(d_C2);
  hipFree(d_EvToOmega);
  hipFree(d_C);

  hipFree(d_pol);
  hipFree(d_pol_state);


  // Free all heap memory
  delete [] x;
  delete [] y;
  delete [] z;

  delete [] bx;
  delete [] by;
  delete [] bz;


  delete [] se;
  delete [] sf;

  delete [] pol;


  return;
}


















__global__ void OSCARSSR_Cuda_PowerDensityGPU (double *x, double *y, double *z, double *bx, double *by, double *bz, double *aocx, double *aocy, double *aocz, double *sx, double *sy, double *sz, double *snx, double *sny, double *snz, double *dt, int *nt, int *ns, double *power_density)
{
  // Get surface id from block and thread number
  int is = threadIdx.x + blockIdx.x * blockDim.x;

  if (is >= *ns) {
    return;
  }




  // If you could copy int ultra-fast memory, something like this:
  //__shared__ double temp[6144];
  //if (threadIdx.x == 0) {
  //  for (int i = 0; i < *nt; ++i) {
  //    if (i <= 6144) {
  //      break;
  //    }
  //    temp[i] = x[i];
  //  }
  //}
  // __syncthreads();



  // Observation point
  double const OX = sx[is];
  double const OY = sy[is];
  double const OZ = sz[is];

  // Normal vector from input
  double const NormalX = snx[is];
  double const NormalY = sny[is];
  double const NormalZ = snz[is];

  double Sum = 0;

  for (int i = 0; i < *nt; ++i) {

    // Normal vector in direction of observation point
    double const R1 = sqrt( pow(OX - x[i], 2) + pow(OY - y[i], 2) + pow(OZ - z[i], 2) );
    double const N1X = (OX - x[i]) / R1;
    double const N1Y = (OY - y[i]) / R1;
    double const N1Z = (OZ - z[i]) / R1;

    // Surface normal dot with vector normal
    double const N1DotNormal = N1X * NormalX + N1Y * NormalY + N1Z * NormalZ;

    // Orthogonal vector 2 & 3
    double N2X;
    double N2Y;
    double N2Z;

    double const xx = N1X < 0.0 ? -N1X : N1X;
    double const yy = N1Y < 0.0 ? -N1Y : N1Y;
    double const zz = N1Z < 0.0 ? -N1Z : N1Z;
    if (xx < yy) {
      if (xx < zz) {
        N2X = 0;
        N2Y = N1Z;
        N2Z = -N1Y;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    } else {
      if (yy < zz) {
        N2X = -N1Z;
        N2Y = 0;
        N2Z = N1X;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    }
    double const R2 = sqrt(N2X * N2X + N2Y * N2Y + N2Z * N2Z);
    N2X /= R2;
    N2Y /= R2;
    N2Z /= R2;

    // Ortohgonal vector N3
    double const N3X = N1Y * N2Z - N1Z * N2Y;
    double const N3Y = N1Z * N2X - N1X * N2Z;
    double const N3Z = N1X * N2Y - N1Y * N2X;





    double const x1 = N1X - bx[i];
    double const y1 = N1Y - by[i];
    double const z1 = N1Z - bz[i];

    double const x2 = y1 * aocz[i] - z1 * aocy[i];
    double const y2 = z1 * aocx[i] - x1 * aocz[i];
    double const z2 = x1 * aocy[i] - y1 * aocx[i];

    // Numerator = N1.Cross( ( (N1 - B).Cross((AoverC)) ) );
    double const x3 = N1Y * z2 - N1Z * y2;
    double const y3 = N1Z * x2 - N1X * z2;
    double const z3 = N1X * y2 - N1Y * x2;

    double const BdotN1 = bx[i] * N1X + by[i] * N1Y + bz[i] * N1Z;
    double const Denominator = pow(1. - BdotN1, 5);

    Sum += pow( x3 * N2X + y3 * N2Y + z3 * N2Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
    Sum += pow( x3 * N3X + y3 * N3Y + z3 * N3Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
  }

  power_density[is] = Sum * (*dt);

  return;
}



extern "C" void OSCARSSR_Cuda_CalculatePowerDensityGPU (TParticleA& Particle,
                                                        TSurfacePoints const& Surface,
                                                        T3DScalarContainer& PowerDensityContainer,
                                                        bool const Directional,
                                                        double const Weight)
{

  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Grab the Trajectory
  TParticleTrajectoryPoints& T = Particle.GetTrajectory();

  // Number of points in Trajectory
  int const NTPoints = (int) T.GetNPoints();

  // Timestep from trajectory
  double const DeltaT = T.GetDeltaT();

  double *x     = new double[NTPoints];
  double *y     = new double[NTPoints];
  double *z     = new double[NTPoints];
  double *bx    = new double[NTPoints];
  double *by    = new double[NTPoints];
  double *bz    = new double[NTPoints];
  double *aocx  = new double[NTPoints];
  double *aocy  = new double[NTPoints];
  double *aocz  = new double[NTPoints];

  int const NSPoints = (int) Surface.GetNPoints();

  double *sx     = new double[NSPoints];
  double *sy     = new double[NSPoints];
  double *sz     = new double[NSPoints];

  double *snx    = new double[NSPoints];
  double *sny    = new double[NSPoints];
  double *snz    = new double[NSPoints];

  double *power_density = new double[NSPoints];


  for (size_t i = 0; i < NTPoints; ++i) {
    x[i] = T.GetX(i).GetX();
    y[i] = T.GetX(i).GetY();
    z[i] = T.GetX(i).GetZ();

    bx[i] = T.GetB(i).GetX();
    by[i] = T.GetB(i).GetY();
    bz[i] = T.GetB(i).GetZ();

    aocx[i] = T.GetAoverC(i).GetX();
    aocy[i] = T.GetAoverC(i).GetY();
    aocz[i] = T.GetAoverC(i).GetZ();
  }



  for (size_t i = 0; i < NSPoints; ++i) {
    sx[i] = Surface.GetPoint(i).GetX();
    sy[i] = Surface.GetPoint(i).GetY();
    sz[i] = Surface.GetPoint(i).GetZ();

    snx[i] = Surface.GetPoint(i).GetNormalX();
    sny[i] = Surface.GetPoint(i).GetNormalY();
    snz[i] = Surface.GetPoint(i).GetNormalZ();
  }



  double *d_x, *d_y, *d_z;
  double *d_bx, *d_by, *d_bz;
  double *d_aocx, *d_aocy, *d_aocz;
  double *d_sx, *d_sy, *d_sz;
  double *d_snx, *d_sny, *d_snz;
  double *d_power_density;
  double *d_dt;
  int    *d_nt, *d_ns;

  int const size_x = NTPoints * sizeof(double);
  int const size_s = NSPoints * sizeof(double);

  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_y, size_x);
  hipMalloc((void **) &d_z, size_x);

  hipMalloc((void **) &d_bx, size_x);
  hipMalloc((void **) &d_by, size_x);
  hipMalloc((void **) &d_bz, size_x);

  hipMalloc((void **) &d_aocx, size_x);
  hipMalloc((void **) &d_aocy, size_x);
  hipMalloc((void **) &d_aocz, size_x);

  hipMalloc((void **) &d_sx, size_s);
  hipMalloc((void **) &d_sy, size_s);
  hipMalloc((void **) &d_sz, size_s);

  hipMalloc((void **) &d_snx, size_s);
  hipMalloc((void **) &d_sny, size_s);
  hipMalloc((void **) &d_snz, size_s);

  hipMalloc((void **) &d_power_density, size_s);

  hipMalloc((void **) &d_dt, sizeof(double));

  hipMalloc((void **) &d_nt, sizeof(int));
  hipMalloc((void **) &d_ns, sizeof(int));


  hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_bx, bx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_by, by, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_bz, bz, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_aocx, aocx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_aocy, aocy, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_aocz, aocz, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_sx, sx, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sy, sy, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sz, sz, size_s, hipMemcpyHostToDevice);

  hipMemcpy(d_snx, snx, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sny, sny, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_snz, snz, size_s, hipMemcpyHostToDevice);

  hipMemcpy(d_dt, &DeltaT, sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_nt, &NTPoints, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_ns, &NSPoints, sizeof(int), hipMemcpyHostToDevice);


  // Send computation to gpu
  int const NBlocks = NSPoints / NTHREADS_PER_BLOCK + 1;
  OSCARSSR_Cuda_PowerDensityGPU<<<NBlocks, NTHREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_bx, d_by, d_bz, d_aocx, d_aocy, d_aocz, d_sx, d_sy, d_sz, d_snx, d_sny, d_snz, d_dt, d_nt, d_ns, d_power_density);

  // Copy result back from GPU
  hipMemcpy(power_density, d_power_density, size_s, hipMemcpyDeviceToHost);



  // Add result to power density container
  for (size_t i = 0; i < NSPoints; ++i) {
    PowerDensityContainer.AddToPoint(i, power_density[i] * fabs(Particle.GetQ() * Particle.GetCurrent()) / (16 * TOSCARSSR::Pi2() * TOSCARSSR::Epsilon0() * TOSCARSSR::C()) / 1e6 * Weight);
  }


  // Free all gpu memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);

  hipFree(d_aocx);
  hipFree(d_aocy);
  hipFree(d_aocz);

  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);

  hipFree(d_snx);
  hipFree(d_sny);
  hipFree(d_snz);

  hipFree(d_dt);
  hipFree(d_nt);
  hipFree(d_ns);

  hipFree(d_power_density);





  // Free all heap memory
  delete [] x;
  delete [] y;
  delete [] z;

  delete [] bx;
  delete [] by;
  delete [] bz;

  delete [] aocx;
  delete [] aocy;
  delete [] aocz;

  delete [] sx;
  delete [] sy;
  delete [] sz;

  delete [] snx;
  delete [] sny;
  delete [] snz;

  delete [] power_density;

  return;
}





