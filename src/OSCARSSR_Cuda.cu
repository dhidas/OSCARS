#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////
//
// Dean Andrew Hidas <dhidas@bnl.gov>
//
// Created on: Tue May 24 11:33:19 EDT 2016
//
////////////////////////////////////////////////////////////////////

#include <hip/hip_complex.h>

#include "OSCARSSR_Cuda.h"

#include "OSCARSSR.h"

#include <cmath>
#include <fstream>
#include <sstream>
#include <algorithm>

#include "TVector3DC.h"
#include "TSpectrumContainer.h"


#define NTHREADS_PER_BLOCK 512






extern "C" int OSCARSSR_Cuda_GetDeviceCount ()
{
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);

  return ngpu;
}





std::string OSCARSSR_Cuda_GetDeviceProperties (int const i)
{
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);

  char buf[300];

  if (i >= ngpu) {
    sprintf(buf, "ERROR: GPU %i Not available", i);
    return std::string(buf);
  }

  std::string ret = "";

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, i);

  sprintf(buf, "  Device name: %s\n", prop.name);
  ret += std::string(buf);
  sprintf(buf, "  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
  ret += std::string(buf);
  sprintf(buf, "  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
  ret += std::string(buf);
  sprintf(buf, "  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  ret += std::string(buf);

  return ret;
}





__device__ static __inline__ void Orthogonal(double *a, double *b)
{
  // Return a vector which is orthogonal vector a
  double xx = a[0] < 0.0 ? -a[0] : a[0];
  double yy = a[1] < 0.0 ? -a[1] : a[1];
  double zz = a[2] < 0.0 ? -a[2] : a[2];
  if (xx < yy) {
    if (xx < zz) {
      b[0] = 0;
      b[1] = a[2];
      b[2] = -a[1];
    } else {
      b[0] = a[1];
      b[1] = -a[0];
      b[2] = 0;
    }
  } else {
    if (yy < zz) {
      b[0] = -a[2];
      b[1] = 0;
      b[2] = a[0];
    } else {
      b[0] = a[1];
      b[1] = -a[0];
      b[2] = 0;
    }
  }
  return;
}



__host__ __device__ static __inline__ hipDoubleComplex cuCexp(hipDoubleComplex x)
{
  double factor = exp(x.x);
  return make_hipDoubleComplex(factor * cos(x.y), factor * sin(x.y));
}









__global__ void OSCARSSR_Cuda_FluxGPUMulti (double *x, double *y, double *z, double *bx, double *by, double *bz, double *sx, double *sy, double *sz, double *dt, int *nt, int *ns, double *C0, double *C2, double *C, double *Omega, int *ifirst, double *flux)
{
  // Check that this is within the number of spectrum points requested
  int const ith = threadIdx.x + blockIdx.x * blockDim.x;
  int const is = ith + *ifirst;
  if (is >= *ns) {
    return;
  }


  // Complex i
  hipDoubleComplex I = make_hipDoubleComplex(0, 1);

  hipDoubleComplex ICoverOmega = make_hipDoubleComplex(0, (*C) / (*Omega));

  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);


  // Loop over all points in trajectory
  for (int i = 0; i < *nt; ++i) {

    // Distance to observer
    double const D = sqrt( pow( (ox) - x[i], 2) + pow( (oy) - y[i], 2) + pow((oz) - z[i], 2) );

    // Normal in direction of observer
    double const NX = ((ox) - x[i]) / D;
    double const NY = ((oy) - y[i]) / D;
    double const NZ = ((oz) - z[i]) / D;

    // Exponent for fourier transformed field
    hipDoubleComplex Exponent = make_hipDoubleComplex(0, (*Omega) * ((*dt) * i + D / (*C)));

    hipDoubleComplex X1 = make_hipDoubleComplex((bx[i] - NX) / D, -(*C) * NX / ((*Omega) * D * D));
    hipDoubleComplex Y1 = make_hipDoubleComplex((by[i] - NY) / D, -(*C) * NY / ((*Omega) * D * D));
    hipDoubleComplex Z1 = make_hipDoubleComplex((bz[i] - NZ) / D, -(*C) * NZ / ((*Omega) * D * D));

    hipDoubleComplex MyEXP = cuCexp(Exponent);
    //hipDoubleComplex MyEXP = make_hipDoubleComplex( exp(Exponent.x) * cos(Exponent.y), exp(Exponent.x) * sin(Exponent.y));

    hipDoubleComplex X2 = hipCmul(X1, MyEXP);
    hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
    hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


    SumEX = hipCadd(SumEX, X2);
    SumEY = hipCadd(SumEY, Y2);
    SumEZ = hipCadd(SumEZ, Z2);

    // Sum in fourier transformed field (integral)
    //SumEX += (TVector3DC(B) - (N *     (One + (ICoverOmega / (D)))     )) / D * std::exp(Exponent);
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEZ);


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;

  // Multiply field by Constant C1 and time step
  //SumE *= C1 * DeltaT;

  // Set the flux for this frequency / energy point
  //Spectrum.AddToFlux(i, C2 *  SumE.Dot( SumE.CC() ).real() * Weight);

  flux[ith] = (*C2) * (EX + EY + EZ);

  return;
}








__global__ void OSCARSSR_Cuda_FluxGPU4 (double *x, double *y, double *z, double *bx, double *by, double *bz, double *sx, double *sy, double *sz, double *dt, int *nt, int *ns, double *C0, double *C2, double *C, double *Omega, double *flux)
{
  // Check that this is within the number of spectrum points requested
  int is = threadIdx.x + blockIdx.x * blockDim.x;
  // Max number for shared memory
  int const NSHAREDMAX = 1000;

  // Number for each thread to copy from global to shared memory
  int const NToCopyPerThread = (int) NSHAREDMAX / NTHREADS_PER_BLOCK;

  // Actual number of elements in shared memory to use
  int const NSHARED = NToCopyPerThread * NTHREADS_PER_BLOCK;

  // Define the shared memory
  __shared__ double sh_x[NSHARED];
  __shared__ double sh_y[NSHARED];
  __shared__ double sh_z[NSHARED];
  __shared__ double sh_bx[NSHARED];
  __shared__ double sh_by[NSHARED];
  __shared__ double sh_bz[NSHARED];


  // observer
  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);



  // Total number of copies
  int const NTotalCopies = *nt / NSHARED + 1;

  // Local offset for this thread in shared memory
  int const ThreadOffset = NToCopyPerThread * threadIdx.x;

  // icp is Copy Number
  for (int icp = 0; icp < NTotalCopies; ++icp) {

    // Offset instarting point in global array
    int const GlobalOffset = icp * NSHARED;


    __syncthreads();

    // icpth is the copy number in a thread
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_x[ThisThreadSharedIndex]  = x[GlobalIndex];
    }
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_y[ThisThreadSharedIndex]  = y[GlobalIndex];
    }
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_z[ThisThreadSharedIndex]  = z[GlobalIndex];
    }
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_bx[ThisThreadSharedIndex] = bx[GlobalIndex];
    }
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_by[ThisThreadSharedIndex] = by[GlobalIndex];
    }
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_bz[ThisThreadSharedIndex] = bz[GlobalIndex];
    }


    __syncthreads();

    if (is >= *ns) {
      continue;
    }

    for (int ish = 0; ish < NSHARED; ++ish) {
      int const i = GlobalOffset + ish;

      if (i >= *nt) {
        break;
      }



      // Distance to observer
      double const D = sqrt( pow( (ox) - sh_x[ish], 2) + pow( (oy) - sh_y[ish], 2) + pow((oz) - sh_z[ish], 2) );

      // Normal in direction of observer
      double const NX = ((ox) - sh_x[ish]) / D;
      double const NY = ((oy) - sh_y[ish]) / D;
      double const NZ = ((oz) - sh_z[ish]) / D;

      // Exponent for fourier transformed field
      hipDoubleComplex Exponent = make_hipDoubleComplex(0, (*Omega) * ((*dt) * i + D / (*C)));

      hipDoubleComplex X1 = make_hipDoubleComplex((sh_bx[ish] - NX) / D, -(*C) * NX / ((*Omega) * D * D));
      hipDoubleComplex Y1 = make_hipDoubleComplex((sh_by[ish] - NY) / D, -(*C) * NY / ((*Omega) * D * D));
      hipDoubleComplex Z1 = make_hipDoubleComplex((sh_bz[ish] - NZ) / D, -(*C) * NZ / ((*Omega) * D * D));

      hipDoubleComplex MyEXP = cuCexp(Exponent);

      hipDoubleComplex X2 = hipCmul(X1, MyEXP);
      hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
      hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


      SumEX = hipCadd(SumEX, X2);
      SumEY = hipCadd(SumEY, Y2);
      SumEZ = hipCadd(SumEZ, Z2);








    }
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEZ);


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;


  flux[is] = (*C2) * (EX + EY + EZ);

  return;
}


























__global__ void OSCARSSR_Cuda_FluxGPU3 (double *x, double *y, double *z, double *bx, double *by, double *bz, double *sx, double *sy, double *sz, double *dt, int *nt, int *ns, double *C0, double *C2, double *C, double *Omega, double *flux)
{
  // Check that this is within the number of spectrum points requested
  int is = threadIdx.x + blockIdx.x * blockDim.x;
  // Max number for shared memory

  int const NSHARED = 1000;

  // Define the shared memory
  __shared__ double sh_x[1000];
  __shared__ double sh_y[1000];
  __shared__ double sh_z[1000];
  __shared__ double sh_bx[1000];
  __shared__ double sh_by[1000];
  __shared__ double sh_bz[1000];


  // observer
  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);



  // Total number of copies
  int const NTotalCopies = *nt / NSHARED + 1;

  // icp is Copy Number
  for (int icp = 0; icp < NTotalCopies; ++icp) {

    // Offset instarting point in global array
    int const GlobalOffset = icp * NSHARED;

    __syncthreads();

    if (threadIdx.x == 0) {
      // icpth is the copy number in a thread
      for (int icpth = 0; icpth < NSHARED; ++icpth) {

        // index of *this* shared memory
        int const ThisThreadSharedIndex = icpth;

        // Global index of *this*
        int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

        // Check if we are within the nt bound
        if (GlobalIndex >= *nt) {
          break;
        }

        // Copy global memory to shared bank
        sh_x[ThisThreadSharedIndex]  = x[GlobalIndex];
        sh_y[ThisThreadSharedIndex]  = y[GlobalIndex];
        sh_z[ThisThreadSharedIndex]  = z[GlobalIndex];
        sh_bx[ThisThreadSharedIndex] = bx[GlobalIndex];
        sh_by[ThisThreadSharedIndex] = by[GlobalIndex];
        sh_bz[ThisThreadSharedIndex] = bz[GlobalIndex];
      }
    }
    __syncthreads();

    if (is >= *ns) {
      continue;
    }

    for (int ish = 0; ish < NSHARED; ++ish) {
      int const i = GlobalOffset + ish;

      if (i >= *nt) {
        break;
      }

      // Distance to observer
      double const D = sqrt( pow( (ox) - sh_x[ish], 2) + pow( (oy) - sh_y[ish], 2) + pow((oz) - sh_z[ish], 2) );

      // Normal in direction of observer
      double const NX = ((ox) - sh_x[ish]) / D;
      double const NY = ((oy) - sh_y[ish]) / D;
      double const NZ = ((oz) - sh_z[ish]) / D;

      // Exponent for fourier transformed field
      hipDoubleComplex Exponent = make_hipDoubleComplex(0, (*Omega) * ((*dt) * i + D / (*C)));

      hipDoubleComplex X1 = make_hipDoubleComplex((sh_bx[ish] - NX) / D, -(*C) * NX / ((*Omega) * D * D));
      hipDoubleComplex Y1 = make_hipDoubleComplex((sh_by[ish] - NY) / D, -(*C) * NY / ((*Omega) * D * D));
      hipDoubleComplex Z1 = make_hipDoubleComplex((sh_bz[ish] - NZ) / D, -(*C) * NZ / ((*Omega) * D * D));

      hipDoubleComplex MyEXP = cuCexp(Exponent);

      hipDoubleComplex X2 = hipCmul(X1, MyEXP);
      hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
      hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


      SumEX = hipCadd(SumEX, X2);
      SumEY = hipCadd(SumEY, Y2);
      SumEZ = hipCadd(SumEZ, Z2);
    }
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEZ);


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;


  flux[is] = (*C2) * (EX + EY + EZ);

  return;
}











__global__ void OSCARSSR_Cuda_FluxGPU2 (double *x, double *y, double *z, double *bx, double *by, double *bz, double *sx, double *sy, double *sz, double *dt, int *nt, int *ns, double *C0, double *C2, double *C, double *Omega, double *flux)
{
  // Check that this is within the number of spectrum points requested
  int is = threadIdx.x + blockIdx.x * blockDim.x;
  // Max number for shared memory
  int const NSHAREDMAX = 1000;

  // Number for each thread to copy from global to shared memory
  int const NToCopyPerThread = (int) NSHAREDMAX / NTHREADS_PER_BLOCK;

  // Actual number of elements in shared memory to use
  int const NSHARED = NToCopyPerThread * NTHREADS_PER_BLOCK;

  // Define the shared memory
  __shared__ double sh_x[NSHARED];
  __shared__ double sh_y[NSHARED];
  __shared__ double sh_z[NSHARED];
  __shared__ double sh_bx[NSHARED];
  __shared__ double sh_by[NSHARED];
  __shared__ double sh_bz[NSHARED];


  // observer
  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);



  // Total number of copies
  int const NTotalCopies = *nt / NSHARED + 1;

  // icp is Copy Number
  for (int icp = 0; icp < NTotalCopies; ++icp) {

    // Offset instarting point in global array
    int const GlobalOffset = icp * NSHARED;

    // Local offset for this thread in shared memory
    int const ThreadOffset = NToCopyPerThread * threadIdx.x;
    //flux[is] = shoffset; return;

    __syncthreads();

    // icpth is the copy number in a thread
    for (int icpth = 0; icpth < NToCopyPerThread; ++icpth) {

      // index of *this* shared memory
      int const ThisThreadSharedIndex = ThreadOffset + icpth;

      // Global index of *this*
      int const GlobalIndex = GlobalOffset + ThisThreadSharedIndex;

      // Check if we are within the nt bound
      if (GlobalIndex >= *nt) {
        break;
      }

      // Copy global memory to shared bank
      sh_x[ThisThreadSharedIndex]  = x[GlobalIndex];
      sh_y[ThisThreadSharedIndex]  = y[GlobalIndex];
      sh_z[ThisThreadSharedIndex]  = z[GlobalIndex];
      sh_bx[ThisThreadSharedIndex] = bx[GlobalIndex];
      sh_by[ThisThreadSharedIndex] = by[GlobalIndex];
      sh_bz[ThisThreadSharedIndex] = bz[GlobalIndex];
    }
    __syncthreads();

    if (is >= *ns) {
      continue;
    }

    for (int ish = 0; ish < NSHARED; ++ish) {
      int const i = GlobalOffset + ish;

      if (i >= *nt) {
        break;
      }



      // Distance to observer
      double const D = sqrt( pow( (ox) - sh_x[ish], 2) + pow( (oy) - sh_y[ish], 2) + pow((oz) - sh_z[ish], 2) );

      // Normal in direction of observer
      double const NX = ((ox) - sh_x[ish]) / D;
      double const NY = ((oy) - sh_y[ish]) / D;
      double const NZ = ((oz) - sh_z[ish]) / D;

      // Exponent for fourier transformed field
      hipDoubleComplex Exponent = make_hipDoubleComplex(0, (*Omega) * ((*dt) * i + D / (*C)));

      hipDoubleComplex X1 = make_hipDoubleComplex((sh_bx[ish] - NX) / D, -(*C) * NX / ((*Omega) * D * D));
      hipDoubleComplex Y1 = make_hipDoubleComplex((sh_by[ish] - NY) / D, -(*C) * NY / ((*Omega) * D * D));
      hipDoubleComplex Z1 = make_hipDoubleComplex((sh_bz[ish] - NZ) / D, -(*C) * NZ / ((*Omega) * D * D));

      hipDoubleComplex MyEXP = cuCexp(Exponent);

      hipDoubleComplex X2 = hipCmul(X1, MyEXP);
      hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
      hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


      SumEX = hipCadd(SumEX, X2);
      SumEY = hipCadd(SumEY, Y2);
      SumEZ = hipCadd(SumEZ, Z2);








    }
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEZ);


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;


  flux[is] = (*C2) * (EX + EY + EZ);

  return;
}








__global__ void OSCARSSR_Cuda_FluxGPU (double *x, double *y, double *z, double *bx, double *by, double *bz, double *sx, double *sy, double *sz, double *dt, int *nt, int *ns, double *C0, double *C2, double *C, double *Omega, double *flux)
{
  // Check that this is within the number of spectrum points requested
  int is = threadIdx.x + blockIdx.x * blockDim.x;
  if (is >= *ns) {
    return;
  }

  // Complex i
  hipDoubleComplex I = make_hipDoubleComplex(0, 1);

  hipDoubleComplex ICoverOmega = make_hipDoubleComplex(0, (*C) / (*Omega));

  double const ox = sx[is];
  double const oy = sy[is];
  double const oz = sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);


  // Loop over all points in trajectory
  for (int i = 0; i < *nt; ++i) {

    // Distance to observer
    double const D = sqrt( pow( (ox) - x[i], 2) + pow( (oy) - y[i], 2) + pow((oz) - z[i], 2) );

    // Normal in direction of observer
    double const NX = ((ox) - x[i]) / D;
    double const NY = ((oy) - y[i]) / D;
    double const NZ = ((oz) - z[i]) / D;

    // Exponent for fourier transformed field
    hipDoubleComplex Exponent = make_hipDoubleComplex(0, (*Omega) * ((*dt) * i + D / (*C)));

    hipDoubleComplex X1 = make_hipDoubleComplex((bx[i] - NX) / D, -(*C) * NX / ((*Omega) * D * D));
    hipDoubleComplex Y1 = make_hipDoubleComplex((by[i] - NY) / D, -(*C) * NY / ((*Omega) * D * D));
    hipDoubleComplex Z1 = make_hipDoubleComplex((bz[i] - NZ) / D, -(*C) * NZ / ((*Omega) * D * D));

    hipDoubleComplex MyEXP = cuCexp(Exponent);
    //hipDoubleComplex MyEXP = make_hipDoubleComplex( exp(Exponent.x) * cos(Exponent.y), exp(Exponent.x) * sin(Exponent.y));

    hipDoubleComplex X2 = hipCmul(X1, MyEXP);
    hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
    hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


    SumEX = hipCadd(SumEX, X2);
    SumEY = hipCadd(SumEY, Y2);
    SumEZ = hipCadd(SumEZ, Z2);

    // Sum in fourier transformed field (integral)
    //SumEX += (TVector3DC(B) - (N *     (One + (ICoverOmega / (D)))     )) / D * std::exp(Exponent);
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * (*Omega) * (*dt)), SumEZ);


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;

  // Multiply field by Constant C1 and time step
  //SumE *= C1 * DeltaT;

  // Set the flux for this frequency / energy point
  //Spectrum.AddToFlux(i, C2 *  SumE.Dot( SumE.CC() ).real() * Weight);

  flux[is] = (*C2) * (EX + EY + EZ);

  return;
}


















extern "C" void OSCARSSR_Cuda_CalculateFluxGPU (TParticleA& Particle,
                                                TSurfacePoints const& Surface,
                                                double const Energy_eV,
                                                T3DScalarContainer& FluxContainer,
                                                std::string const& Polarization,
                                                double const Angle,
                                                TVector3D const& HorizontalDirection,
                                                TVector3D const& PropogationDirection,
                                                double const Weight)
{
  // Do the setup for and call the GPU calculation of flux.  Your limitation here is only GPU memory.

  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Grab the Trajectory
  TParticleTrajectoryPoints& T = Particle.GetTrajectory();

  // Number of points in Trajectory
  int const NTPoints = (int) T.GetNPoints();

  // Timestep from trajectory
  double const DeltaT = T.GetDeltaT();

  double *x     = new double[NTPoints];
  double *y     = new double[NTPoints];
  double *z     = new double[NTPoints];
  double *bx    = new double[NTPoints];
  double *by    = new double[NTPoints];
  double *bz    = new double[NTPoints];


  int const NSPoints = (int) Surface.GetNPoints();

  // Observer
  double *sx     = new double[NSPoints];
  double *sy     = new double[NSPoints];
  double *sz     = new double[NSPoints];

  // Constants
  double const C = TOSCARSSR::C();
  double const Omega = TOSCARSSR::EvToAngularFrequency(Energy_eV);

  // Flux
  double *flux = new double[NSPoints];


  // Set trajectory
  for (size_t i = 0; i < NTPoints; ++i) {
    x[i] = T.GetX(i).GetX();
    y[i] = T.GetX(i).GetY();
    z[i] = T.GetX(i).GetZ();

    bx[i] = T.GetB(i).GetX();
    by[i] = T.GetB(i).GetY();
    bz[i] = T.GetB(i).GetZ();
  }

  // Set the surface points
  for (size_t i = 0; i < NSPoints; ++i) {
    sx[i] = Surface.GetPoint(i).GetX();
    sy[i] = Surface.GetPoint(i).GetY();
    sz[i] = Surface.GetPoint(i).GetZ();
  }




  double *d_x, *d_y, *d_z;
  double *d_bx, *d_by, *d_bz;
  double *d_sx, *d_sy, *d_sz;
  double *d_flux;
  double *d_dt;
  int    *d_nt, *d_ns;

  int const size_x = NTPoints * sizeof(double);
  int const size_s = NSPoints * sizeof(double);

  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_y, size_x);
  hipMalloc((void **) &d_z, size_x);

  hipMalloc((void **) &d_bx, size_x);
  hipMalloc((void **) &d_by, size_x);
  hipMalloc((void **) &d_bz, size_x);

  hipMalloc((void **) &d_sx, size_s);
  hipMalloc((void **) &d_sy, size_s);
  hipMalloc((void **) &d_sz, size_s);


  hipMalloc((void **) &d_dt, sizeof(double));
  hipMalloc((void **) &d_nt, sizeof(int));
  hipMalloc((void **) &d_ns, sizeof(int));

  hipMalloc((void **) &d_flux, size_s);


  hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_bx, bx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_by, by, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_bz, bz, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_sx, sx, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sy, sy, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sz, sz, size_s, hipMemcpyHostToDevice);



  hipMemcpy(d_dt, &DeltaT, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nt, &NTPoints, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_ns, &NSPoints, sizeof(int), hipMemcpyHostToDevice);



  // Constant C0 for calculation
  double const C0 = Particle.GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());

  // Constant for flux calculation at the end
  double const C2 = TOSCARSSR::FourPi() * Particle.GetCurrent() / (TOSCARSSR::H() * fabs(Particle.GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;

  // Constants to send in to GPU
  double *d_C0, *d_C2, *d_Omega, *d_C;

  // Allocate memory for constants
  hipMalloc((void **) &d_C0,        sizeof(double));
  hipMalloc((void **) &d_C2,        sizeof(double));
  hipMalloc((void **) &d_Omega,     sizeof(double));
  hipMalloc((void **) &d_C,         sizeof(double));

  // Copy constants to GPU
  hipMemcpy(d_C0,        &C0,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_C2,        &C2,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_Omega,     &Omega,     sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_C,         &C,         sizeof(double), hipMemcpyHostToDevice);


  // Send computation to gpu
  int const NBlocks = NSPoints / NTHREADS_PER_BLOCK + 1;
  OSCARSSR_Cuda_FluxGPU<<<NBlocks, NTHREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_bx, d_by, d_bz, d_sx, d_sy, d_sz, d_dt, d_nt, d_ns, d_C0, d_C2, d_C, d_Omega, d_flux);

  // Copy result back from GPU
  hipMemcpy(flux, d_flux, size_s, hipMemcpyDeviceToHost);



  // Add result to power density container
  for (size_t i = 0; i < NSPoints; ++i) {
    FluxContainer.AddToPoint(i, flux[i] * Weight);
  }


  // Free all gpu memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);

  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);

  hipFree(d_dt);
  hipFree(d_nt);
  hipFree(d_ns);

  hipFree(d_flux);

  hipFree(d_C0);
  hipFree(d_C2);
  hipFree(d_Omega);
  hipFree(d_C);


  // Free all heap memory
  delete [] x;
  delete [] y;
  delete [] z;

  delete [] bx;
  delete [] by;
  delete [] bz;

  delete [] sx;
  delete [] sy;
  delete [] sz;


  delete [] flux;


  return;
}






extern "C" void OSCARSSR_Cuda_CalculateFluxGPU2 (OSCARSSR& OSR,
                                                TSurfacePoints const& Surface,
                                                double const Energy_eV,
                                                T3DScalarContainer& FluxContainer,
                                                std::string const& Polarization,
                                                double const Angle,
                                                TVector3D const& HorizontalDirection,
                                                TVector3D const& PropogationDirection,
                                                int const NParticles,
                                                std::vector<int> const& GPUVector)
{
  // Calculate the flux for NParticles using the GPUs given in GPUVector.  Each particle's
  // trajectory will be sent to all GPUs for processing, meanwhile a new trajectory will
  // be calculated

  // Number of available GPUs
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Make sure that a gpu listed is within the range and not a duplicate
  std::vector<int> GPUsToUse;
  for (std::vector<int>::const_iterator it = GPUVector.begin(); it != GPUVector.end(); ++it) {
    if ( !(std::find(GPUsToUse.begin(), GPUsToUse.end(), *it) != GPUsToUse.end() && (*it < ngpu)) ) {
      GPUsToUse.push_back(*it);
    }
  }

  // Make sure we have at least one
  if (GPUsToUse.size() == 0) {
    throw std::invalid_argument("GPUs selected do not match hardware");
  }
  int const NGPUsToUse = (int) GPUsToUse.size();

  // Do we calculate for the current particle?
  bool const ThisParticleOnly = NParticles == 0 ? true : false;
  int  const NParticlesReally = ThisParticleOnly ? 1 : NParticles;
  // Type check, new particle if no type


  int *h_nt, *h_ns;
  double *h_dt;
  hipHostAlloc((void**) &h_nt, sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ns, sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_dt, sizeof(double), hipHostMallocWriteCombined | hipHostMallocMapped);

  // First one, set particle and trajectory
  if (!ThisParticleOnly) {
    OSR.SetNewParticle();
  }
  if (OSR.GetTrajectory().GetNPoints() == 0) {
    OSR.CalculateTrajectory();
  }

  // Needed number of points in the track and time step
  *h_nt = (int) OSR.GetTrajectory().GetNPoints();
  *h_ns = (int) Surface.GetNPoints();
  *h_dt = (double) OSR.GetTrajectory().GetDeltaT();


  int const NThreads = *h_ns;
  int const NThreadsPerBlock = 32*16;
  int const NThreadsRemainder = NThreads % NThreadsPerBlock;
  int const NBlocksTotal = (NThreads - 1) / NThreadsPerBlock + 1;
  int const NBlocksPerGPU = NBlocksTotal / NGPUsToUse;
  int const NRemainderBlocks = NBlocksTotal % NGPUsToUse;
  // UPDATE: To be modified
  int const NFlux = NThreadsPerBlock * (NBlocksPerGPU + (NRemainderBlocks > 0 ? 1 : 0));

  std::vector<int> NBlocksThisGPU(NGPUsToUse, NBlocksPerGPU);
  for (int i = 0; i < NRemainderBlocks; ++i) {
    ++NBlocksThisGPU[i];
  }

  // Memory allocation for Host
  double  *h_x,  *h_y,  *h_z,  *h_bx,  *h_by,  *h_bz,  *h_sx,  *h_sy,  *h_sz,   *h_c0,  *h_c2,  *h_c,  *h_omega;
  int     *h_ifirst;
  double **h_flux;
  hipHostAlloc((void**) &h_x,           *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_y,           *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_z,           *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bx,          *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_by,          *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bz,          *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sx,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sy,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sz,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c0,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c2,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c,                   sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_omega,               sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ifirst, NGPUsToUse * sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_flux,   NGPUsToUse * sizeof(double*), hipHostMallocWriteCombined | hipHostMallocMapped);
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    hipHostAlloc((void**) &(h_flux[i]), NFlux * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  }

  // First surface point for each gpu
  int NBlocksUsed = 0;
  for (int i = 0; i < NGPUsToUse; ++i) {
    h_ifirst[i] = NBlocksUsed * NThreadsPerBlock;
    std::cout << "ifirst " << h_ifirst[i] << std::endl;
    NBlocksUsed += NBlocksThisGPU[i];
  }


  // Memor allocations for GPU
  int    **d_nt;
  int    **d_ns;
  double **d_dt;
  double **d_x;
  double **d_y;
  double **d_z;
  double **d_bx;
  double **d_by;
  double **d_bz;
  double **d_sx;
  double **d_sy;
  double **d_sz;
  double **d_c0;
  double **d_c2;
  double **d_c;
  double **d_omega;
  int    **d_ifirst;
  double **d_flux;

  hipHostAlloc((void **) &d_nt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ns,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_dt,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_x,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_y,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_z,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_by,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sy,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c0,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c2,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_omega,  NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ifirst, NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_flux,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipMalloc((void **) &d_nt[i],             sizeof(int));
    hipMalloc((void **) &d_ns[i],             sizeof(int));
    hipMalloc((void **) &d_dt[i],             sizeof(double));
    hipMalloc((void **) &d_x[i],      *h_nt * sizeof(double));
    hipMalloc((void **) &d_y[i],      *h_nt * sizeof(double));
    hipMalloc((void **) &d_z[i],      *h_nt * sizeof(double));
    hipMalloc((void **) &d_bx[i],     *h_nt * sizeof(double));
    hipMalloc((void **) &d_by[i],     *h_nt * sizeof(double));
    hipMalloc((void **) &d_bz[i],     *h_nt * sizeof(double));
    hipMalloc((void **) &d_sx[i],     *h_ns * sizeof(double));
    hipMalloc((void **) &d_sy[i],     *h_ns * sizeof(double));
    hipMalloc((void **) &d_sz[i],     *h_ns * sizeof(double));
    hipMalloc((void **) &d_c0[i],             sizeof(double));
    hipMalloc((void **) &d_c2[i],             sizeof(double));
    hipMalloc((void **) &d_c[i],              sizeof(double));
    hipMalloc((void **) &d_omega[i],          sizeof(double));
    hipMalloc((void **) &d_ifirst[i],         sizeof(int));
    hipMalloc((void **) &d_flux[i],   NFlux * sizeof(double));

    // Copy device number to device
    hipMemcpyAsync(d_ifirst[i], &(h_ifirst[i]), sizeof(int), hipMemcpyHostToDevice);
  }

  // Compute known host values
  *h_c0    = OSR.GetCurrentParticle().GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());
  *h_c2    = TOSCARSSR::FourPi() * OSR.GetCurrentParticle().GetCurrent() / (TOSCARSSR::H() * fabs(OSR.GetCurrentParticle().GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;
  *h_c     = TOSCARSSR::C();
  *h_omega = TOSCARSSR::EvToAngularFrequency(Energy_eV);
  for (size_t i = 0; i < *h_ns; ++i) {
    h_sx[i] = Surface.GetPoint(i).GetX();
    h_sy[i] = Surface.GetPoint(i).GetY();
    h_sz[i] = Surface.GetPoint(i).GetZ();
  }

  // Copy constants to first device (async)
  int const d0 = GPUsToUse[0];
  hipSetDevice(d0);
  hipMemcpyAsync(d_nt[0],    h_nt,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_ns[0],    h_ns,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_dt[0],    h_dt,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c0[0],    h_c0,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c2[0],    h_c2,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c[0],     h_c,           sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_omega[0], h_omega,       sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sx[0],    h_sx,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sy[0],    h_sy,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sz[0],    h_sz,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_nt[i+1],     d1, d_nt[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_ns[i+1],     d1, d_ns[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_dt[i+1],     d1, d_dt[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c0[i+1],     d1, d_c0[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c2[i+1],     d1, d_c2[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c[i+1],      d1, d_c[i],      d, sizeof(double));
    hipMemcpyPeerAsync( d_omega[i+1],  d1, d_omega[i],  d, sizeof(double));
    hipMemcpyPeerAsync( d_sx[i+1],     d1, d_sx[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sy[i+1],     d1, d_sy[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sz[i+1],     d1, d_sz[i],     d, *h_ns * sizeof(double));
  }

  // Set first trajectory
  TParticleTrajectoryPoints const& T = OSR.GetTrajectory();
  for (size_t i = 0; i < *h_nt; ++i) {
    h_x[i]  = T.GetX(i).GetX();
    h_y[i]  = T.GetX(i).GetY();
    h_z[i]  = T.GetX(i).GetZ();
    h_bx[i] = T.GetB(i).GetX();
    h_by[i] = T.GetB(i).GetY();
    h_bz[i] = T.GetB(i).GetZ();
  }

  // Set the surface points
  // GPU events
  hipEvent_t *event_fluxcopy = new hipEvent_t[NGPUsToUse];
  for (int ig = 0; ig < NGPUsToUse; ++ig) {
    int const d = GPUsToUse[ig];
    hipSetDevice(d);
    hipEventCreate(&(event_fluxcopy[ig]));
  }

  // Enable peer (direct gpu-gpu) writes
  for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
    // Device number
    int const d  = GPUsToUse[ig];
    int const d1 = GPUsToUse[ig+1];
    int access;
    hipDeviceCanAccessPeer(&access, d, d1);
    if (access == 1) {
      hipSetDevice(d);
      hipDeviceEnablePeerAccess(d1, 0);
    }
  }

  // Loop over number of particles
  for (int ip = 0; ip < NParticlesReally; ++ip) {

    // Copy trajectory to first GPU, then internal async transfers (where possible)
    hipSetDevice(d0);
    hipMemcpyAsync(d_x[0],  h_x,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_y[0],  h_y,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_z[0],  h_z,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bx[0], h_bx, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_by[0], h_by, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bz[0], h_bz, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
      // Device number
      int const d  = GPUsToUse[ig];
      int const d1 = GPUsToUse[ig+1];
      hipSetDevice(d);
      hipMemcpyPeerAsync(d_x[ig+1],  d1, d_x[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_y[ig+1],  d1, d_y[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_z[ig+1],  d1, d_z[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bx[ig+1], d1, d_bx[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_by[ig+1], d1, d_by[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bz[ig+1], d1, d_bz[ig], d, *h_nt * sizeof(double));

    }

    // Wait for previous copy, start next one
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d = GPUsToUse[ig];
      hipSetDevice(d);
      hipEventSynchronize(event_fluxcopy[ig]);
      OSCARSSR_Cuda_FluxGPUMulti<<<NBlocksThisGPU[ig], NThreadsPerBlock>>>(d_x[ig], d_y[ig], d_z[ig], d_bx[ig], d_by[ig], d_bz[ig], d_sx[ig], d_sy[ig], d_sz[ig], d_dt[ig], d_nt[ig], d_ns[ig], d_c0[ig], d_c2[ig], d_c[ig], d_omega[ig], d_ifirst[ig], d_flux[ig]);
    }


    // Add result to flux container (from **previous**)
    if (ip > 0) {
      int NBlocksUsed = 0;
      for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
        for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
          if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
            break;
          }
          int iss = ith + NThreadsPerBlock * NBlocksUsed;
          FluxContainer.AddToPoint(iss, h_flux[ig][ith]);
        }
        NBlocksUsed += NBlocksThisGPU[ig];
      }
    }

    // Add copy back to streams
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d  = GPUsToUse[ig];
      hipSetDevice(d);
      hipMemcpyAsync(h_flux[ig],  d_flux[ig],  NFlux * sizeof(double), hipMemcpyDeviceToHost);
      hipEventRecord(event_fluxcopy[ig]);
    }

    // If it's not the last one, calculate a new trajectory
    if (ip < NParticlesReally - 1) {
      OSR.SetNewParticle();
      OSR.CalculateTrajectory();
      TParticleTrajectoryPoints const& T = OSR.GetTrajectory();

      for (size_t it = 0; it < *h_nt; ++it) {
        h_x[it]  = T.GetX(it).GetX();
        h_y[it]  = T.GetX(it).GetY();
        h_z[it]  = T.GetX(it).GetZ();
        h_bx[it] = T.GetB(it).GetX();
        h_by[it] = T.GetB(it).GetY();
        h_bz[it] = T.GetB(it).GetZ();
      }
    }
  }

  // Wait for last copy
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    hipEventSynchronize(event_fluxcopy[ig]);
  }

  // Add result to flux container (from **previous**)
  NBlocksUsed = 0;
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
      if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
        break;
      }
      int iss = ith + NThreadsPerBlock * NBlocksUsed;
      FluxContainer.AddToPoint(iss, h_flux[ig][ith]);
    }
    NBlocksUsed += NBlocksThisGPU[ig];
  }

  // Weighting for multi-particle
  double const Weight = 1.0 / (double) NParticlesReally;
  FluxContainer.WeightAll(Weight);

  // Free host memory
  hipHostFree(h_nt);
  hipHostFree(h_ns);
  hipHostFree(h_dt);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_bx);
  hipHostFree(h_by);
  hipHostFree(h_bz);
  hipHostFree(h_sx);
  hipHostFree(h_sy);
  hipHostFree(h_sz);
  hipHostFree(h_c0);
  hipHostFree(h_c2);
  hipHostFree(h_c);
  hipHostFree(h_omega);
  hipHostFree(h_ifirst);
  // Free host and GPU memory
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {

    hipHostFree(h_flux[i]);

    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipFree(d_nt[i]);
    hipFree(d_ns[i]);
    hipFree(d_dt[i]);
    hipFree(d_x[i]);
    hipFree(d_y[i]);
    hipFree(d_z[i]);
    hipFree(d_bx[i]);
    hipFree(d_by[i]);
    hipFree(d_bz[i]);
    hipFree(d_sx[i]);
    hipFree(d_sy[i]);
    hipFree(d_sz[i]);
    hipFree(d_c0[i]);
    hipFree(d_c2[i]);
    hipFree(d_c[i]);
    hipFree(d_omega[i]);
    hipFree(d_ifirst[i]);
    hipFree(d_flux[i]);
  }
  hipFree(h_flux);

  hipFree(d_nt);
  hipFree(d_ns);
  hipFree(d_dt);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);
  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);
  hipFree(d_c0);
  hipFree(d_c2);
  hipFree(d_c);
  hipFree(d_omega);
  hipFree(h_ifirst);
  hipFree(d_flux);

  // Delete host gpu pointer arrays
  delete [] event_fluxcopy;


  return;
}



































__global__ void OSCARSSR_Cuda_SpectrumGPU (double *x, double *y, double *z, double *bx, double *by, double *bz, double *ox, double *oy, double *oz, double *dt, int *nt, int *ns, double *C0, double *C2, double *EvToOmega, double *C, double *se, double *sf, hipDoubleComplex* pol, int *pol_state)
{
  // Check that this is within the number of spectrum points requested
  int is = threadIdx.x + blockIdx.x * blockDim.x;
  if (is >= *ns) {
    return;
  }

  // Complex i
  hipDoubleComplex I = make_hipDoubleComplex(0, 1);

  double const Omega = *EvToOmega * se[is];
  hipDoubleComplex ICoverOmega = make_hipDoubleComplex(0, (*C) / Omega);


  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);


  // Loop over all points in trajectory
  for (int i = 0; i < *nt; ++i) {

    // Distance to observer
    double const D = sqrt( pow( (*ox) - x[i], 2) + pow( (*oy) - y[i], 2) + pow((*oz) - z[i], 2) );

    // Normal in direction of observer
    double const NX = ((*ox) - x[i]) / D;
    double const NY = ((*oy) - y[i]) / D;
    double const NZ = ((*oz) - z[i]) / D;

    // Exponent for fourier transformed field
    hipDoubleComplex Exponent = make_hipDoubleComplex(0, Omega * ((*dt) * i + D / (*C)));

    hipDoubleComplex X1 = make_hipDoubleComplex((bx[i] - NX) / D, -(*C) * NX / (Omega * D * D));
    hipDoubleComplex Y1 = make_hipDoubleComplex((by[i] - NY) / D, -(*C) * NY / (Omega * D * D));
    hipDoubleComplex Z1 = make_hipDoubleComplex((bz[i] - NZ) / D, -(*C) * NZ / (Omega * D * D));

    hipDoubleComplex MyEXP = cuCexp(Exponent);
    //hipDoubleComplex MyEXP = make_hipDoubleComplex( exp(Exponent.x) * cos(Exponent.y), exp(Exponent.x) * sin(Exponent.y));

    hipDoubleComplex X2 = hipCmul(X1, MyEXP);
    hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
    hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);


    SumEX = hipCadd(SumEX, X2);
    SumEY = hipCadd(SumEY, Y2);
    SumEZ = hipCadd(SumEZ, Z2);

    // Sum in fourier transformed field (integral)
    //SumEX += (TVector3DC(B) - (N *     (One + (ICoverOmega / (D)))     )) / D * std::exp(Exponent);
  }

  SumEX = hipCmul(make_hipDoubleComplex(0, (*C0) * Omega * (*dt)), SumEX);
  SumEY = hipCmul(make_hipDoubleComplex(0, (*C0) * Omega * (*dt)), SumEY);
  SumEZ = hipCmul(make_hipDoubleComplex(0, (*C0) * Omega * (*dt)), SumEZ);


  // Check for polarization state
  if (*pol_state == 0) {
    // Do nothing
  } else if (*pol_state == 1) {
    // Linear, just dot with vector and put in direction of vector
    hipDoubleComplex Magnitude = hipCadd(hipCadd(hipCmul(SumEX, pol[0]), hipCmul(SumEY, pol[1])),  hipCmul(SumEZ, pol[2]));
    SumEX = hipCmul(Magnitude, pol[0]);
    SumEY = hipCmul(Magnitude, pol[1]);
    SumEZ = hipCmul(Magnitude, pol[2]);
  } else if (*pol_state == 2) {
    hipDoubleComplex Magnitude = hipCadd(hipCadd(hipCmul(SumEX, hipConj(pol[0])), hipCmul(SumEY, hipConj(pol[1]))),  hipCmul(SumEZ, hipConj(pol[2])));
    SumEX = hipCmul(Magnitude, pol[0]);
    SumEY = hipCmul(Magnitude, pol[1]);
    SumEZ = hipCmul(Magnitude, pol[2]);
  } else {
    // UPDATE: Serious problem
  }


  double const EX = SumEX.x * SumEX.x + SumEX.y * SumEX.y;
  double const EY = SumEY.x * SumEY.x + SumEY.y * SumEY.y;
  double const EZ = SumEZ.x * SumEZ.x + SumEZ.y * SumEZ.y;

  // Multiply field by Constant C1 and time step
  //SumE *= C1 * DeltaT;

  // Set the flux for this frequency / energy point
  //Spectrum.AddToFlux(i, C2 *  SumE.Dot( SumE.CC() ).real() * Weight);

  sf[is] = (*C2) * (EX + EY + EZ);

  return;
}








extern "C" void OSCARSSR_Cuda_CalculateSpectrumGPU (TParticleA& Particle,
                                                    TVector3D const& ObservationPoint,
                                                    TSpectrumContainer& Spectrum,
                                                    std::string const& Polarization,
                                                    double const Angle,
                                                    TVector3D const& HorizontalDirection,
                                                    TVector3D const& PropogationDirection,
                                                    double const Weight)
{

  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Grab the Trajectory
  TParticleTrajectoryPoints& T = Particle.GetTrajectory();

  // Number of points in Trajectory
  int const NTPoints = (int) T.GetNPoints();

  // Timestep from trajectory
  double const DeltaT = T.GetDeltaT();

  double *x     = new double[NTPoints];
  double *y     = new double[NTPoints];
  double *z     = new double[NTPoints];
  double *bx    = new double[NTPoints];
  double *by    = new double[NTPoints];
  double *bz    = new double[NTPoints];

  int const NSPoints = (int) Spectrum.GetNPoints();

  // Observer
  double ox = ObservationPoint.GetX();
  double oy = ObservationPoint.GetY();
  double oz = ObservationPoint.GetZ();

  // Constants
  double const C = TOSCARSSR::C();
  double const EvToOmega = TOSCARSSR::EvToAngularFrequency(1);

  // Spectrum energy and flux
  double *se     = new double[NSPoints];
  double *sf     = new double[NSPoints];



  // Imaginary "i" and complxe 1+0i
  std::complex<double> const I(0, 1);
  std::complex<double> const One(1, 0);

  // Photon vertical direction and positive and negative helicity
  TVector3D const VerticalDirection = PropogationDirection.Cross(HorizontalDirection).UnitVector();
  TVector3DC const Positive = 1. / sqrt(2) * (TVector3DC(HorizontalDirection) + VerticalDirection * I );
  TVector3DC const Negative = 1. / sqrt(2) * (TVector3DC(HorizontalDirection) - VerticalDirection * I );

  // For polarization input to the gpu
  hipDoubleComplex *pol = new hipDoubleComplex[3];

  // State of polarization: 0 for all, 1 for linear, 2 for circular
  // (requires different threatment of vector pol interally)
  int pol_state = 1;

  if (Polarization == "all") {
    // Do nothing, it is already ALL
    pol_state = 0;
  } else if (Polarization == "linear-horizontal") {
    pol[0] = make_hipDoubleComplex(HorizontalDirection.GetX(), 0);
    pol[1] = make_hipDoubleComplex(HorizontalDirection.GetY(), 0);
    pol[2] = make_hipDoubleComplex(HorizontalDirection.GetZ(), 0);
  } else if (Polarization == "linear-vertical") {
    pol[0] = make_hipDoubleComplex(VerticalDirection.GetX(), 0);
    pol[1] = make_hipDoubleComplex(VerticalDirection.GetY(), 0);
    pol[2] = make_hipDoubleComplex(VerticalDirection.GetZ(), 0);
  } else if (Polarization == "linear") {
    TVector3D PolarizationAngle = HorizontalDirection;
    PolarizationAngle.RotateSelf(Angle, PropogationDirection);
    pol[0] = make_hipDoubleComplex(PolarizationAngle.GetX(), 0);
    pol[1] = make_hipDoubleComplex(PolarizationAngle.GetY(), 0);
    pol[2] = make_hipDoubleComplex(PolarizationAngle.GetZ(), 0);
  } else if (Polarization == "circular-left") {
    //SumE = SumE.Dot(Positive.CC()) * Positive;
    pol_state = 2;
    pol[0] = make_hipDoubleComplex(Positive.CC().GetX().real(), Positive.CC().GetX().imag());
    pol[1] = make_hipDoubleComplex(Positive.CC().GetY().real(), Positive.CC().GetY().imag());
    pol[2] = make_hipDoubleComplex(Positive.CC().GetZ().real(), Positive.CC().GetZ().imag());
  } else if (Polarization == "circular-right") {
    //SumE = SumE.Dot(Negative.CC()) * Negative;
    pol_state = 2;
    pol[0] = make_hipDoubleComplex(Negative.CC().GetX().real(), Negative.CC().GetX().imag());
    pol[1] = make_hipDoubleComplex(Negative.CC().GetY().real(), Negative.CC().GetY().imag());
    pol[2] = make_hipDoubleComplex(Negative.CC().GetZ().real(), Negative.CC().GetZ().imag());
  } else {
    // Throw invalid argument if polarization is not recognized
    //throw std::invalid_argument("Polarization requested not recognized");
  }

  // Set trajectory
  for (size_t i = 0; i < NTPoints; ++i) {
    x[i] = T.GetX(i).GetX();
    y[i] = T.GetX(i).GetY();
    z[i] = T.GetX(i).GetZ();

    bx[i] = T.GetB(i).GetX();
    by[i] = T.GetB(i).GetY();
    bz[i] = T.GetB(i).GetZ();
  }



  // Set energy to value
  for (size_t i = 0; i < NSPoints; ++i) {
    se[i] = Spectrum.GetEnergy(i);
  }



  double *d_x, *d_y, *d_z;
  double *d_bx, *d_by, *d_bz;
  double *d_ox, *d_oy, *d_oz;
  double *d_se, *d_sf;
  double *d_dt;
  int    *d_nt, *d_ns;

  // For polarization
  hipDoubleComplex *d_pol;
  int *d_pol_state;


  int const size_x = NTPoints * sizeof(double);
  int const size_s = NSPoints * sizeof(double);

  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_y, size_x);
  hipMalloc((void **) &d_z, size_x);

  hipMalloc((void **) &d_bx, size_x);
  hipMalloc((void **) &d_by, size_x);
  hipMalloc((void **) &d_bz, size_x);

  hipMalloc((void **) &d_ox, sizeof(double));
  hipMalloc((void **) &d_oy, sizeof(double));
  hipMalloc((void **) &d_oz, sizeof(double));

  hipMalloc((void **) &d_dt, sizeof(double));
  hipMalloc((void **) &d_nt, sizeof(int));
  hipMalloc((void **) &d_ns, sizeof(int));

  hipMalloc((void **) &d_se, size_s);
  hipMalloc((void **) &d_sf, size_s);

  // Polarization
  hipMalloc((void **) &d_pol, 3*sizeof(hipDoubleComplex));
  hipMalloc((void **) &d_pol_state, sizeof(int));


  hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_bx, bx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_by, by, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_bz, bz, size_x, hipMemcpyHostToDevice);


  hipMemcpy(d_ox, &ox, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_oy, &oy, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_oz, &oz, sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_dt, &DeltaT, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_nt, &NTPoints, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_ns, &NSPoints, sizeof(int), hipMemcpyHostToDevice);

  hipMemcpy(d_se, se, size_s, hipMemcpyHostToDevice);

  hipMemcpy(d_pol, pol, 3*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  hipMemcpy(d_pol_state, &pol_state, sizeof(int), hipMemcpyHostToDevice);


  // Constant C0 for calculation
  double const C0 = Particle.GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());

  // Constant for flux calculation at the end
  double const C2 = TOSCARSSR::FourPi() * Particle.GetCurrent() / (TOSCARSSR::H() * fabs(Particle.GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;

  // Constants to send in to GPU
  double *d_C0, *d_C2, *d_EvToOmega, *d_C;

  // Allocate memory for constants
  hipMalloc((void **) &d_C0,        sizeof(double));
  hipMalloc((void **) &d_C2,        sizeof(double));
  hipMalloc((void **) &d_EvToOmega, sizeof(double));
  hipMalloc((void **) &d_C,         sizeof(double));

  // Copy constants to GPU
  hipMemcpy(d_C0,        &C0,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_C2,        &C2,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_EvToOmega, &EvToOmega, sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_C,         &C,         sizeof(double), hipMemcpyHostToDevice);


  // Send computation to gpu
  int const NBlocks = NSPoints / NTHREADS_PER_BLOCK + 1;
  OSCARSSR_Cuda_SpectrumGPU<<<NBlocks, NTHREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_bx, d_by, d_bz, d_ox, d_oy, d_oz, d_dt, d_nt, d_ns, d_C0, d_C2, d_EvToOmega, d_C, d_se, d_sf, d_pol, d_pol_state);

  // Copy result back from GPU
  hipMemcpy(sf, d_sf, size_s, hipMemcpyDeviceToHost);



  // Add result to power density container
  for (size_t i = 0; i < NSPoints; ++i) {
    Spectrum.AddToFlux(i, sf[i] * Weight);
  }


  // Free all gpu memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);

  hipFree(d_ox);
  hipFree(d_oy);
  hipFree(d_oz);

  hipFree(d_dt);
  hipFree(d_nt);
  hipFree(d_ns);

  hipFree(d_se);
  hipFree(d_sf);

  hipFree(d_C0);
  hipFree(d_C2);
  hipFree(d_EvToOmega);
  hipFree(d_C);

  hipFree(d_pol);
  hipFree(d_pol_state);


  // Free all heap memory
  delete [] x;
  delete [] y;
  delete [] z;

  delete [] bx;
  delete [] by;
  delete [] bz;


  delete [] se;
  delete [] sf;

  delete [] pol;


  return;
}


















__global__ void OSCARSSR_Cuda_PowerDensityGPU (double *x, double *y, double *z, double *bx, double *by, double *bz, double *aocx, double *aocy, double *aocz, double *sx, double *sy, double *sz, double *snx, double *sny, double *snz, double *dt, int *nt, int *ns, double *power_density)
{
  // Get surface id from block and thread number
  int is = threadIdx.x + blockIdx.x * blockDim.x;

  if (is >= *ns) {
    return;
  }




  // If you could copy int ultra-fast memory, something like this:
  //__shared__ double temp[6144];
  //if (threadIdx.x == 0) {
  //  for (int i = 0; i < *nt; ++i) {
  //    if (i <= 6144) {
  //      break;
  //    }
  //    temp[i] = x[i];
  //  }
  //}
  // __syncthreads();



  // Observation point
  double const OX = sx[is];
  double const OY = sy[is];
  double const OZ = sz[is];

  // Normal vector from input
  double const NormalX = snx[is];
  double const NormalY = sny[is];
  double const NormalZ = snz[is];

  double Sum = 0;

  for (int i = 0; i < *nt; ++i) {

    // Normal vector in direction of observation point
    double const R1 = sqrt( pow(OX - x[i], 2) + pow(OY - y[i], 2) + pow(OZ - z[i], 2) );
    double const N1X = (OX - x[i]) / R1;
    double const N1Y = (OY - y[i]) / R1;
    double const N1Z = (OZ - z[i]) / R1;

    // Surface normal dot with vector normal
    double const N1DotNormal = N1X * NormalX + N1Y * NormalY + N1Z * NormalZ;

    // Orthogonal vector 2 & 3
    double N2X;
    double N2Y;
    double N2Z;

    double const xx = N1X < 0.0 ? -N1X : N1X;
    double const yy = N1Y < 0.0 ? -N1Y : N1Y;
    double const zz = N1Z < 0.0 ? -N1Z : N1Z;
    if (xx < yy) {
      if (xx < zz) {
        N2X = 0;
        N2Y = N1Z;
        N2Z = -N1Y;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    } else {
      if (yy < zz) {
        N2X = -N1Z;
        N2Y = 0;
        N2Z = N1X;
      } else {
        N2X = N1Y;
        N2Y = -N1X;
        N2Z = 0;
      }
    }
    double const R2 = sqrt(N2X * N2X + N2Y * N2Y + N2Z * N2Z);
    N2X /= R2;
    N2Y /= R2;
    N2Z /= R2;

    // Ortohgonal vector N3
    double const N3X = N1Y * N2Z - N1Z * N2Y;
    double const N3Y = N1Z * N2X - N1X * N2Z;
    double const N3Z = N1X * N2Y - N1Y * N2X;





    double const x1 = N1X - bx[i];
    double const y1 = N1Y - by[i];
    double const z1 = N1Z - bz[i];

    double const x2 = y1 * aocz[i] - z1 * aocy[i];
    double const y2 = z1 * aocx[i] - x1 * aocz[i];
    double const z2 = x1 * aocy[i] - y1 * aocx[i];

    // Numerator = N1.Cross( ( (N1 - B).Cross((AoverC)) ) );
    double const x3 = N1Y * z2 - N1Z * y2;
    double const y3 = N1Z * x2 - N1X * z2;
    double const z3 = N1X * y2 - N1Y * x2;

    double const BdotN1 = bx[i] * N1X + by[i] * N1Y + bz[i] * N1Z;
    double const Denominator = pow(1. - BdotN1, 5);

    Sum += pow( x3 * N2X + y3 * N2Y + z3 * N2Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
    Sum += pow( x3 * N3X + y3 * N3Y + z3 * N3Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
  }

  power_density[is] = Sum * (*dt);

  return;
}



extern "C" void OSCARSSR_Cuda_CalculatePowerDensityGPU (TParticleA& Particle,
                                                        TSurfacePoints const& Surface,
                                                        T3DScalarContainer& PowerDensityContainer,
                                                        bool const Directional,
                                                        double const Weight)
{

  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Grab the Trajectory
  TParticleTrajectoryPoints& T = Particle.GetTrajectory();

  // Number of points in Trajectory
  int const NTPoints = (int) T.GetNPoints();

  // Timestep from trajectory
  double const DeltaT = T.GetDeltaT();

  double *x     = new double[NTPoints];
  double *y     = new double[NTPoints];
  double *z     = new double[NTPoints];
  double *bx    = new double[NTPoints];
  double *by    = new double[NTPoints];
  double *bz    = new double[NTPoints];
  double *aocx  = new double[NTPoints];
  double *aocy  = new double[NTPoints];
  double *aocz  = new double[NTPoints];

  int const NSPoints = (int) Surface.GetNPoints();

  double *sx     = new double[NSPoints];
  double *sy     = new double[NSPoints];
  double *sz     = new double[NSPoints];

  double *snx    = new double[NSPoints];
  double *sny    = new double[NSPoints];
  double *snz    = new double[NSPoints];

  double *power_density = new double[NSPoints];


  for (size_t i = 0; i < NTPoints; ++i) {
    x[i] = T.GetX(i).GetX();
    y[i] = T.GetX(i).GetY();
    z[i] = T.GetX(i).GetZ();

    bx[i] = T.GetB(i).GetX();
    by[i] = T.GetB(i).GetY();
    bz[i] = T.GetB(i).GetZ();

    aocx[i] = T.GetAoverC(i).GetX();
    aocy[i] = T.GetAoverC(i).GetY();
    aocz[i] = T.GetAoverC(i).GetZ();
  }



  for (size_t i = 0; i < NSPoints; ++i) {
    sx[i] = Surface.GetPoint(i).GetX();
    sy[i] = Surface.GetPoint(i).GetY();
    sz[i] = Surface.GetPoint(i).GetZ();

    snx[i] = Surface.GetPoint(i).GetNormalX();
    sny[i] = Surface.GetPoint(i).GetNormalY();
    snz[i] = Surface.GetPoint(i).GetNormalZ();
  }



  double *d_x, *d_y, *d_z;
  double *d_bx, *d_by, *d_bz;
  double *d_aocx, *d_aocy, *d_aocz;
  double *d_sx, *d_sy, *d_sz;
  double *d_snx, *d_sny, *d_snz;
  double *d_power_density;
  double *d_dt;
  int    *d_nt, *d_ns;

  int const size_x = NTPoints * sizeof(double);
  int const size_s = NSPoints * sizeof(double);

  hipMalloc((void **) &d_x, size_x);
  hipMalloc((void **) &d_y, size_x);
  hipMalloc((void **) &d_z, size_x);

  hipMalloc((void **) &d_bx, size_x);
  hipMalloc((void **) &d_by, size_x);
  hipMalloc((void **) &d_bz, size_x);

  hipMalloc((void **) &d_aocx, size_x);
  hipMalloc((void **) &d_aocy, size_x);
  hipMalloc((void **) &d_aocz, size_x);

  hipMalloc((void **) &d_sx, size_s);
  hipMalloc((void **) &d_sy, size_s);
  hipMalloc((void **) &d_sz, size_s);

  hipMalloc((void **) &d_snx, size_s);
  hipMalloc((void **) &d_sny, size_s);
  hipMalloc((void **) &d_snz, size_s);

  hipMalloc((void **) &d_power_density, size_s);

  hipMalloc((void **) &d_dt, sizeof(double));

  hipMalloc((void **) &d_nt, sizeof(int));
  hipMalloc((void **) &d_ns, sizeof(int));


  hipMemcpy(d_x, x, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_z, z, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_bx, bx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_by, by, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_bz, bz, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_aocx, aocx, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_aocy, aocy, size_x, hipMemcpyHostToDevice);
  hipMemcpy(d_aocz, aocz, size_x, hipMemcpyHostToDevice);

  hipMemcpy(d_sx, sx, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sy, sy, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sz, sz, size_s, hipMemcpyHostToDevice);

  hipMemcpy(d_snx, snx, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_sny, sny, size_s, hipMemcpyHostToDevice);
  hipMemcpy(d_snz, snz, size_s, hipMemcpyHostToDevice);

  hipMemcpy(d_dt, &DeltaT, sizeof(double), hipMemcpyHostToDevice);

  hipMemcpy(d_nt, &NTPoints, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_ns, &NSPoints, sizeof(int), hipMemcpyHostToDevice);


  // Send computation to gpu
  int const NBlocks = NSPoints / NTHREADS_PER_BLOCK + 1;
  OSCARSSR_Cuda_PowerDensityGPU<<<NBlocks, NTHREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_bx, d_by, d_bz, d_aocx, d_aocy, d_aocz, d_sx, d_sy, d_sz, d_snx, d_sny, d_snz, d_dt, d_nt, d_ns, d_power_density);

  // Copy result back from GPU
  hipMemcpy(power_density, d_power_density, size_s, hipMemcpyDeviceToHost);



  // Add result to power density container
  for (size_t i = 0; i < NSPoints; ++i) {
    PowerDensityContainer.AddToPoint(i, power_density[i] * fabs(Particle.GetQ() * Particle.GetCurrent()) / (16 * TOSCARSSR::Pi2() * TOSCARSSR::Epsilon0() * TOSCARSSR::C()) / 1e6 * Weight);
  }


  // Free all gpu memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);

  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);

  hipFree(d_aocx);
  hipFree(d_aocy);
  hipFree(d_aocz);

  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);

  hipFree(d_snx);
  hipFree(d_sny);
  hipFree(d_snz);

  hipFree(d_dt);
  hipFree(d_nt);
  hipFree(d_ns);

  hipFree(d_power_density);





  // Free all heap memory
  delete [] x;
  delete [] y;
  delete [] z;

  delete [] bx;
  delete [] by;
  delete [] bz;

  delete [] aocx;
  delete [] aocy;
  delete [] aocz;

  delete [] sx;
  delete [] sy;
  delete [] sz;

  delete [] snx;
  delete [] sny;
  delete [] snz;

  delete [] power_density;

  return;
}





