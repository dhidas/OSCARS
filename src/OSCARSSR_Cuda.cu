#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////
//
// Dean Andrew Hidas <dhidas@bnl.gov>
//
// Created on: Tue May 24 11:33:19 EDT 2016
//
////////////////////////////////////////////////////////////////////

#include <hip/hip_complex.h>

#include "OSCARSSR_Cuda.h"

#include "OSCARSSR.h"

#include <cmath>
#include <fstream>
#include <sstream>
#include <algorithm>

#include "TVector3DC.h"
#include "TSpectrumContainer.h"


#define NTHREADS_PER_BLOCK 512






extern "C" int OSCARSSR_Cuda_GetDeviceCount ()
{
  static int ngpu = 0;
  hipGetDeviceCount(&ngpu);

  return ngpu;
}





std::string OSCARSSR_Cuda_GetDeviceProperties (int const i)
{
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);

  char buf[300];

  if (i >= ngpu) {
    sprintf(buf, "ERROR: GPU %i Not available", i);
    return std::string(buf);
  }

  std::string ret = "";

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, i);

  sprintf(buf, "  Device name: %s\n", prop.name);
  ret += std::string(buf);
  sprintf(buf, "  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
  ret += std::string(buf);
  sprintf(buf, "  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
  ret += std::string(buf);
  sprintf(buf, "  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  ret += std::string(buf);

  return ret;
}





__device__ static __inline__ void Orthogonal(double *a, double *b)
{
  // Return a vector which is orthogonal vector a
  double xx = a[0] < 0.0 ? -a[0] : a[0];
  double yy = a[1] < 0.0 ? -a[1] : a[1];
  double zz = a[2] < 0.0 ? -a[2] : a[2];
  if (xx < yy) {
    if (xx < zz) {
      b[0] = 0;
      b[1] = a[2];
      b[2] = -a[1];
    } else {
      b[0] = a[1];
      b[1] = -a[0];
      b[2] = 0;
    }
  } else {
    if (yy < zz) {
      b[0] = -a[2];
      b[1] = 0;
      b[2] = a[0];
    } else {
      b[0] = a[1];
      b[1] = -a[0];
      b[2] = 0;
    }
  }
  return;
}



__host__ __device__ static __inline__ hipDoubleComplex cuCexp(hipDoubleComplex x)
{
  double factor = exp(x.x);
  return make_hipDoubleComplex(factor * cos(x.y), factor * sin(x.y));
}




__device__ static __inline__ void GetInterpolatingIMinIMax(double *fx, int* nx, double* x, int *imin, int* imax)
{
  // get the klo and khi for interpolation

  int klo=0;
  int khi = *nx - 1;
  int k;
  while (khi - klo > 1) {
    k = (khi + klo) >> 1;
    if (fx[k] > *x) {
      khi = k;
    } else {
      klo = k;
    }
  }

  *imin = klo;
  *imax = khi;

  return;
}








__device__ static __inline__ double Interpolate (double *fx, double* fy, double *fypp, double* x, int* klo, int* khi)
{
  // Return the Y-value according to spline


  // Distance between points, check that it isn't zero!
  double const h = fx[*khi] - fx[*klo];
  if (h == 0) {
    // UPDATE: supposed to throw CUDA
  }

  // Fractional distance to the points on either side
  double const a = (fx[*khi] - *x) / h;
  double const b = (*x - fx[*klo]) / h;

  // Return the value of Y
  return a * fy[*klo] + b * fy[*khi] + ((a * a * a - a) * fypp[*klo] + (b * b * b - b) * fypp[*khi]) * (h * h) / 6.;
}











__device__ static __inline__ double Interpolate (double *fx, int* nx, double* fy, double *fypp, double* x)
{
  // Return the Y-value according to spline

  int klo=0;
  int khi = *nx - 1;
  int k;
  while (khi - klo > 1) {
    k = (khi + klo) >> 1;
    if (fx[k] > *x) {
      khi = k;
    } else {
      klo = k;
    }
  }

  // Distance between points, check that it isn't zero!
  double const h = fx[khi] - fx[klo];
  if (h == 0) {
    // UPDATE: supposed to throw CUDA
  }

  // Fractional distance to the points on either side
  double const a = (fx[khi] - *x) / h;
  double const b = (*x - fx[klo]) / h;

  // Return the value of Y
  return a * fy[klo] + b * fy[khi] + ((a * a * a - a) * fypp[klo] + (b * b * b - b) * fypp[khi]) * (h * h) / 6.;
}





























__global__ void OSCARSSR_Cuda_FluxGPU (double *t,
                                       double  *x,   double *y,   double *z,
                                       double  *xp,  double *yp,  double *zp,
                                       double  *bx,  double *by,  double *bz,
                                       double  *bxp, double *byp, double *bzp,
                                       double  *ax,  double *ay,  double *az,
                                       double  *axp, double *ayp, double *azp,
                                       double  *sx,  double *sy,  double *sz,
                                       hipDoubleComplex *pol,                             // photon polarization vector [3]
                                       double  *tstart, double *tstop,
                                       int *nt,
                                       int *ns,
                                       double *C0,   double *C2,  double *C,
                                       double *Omega,
                                       int *ifirst,
                                       int *ml,
                                       double *prec,
                                       int    *rt,
                                       double *result)
{
  // Thread number and surface number
  int const ith = threadIdx.x + blockIdx.x * blockDim.x;
  int const is = ith + *ifirst;

  // Alternative returns
  double result_precision = -1;
  int    result_level = -1;

  // Observation point
  double const ox = is >= *ns ? 0 : sx[is];
  double const oy = is >= *ns ? 0 : sy[is];
  double const oz = is >= *ns ? 0 : sz[is];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);


  bool use_pol = hipCabs(pol[0]) > 0.1 || hipCabs(pol[1]) > 0.1 || hipCabs(pol[2]) > 0.1 ? true : false;

  // Trajectory interpolated on fly and stored in shared memory for this block
  __shared__ double _t[NTHREADS_PER_BLOCK];
  __shared__ double _x[NTHREADS_PER_BLOCK];
  __shared__ double _y[NTHREADS_PER_BLOCK];
  __shared__ double _z[NTHREADS_PER_BLOCK];
  __shared__ double _bx[NTHREADS_PER_BLOCK];
  __shared__ double _by[NTHREADS_PER_BLOCK];
  __shared__ double _bz[NTHREADS_PER_BLOCK];
  __shared__ double _ax[NTHREADS_PER_BLOCK];
  __shared__ double _ay[NTHREADS_PER_BLOCK];
  __shared__ double _az[NTHREADS_PER_BLOCK];

  // Are all threads in this block done?
  __shared__ bool _all_done;

  // Is this thread a valid surface point?
  bool const in_surface = ((is < *ns) ? true : false);

  // I will consider you "done" if you are not a valid surface
  bool done = !in_surface;

  // Start off with the assumption that not all threads are done
  _all_done = false;


  // Number of trajectory points in current level
  int this_nt = 1;


  // Result up to this level and from last level for comparison
  double this_result = 0;
  double last_result = 1;

  // DeltaT for all levels up to this level
  double dt_total = 0;

  // Loop over all levels 
  for (int ilevel = 0; (ilevel <= *ml) && !_all_done; ++ilevel) {

    // DeltaT inclusive up to this level
    dt_total = (*tstop - *tstart) / pow(2., ilevel+1);//(*tstop - *tstart) / (2 * this_nt);

    // deltaT this level and Time start this level
    double const dt = (*tstop - *tstart) / pow(2., ilevel);//(*tstop - *tstart) / this_nt;
    double const ts = *tstart + (*tstop - *tstart) / pow(2., ilevel + 1);//*tstart + (*tstop - *tstart) / (2. * this_nt);

    // Number of trajectory "blocks" we will have to loop over for this ilevel
    int const NTrajectoryBlocks = this_nt / blockDim.x + (this_nt % blockDim.x == 0 ? 0 : 1);

    // Loop over all trajectory blocks
    for (int itb = 0; itb < NTrajectoryBlocks; ++itb) {

      // You better sync threads before you attempt to write to shared memory
      __syncthreads();

      // Calculate time of this thread in this trajectory block
      _t[threadIdx.x] = dt * (itb * blockDim.x + threadIdx.x) + ts;

      // If the time is outside we will not consider it
      if (_t[threadIdx.x] < *tstop) {

        // First get interpolating imin and imax for this time
        int imin, imax;
        GetInterpolatingIMinIMax(t, nt, &(_t[threadIdx.x]), &imin, &imax);

        // Fill trajectory with interpolated data
        _x[threadIdx.x]  = Interpolate(t,  x,  xp, &(_t[threadIdx.x]), &imin, &imax);
        _y[threadIdx.x]  = Interpolate(t,  y,  yp, &(_t[threadIdx.x]), &imin, &imax);
        _z[threadIdx.x]  = Interpolate(t,  z,  zp, &(_t[threadIdx.x]), &imin, &imax);
        _bx[threadIdx.x] = Interpolate(t, bx, bxp, &(_t[threadIdx.x]), &imin, &imax);
        _by[threadIdx.x] = Interpolate(t, by, byp, &(_t[threadIdx.x]), &imin, &imax);
        _bz[threadIdx.x] = Interpolate(t, bz, bzp, &(_t[threadIdx.x]), &imin, &imax);
        _ax[threadIdx.x] = Interpolate(t, ax, axp, &(_t[threadIdx.x]), &imin, &imax);
        _ay[threadIdx.x] = Interpolate(t, ay, ayp, &(_t[threadIdx.x]), &imin, &imax);
        _az[threadIdx.x] = Interpolate(t, az, azp, &(_t[threadIdx.x]), &imin, &imax);
      }

      // Must sync threads to make sure all trajectory data for this trajectory loop is written
      __syncthreads();

      // If this thread is not done then do some more calculations
      if (!done) {
        for (int i = 0; i < blockDim.x; ++i) {

          // Check if we are over the limit of trajectory points
          if (is < *ns && (_t[i] < *tstop)) {

            // Distance to observer
            double const D = sqrt( pow( (ox) - _x[i], 2) + pow( (oy) - _y[i], 2) + pow((oz) - _z[i], 2) );

            // Normal in direction of observer
            double const NX = ((ox) - _x[i]) / D;
            double const NY = ((oy) - _y[i]) / D;
            double const NZ = ((oz) - _z[i]) / D;

            // Magnitude of Beta squared
            double const One_Minus_BMag2 = 1. -  (_bx[i] * _bx[i] + _by[i] * _by[i] + _bz[i] * _bz[i]);

            // N dot Beta
            double const NDotBeta = NX * _bx[i] + NY * _by[i] + NZ * _bz[i];

            // Field contibutions
            double const FarFieldDenominator =  D * (pow(1. - NDotBeta, 2));
            double const NearFieldDenominator = D * FarFieldDenominator;
            double const NearField_X = One_Minus_BMag2 * (NX - _bx[i]) / NearFieldDenominator;
            double const NearField_Y = One_Minus_BMag2 * (NY - _by[i]) / NearFieldDenominator;
            double const NearField_Z = One_Minus_BMag2 * (NZ - _bz[i]) / NearFieldDenominator;

            double const FFX = (NY - _by[i]) * _az[i] - (NZ - _bz[i]) * _ay[i];
            double const FFY = (NZ - _bz[i]) * _ax[i] - (NX - _bx[i]) * _az[i];
            double const FFZ = (NX - _bx[i]) * _ay[i] - (NY - _by[i]) * _ax[i];

            double const FarField_X = (NY * FFZ - NZ * FFY) / FarFieldDenominator;
            double const FarField_Y = (NZ * FFX - NX * FFZ) / FarFieldDenominator;
            double const FarField_Z = (NX * FFY - NY * FFX) / FarFieldDenominator;


            // Exponent for fourier transformed field
            hipDoubleComplex Exponent = make_hipDoubleComplex(0, -(*Omega) * (_t[i] + D / (*C)));

            hipDoubleComplex X1 = make_hipDoubleComplex(NearField_X + FarField_X, 0);
            hipDoubleComplex Y1 = make_hipDoubleComplex(NearField_Y + FarField_Y, 0);
            hipDoubleComplex Z1 = make_hipDoubleComplex(NearField_Z + FarField_Z, 0);

            hipDoubleComplex MyEXP = cuCexp(Exponent);

            hipDoubleComplex X2 = hipCmul(X1, MyEXP);
            hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
            hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);

            // Add to current sums
            SumEX = hipCadd(SumEX, X2);
            SumEY = hipCadd(SumEY, Y2);
            SumEZ = hipCadd(SumEZ, Z2);

          }
        }
      }
    }

    // If this thread is a valid surface point and it is not "done" yet, calculate the result
    if (in_surface && !done) {
      hipDoubleComplex TSumEX = hipCmul(make_hipDoubleComplex((*C0) * (dt_total), 0), SumEX);
      hipDoubleComplex TSumEY = hipCmul(make_hipDoubleComplex((*C0) * (dt_total), 0), SumEY);
      hipDoubleComplex TSumEZ = hipCmul(make_hipDoubleComplex((*C0) * (dt_total), 0), SumEZ);

      if (use_pol) {
        hipDoubleComplex DotProduct = hipCadd(hipCadd(hipCmul(hipConj(TSumEX), pol[0]), hipCmul(hipConj(TSumEY), pol[1])), hipCmul(hipConj(TSumEZ), pol[2]));

        TSumEX = hipCmul(pol[0], DotProduct);
        TSumEY = hipCmul(pol[1], DotProduct);
        TSumEZ = hipCmul(pol[2], DotProduct);
      }

      double const EX = (TSumEX.x * TSumEX.x + TSumEX.y * TSumEX.y);
      double const EY = (TSumEY.x * TSumEY.x + TSumEY.y * TSumEY.y);
      double const EZ = (TSumEZ.x * TSumEZ.x + TSumEZ.y * TSumEZ.y);

      // Result up to this point
      this_result = fabs((*C2) * (EX + EY + EZ));

      result_precision = fabs((last_result - this_result) / last_result);

      // If below desired precision set as done
      if ( ilevel > 8 && result_precision < *prec ) {
        done = true;
        result_level = ilevel;
      }

      // Keep track of last result for precision test
      last_result = this_result;
    }

    // Check if any thread is not done
    _all_done = true;
    __syncthreads();
    if (!done) {
      _all_done = false;
    }
    __syncthreads();

    // Next level has *2 trajectory points
    this_nt *= 2;
  }

  // We're done, but if you're not a surface point just return
  if (!in_surface) {
    return;
  }

  // Set result and return
  switch (*rt) {
    case 1:
      result[ith] = result_precision;
      break;
    case 2:
      result[ith] = (double) result_level;
      break;
    default:
      result[ith] = this_result;
      break;
  }

  return;
}

















extern "C" void OSCARSSR_Cuda_CalculateFluxGPU (OSCARSSR& OSR,
                                                TSurfacePoints const& Surface,
                                                double const Energy_eV,
                                                T3DScalarContainer& FluxContainer,
                                                std::string const& Polarization,
                                                double const Angle,
                                                TVector3D const& HorizontalDirection,
                                                TVector3D const& PropogationDirection,
                                                int const NParticles,
                                                std::vector<int> const& GPUVector,
                                                double const Precision,
                                                int const MaxLevel,
                                                int    const ReturnQuantity)
{
  // Calculate the flux for NParticles using the GPUs given in GPUVector.  Each particle's
  // trajectory will be sent to all GPUs for processing, meanwhile a new trajectory will
  // be calculated

  // Number of available GPUs
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Make sure that a gpu listed is within the range and not a duplicate
  std::vector<int> GPUsToUse;
  for (std::vector<int>::const_iterator it = GPUVector.begin(); it != GPUVector.end(); ++it) {
    if ( !(std::find(GPUsToUse.begin(), GPUsToUse.end(), *it) != GPUsToUse.end() && (*it < ngpu)) ) {
      GPUsToUse.push_back(*it);
    }
  }

  // Make sure we have at least one
  if (GPUsToUse.size() == 0) {
    throw std::invalid_argument("GPUs selected do not match hardware");
  }
  int const NGPUsToUse = (int) GPUsToUse.size();

  // Do we calculate for the current particle?
  bool const ThisParticleOnly = NParticles == 0 ? true : false;
  int  const NParticlesReally = ThisParticleOnly ? 1 : NParticles;

  // Calculate polarization vector to use (0, 0, 0) for 'all'
  TVector3D  const VerticalDirection = PropogationDirection.Cross(HorizontalDirection).UnitVector();
  TVector3DC const Positive = 1. / sqrt(2) * (TVector3DC(HorizontalDirection) + VerticalDirection * std::complex<double>(0, 1) );
  TVector3DC const Negative = 1. / sqrt(2) * (TVector3DC(HorizontalDirection) - VerticalDirection * std::complex<double>(0, 1) );
  TVector3DC PhotonPolarizationVector(0, 0, 0);
  if (Polarization == "all") {
    // Do nothing, it is already ALL
  } else if (Polarization == "linear-horizontal") {
    PhotonPolarizationVector = HorizontalDirection;
  } else if (Polarization == "linear-vertical") {
    PhotonPolarizationVector = VerticalDirection;
  } else if (Polarization == "linear") {
    TVector3D PolarizationAngle = HorizontalDirection;
    PolarizationAngle.RotateSelf(Angle, PropogationDirection);
    PhotonPolarizationVector = PolarizationAngle;
  } else if (Polarization == "circular-left") {
    PhotonPolarizationVector = Positive;
  } else if (Polarization == "circular-right") {
    PhotonPolarizationVector = Negative;
  } else {
    // Throw invalid argument if polarization is not recognized
    throw std::invalid_argument("Polarization requested not recognized");
  }


  int *h_ns;
  int *h_nt;
  hipDoubleComplex *h_pol;
  hipHostAlloc((void**) &h_ns,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nt,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_pol, 3 * sizeof(hipDoubleComplex), hipHostMallocWriteCombined | hipHostMallocMapped);

  // First one, set particle and trajectory
  if (!ThisParticleOnly) {
    OSR.SetNewParticle();
  }
  if (OSR.GetTrajectory().GetNPoints() == 0) {
    OSR.CalculateTrajectory();
  }

  // Needed number of points in the track and time step
  *h_nt     = (int) OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetNPoints();
  *h_ns     = (int) Surface.GetNPoints();

  // Photon polarization
  h_pol[0] = make_hipDoubleComplex(PhotonPolarizationVector.GetX().real(), PhotonPolarizationVector.GetX().imag());
  h_pol[1] = make_hipDoubleComplex(PhotonPolarizationVector.GetY().real(), PhotonPolarizationVector.GetY().imag());
  h_pol[2] = make_hipDoubleComplex(PhotonPolarizationVector.GetZ().real(), PhotonPolarizationVector.GetZ().imag());

  int const NThreads = *h_ns;
  int const NThreadsPerBlock = NTHREADS_PER_BLOCK;
  int const NThreadsRemainder = NThreads % NThreadsPerBlock;
  int const NBlocksTotal = (NThreads - 1) / NThreadsPerBlock + 1;
  int const NBlocksPerGPU = NBlocksTotal / NGPUsToUse;
  int const NRemainderBlocks = NBlocksTotal % NGPUsToUse;
  // UPDATE: To be modified
  int const NTT = NThreadsPerBlock * (NBlocksPerGPU + (NRemainderBlocks > 0 ? 1 : 0));

  std::vector<int> NBlocksThisGPU(NGPUsToUse, NBlocksPerGPU);
  for (int i = 0; i < NRemainderBlocks; ++i) {
    ++NBlocksThisGPU[i];
  }

  // Interpolating structure
  double  *h_t;
  double  *h_tstart, *h_tstop;
  double  *h_x,   *h_y,   *h_z;
  double  *h_xp,  *h_yp,  *h_zp;
  double  *h_bx,  *h_by,  *h_bz;
  double  *h_bxp, *h_byp, *h_bzp;
  double  *h_ax,  *h_ay,  *h_az;
  double  *h_axp, *h_ayp, *h_azp;

  // Surface points
  double  *h_sx,  *h_sy,  *h_sz;

  // Constants and photon frequency
  double *h_c0,  *h_c2,  *h_c,  *h_omega;

  // first point for each thread, max level
  int     *h_ifirst;
  int     *h_ml;

  // Precision desired
  double *h_prec;

  // Return quantity flag
  int    *h_rt;

  // Results
  double **h_result;

  // Allocate host memory
  hipHostAlloc((void**) &h_t,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_tstart,          sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_tstop,           sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_x,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_y,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_z,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_xp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_yp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_zp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bx,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_by,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bz,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bxp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_byp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bzp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ax,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ay,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_az,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_axp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ayp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_azp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_sx,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sy,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sz,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_c0,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c2,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c,                   sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_omega,               sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ifirst, NGPUsToUse * sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ml,                  sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_prec,                sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_rt,                  sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_result,   NGPUsToUse * sizeof(double*), hipHostMallocWriteCombined | hipHostMallocMapped);
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    hipHostAlloc((void**) &(h_result[i]), NTT * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  }

  // First surface point for each gpu
  int NBlocksUsed = 0;
  for (int i = 0; i < NGPUsToUse; ++i) {
    h_ifirst[i] = NBlocksUsed * NThreadsPerBlock;
    NBlocksUsed += NBlocksThisGPU[i];
  }

  *h_ml = MaxLevel; //UPDATE: max level should be an input

  // Precision
  *h_prec = Precision;

  // Return quantity
  *h_rt = ReturnQuantity;

  // Memor allocations for GPU
  int    **d_nt;
  double **d_tstart;
  double **d_tstop;
  int    **d_ns;

  hipDoubleComplex **d_pol;

  double **d_t;

  double **d_x;
  double **d_y;
  double **d_z;
  double **d_xp;
  double **d_yp;
  double **d_zp;

  double **d_bx;
  double **d_by;
  double **d_bz;
  double **d_bxp;
  double **d_byp;
  double **d_bzp;

  double **d_ax;
  double **d_ay;
  double **d_az;
  double **d_axp;
  double **d_ayp;
  double **d_azp;

  double **d_sx;
  double **d_sy;
  double **d_sz;

  double **d_c0;
  double **d_c2;
  double **d_c;
  double **d_omega;

  int    **d_ifirst;
  int    **d_ml;
  double **d_prec;
  int    **d_rt;
  double **d_result;

  hipHostAlloc((void **) &d_nt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_tstart, NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_tstop,  NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ns,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_pol,    NGPUsToUse * sizeof(hipDoubleComplex*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_t,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_x,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_y,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_z,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_xp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_yp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_zp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_bx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_by,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bxp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_byp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bzp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_ax,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ay,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_az,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_axp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ayp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_azp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_sx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sy,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_c0,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c2,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_omega,  NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_ifirst, NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ml,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_prec,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_rt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_result,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipMalloc((void **) &d_nt[i],                 sizeof(int));
    hipMalloc((void **) &d_tstart[i],             sizeof(double));
    hipMalloc((void **) &d_tstop[i],              sizeof(double));
    hipMalloc((void **) &d_ns[i],                 sizeof(int));

    hipMalloc((void **) &d_pol[i],            3 * sizeof(hipDoubleComplex));

    hipMalloc((void **) &d_t[i],          *h_nt * sizeof(double));

    hipMalloc((void **) &d_x[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_y[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_z[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_xp[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_yp[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_zp[i],         *h_nt * sizeof(double));

    hipMalloc((void **) &d_bx[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_by[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_bz[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_bxp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_byp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_bzp[i],        *h_nt * sizeof(double));

    hipMalloc((void **) &d_ax[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_ay[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_az[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_axp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_ayp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_azp[i],        *h_nt * sizeof(double));

    hipMalloc((void **) &d_sx[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sy[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sz[i],         *h_ns * sizeof(double));

    hipMalloc((void **) &d_c0[i],                 sizeof(double));
    hipMalloc((void **) &d_c2[i],                 sizeof(double));
    hipMalloc((void **) &d_c[i],                  sizeof(double));
    hipMalloc((void **) &d_omega[i],              sizeof(double));

    hipMalloc((void **) &d_ifirst[i],             sizeof(int));
    hipMalloc((void **) &d_ml[i],                 sizeof(int));
    hipMalloc((void **) &d_prec[i],               sizeof(double));
    hipMalloc((void **) &d_rt[i],                 sizeof(int));
    hipMalloc((void **) &d_result[i],       NTT * sizeof(double));

    // Copy device number to device
    hipMemcpyAsync(d_ifirst[i], &(h_ifirst[i]), sizeof(int), hipMemcpyHostToDevice);
  }

  // Compute known host values
  *h_c0    = OSR.GetCurrentParticle().GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());
  *h_c2    = TOSCARSSR::FourPi() * OSR.GetCurrentParticle().GetCurrent() / (TOSCARSSR::H() * fabs(OSR.GetCurrentParticle().GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;
  *h_c     = TOSCARSSR::C();
  *h_omega = TOSCARSSR::EvToAngularFrequency(Energy_eV);
  for (size_t i = 0; i < *h_ns; ++i) {
    h_sx[i] = Surface.GetPoint(i).GetX();
    h_sy[i] = Surface.GetPoint(i).GetY();
    h_sz[i] = Surface.GetPoint(i).GetZ();
  }

  // Copy constants to first device (async)
  int const d0 = GPUsToUse[0];
  hipSetDevice(d0);
  hipMemcpyAsync(d_ns[0],    h_ns,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_pol[0],   h_pol,     3 * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c0[0],    h_c0,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c2[0],    h_c2,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c[0],     h_c,           sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_omega[0], h_omega,       sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sx[0],    h_sx,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sy[0],    h_sy,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sz[0],    h_sz,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_ml[0],    h_ml,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_prec[0],  h_prec,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_rt[0],    h_rt,          sizeof(int),    hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_ns[i+1],     d1, d_ns[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_pol[i+1],    d1, d_pol[i],    d, 3 * sizeof(hipDoubleComplex));
    hipMemcpyPeerAsync( d_c0[i+1],     d1, d_c0[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c2[i+1],     d1, d_c2[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c[i+1],      d1, d_c[i],      d, sizeof(double));
    hipMemcpyPeerAsync( d_omega[i+1],  d1, d_omega[i],  d, sizeof(double));
    hipMemcpyPeerAsync( d_sx[i+1],     d1, d_sx[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sy[i+1],     d1, d_sy[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sz[i+1],     d1, d_sz[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_ml[i+1],     d1, d_ml[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_prec[i+1],   d1, d_prec[i],   d, sizeof(double));
    hipMemcpyPeerAsync( d_rt[i+1],     d1, d_rt[i],     d, sizeof(int));
  }

  // Set first trajectory
  TOMATH::TSpline1D3<TParticleTrajectoryPoint> const& T = OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetSpline();
  int const NPointsThisTrajectory = T.GetNPoints();
  *h_nt = NPointsThisTrajectory;
  *h_tstart = T.GetXStart();
  *h_tstop  = T.GetXStop();
  for (size_t i = 0; i < NPointsThisTrajectory; ++i) {
    TParticleTrajectoryPoint const& P  = T.GetY(i);
    TParticleTrajectoryPoint const& PP = T.GetYPP(i);

    h_t[i]   = T.GetX(i);

    h_x[i]   =  P.GetX().GetX();
    h_y[i]   =  P.GetX().GetY();
    h_z[i]   =  P.GetX().GetZ();
    h_xp[i]  = PP.GetX().GetX();
    h_yp[i]  = PP.GetX().GetY();
    h_zp[i]  = PP.GetX().GetZ();

    h_bx[i]  =  P.GetB().GetX();
    h_by[i]  =  P.GetB().GetY();
    h_bz[i]  =  P.GetB().GetZ();
    h_bxp[i] = PP.GetB().GetX();
    h_byp[i] = PP.GetB().GetY();
    h_bzp[i] = PP.GetB().GetZ();

    h_ax[i]  =  P.GetAoverC().GetX();
    h_ay[i]  =  P.GetAoverC().GetY();
    h_az[i]  =  P.GetAoverC().GetZ();
    h_axp[i] = PP.GetAoverC().GetX();
    h_ayp[i] = PP.GetAoverC().GetY();
    h_azp[i] = PP.GetAoverC().GetZ();
  }


  // Set the surface points
  // GPU events
  hipEvent_t *event_resultcopy = new hipEvent_t[NGPUsToUse];
  for (int ig = 0; ig < NGPUsToUse; ++ig) {
    int const d = GPUsToUse[ig];
    hipSetDevice(d);
    hipEventCreate(&(event_resultcopy[ig]));
  }

  // Enable peer (direct gpu-gpu) writes
  for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
    // Device number
    int const d  = GPUsToUse[ig];
    int const d1 = GPUsToUse[ig+1];
    int access;
    hipDeviceCanAccessPeer(&access, d, d1);
    if (access == 1) {
      hipSetDevice(d);
      hipDeviceEnablePeerAccess(d1, 0);
    }
  }

  // Loop over number of particles
  for (int ip = 0; ip < NParticlesReally; ++ip) {

    // Copy trajectory to first GPU, then internal async transfers (where possible)
    hipSetDevice(d0);
    hipMemcpyAsync(d_nt[0],  h_nt,          sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpyAsync(d_t[0],   h_t,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_tstart[0], h_tstart,   sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_tstop[0],  h_tstop,    sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_x[0],   h_x,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_y[0],   h_y,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_z[0],   h_z,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_xp[0],  h_xp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_yp[0],  h_yp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_zp[0],  h_zp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bx[0],  h_bx,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_by[0],  h_by,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bz[0],  h_bz,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bxp[0], h_bxp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_byp[0], h_byp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bzp[0], h_bzp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ax[0],  h_ax,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ay[0],  h_ay,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_az[0],  h_az,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_axp[0], h_axp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ayp[0], h_ayp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_azp[0], h_azp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
      // Device number
      int const d  = GPUsToUse[ig];
      int const d1 = GPUsToUse[ig+1];
      hipSetDevice(d);
      hipMemcpyPeerAsync(d_nt[ig+1],  d1, d_nt[ig],  d,         sizeof(int));
      hipMemcpyPeerAsync(d_t[ig+1],   d1, d_t[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_tstart[ig+1], d1, d_tstart[ig], d,   sizeof(double));
      hipMemcpyPeerAsync(d_tstop[ig+1],  d1, d_tstop[ig],  d,    sizeof(double));
      hipMemcpyPeerAsync(d_x[ig+1],   d1, d_x[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_y[ig+1],   d1, d_y[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_z[ig+1],   d1, d_z[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_xp[ig+1],  d1, d_xp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_yp[ig+1],  d1, d_yp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_zp[ig+1],  d1, d_zp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bx[ig+1],  d1, d_bx[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_by[ig+1],  d1, d_by[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bz[ig+1],  d1, d_bz[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bxp[ig+1], d1, d_bxp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_byp[ig+1], d1, d_byp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bzp[ig+1], d1, d_bzp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ax[ig+1],  d1, d_ax[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ay[ig+1],  d1, d_ay[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_az[ig+1],  d1, d_az[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_axp[ig+1], d1, d_axp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ayp[ig+1], d1, d_ayp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_azp[ig+1], d1, d_azp[ig], d, *h_nt * sizeof(double));

    }

    // Wait for previous copy, start next one
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d = GPUsToUse[ig];
      hipSetDevice(d);
      hipEventSynchronize(event_resultcopy[ig]);
      OSCARSSR_Cuda_FluxGPU<<<NBlocksThisGPU[ig], NThreadsPerBlock>>>(
                                                                      d_t[ig],
                                                                      d_x[ig],   d_y[ig],   d_z[ig],
                                                                      d_xp[ig],  d_yp[ig],  d_zp[ig],
                                                                      d_bx[ig],  d_by[ig],  d_bz[ig],
                                                                      d_bxp[ig], d_byp[ig], d_bzp[ig],
                                                                      d_ax[ig],  d_ay[ig],  d_az[ig],
                                                                      d_axp[ig], d_ayp[ig], d_azp[ig],
                                                                      d_sx[ig],  d_sy[ig],  d_sz[ig],
                                                                      d_pol[ig],
                                                                      d_tstart[ig], d_tstop[ig],
                                                                      d_nt[ig],
                                                                      d_ns[ig],
                                                                      d_c0[ig], d_c2[ig], d_c[ig],
                                                                      d_omega[ig],
                                                                      d_ifirst[ig],
                                                                      d_ml[ig],
                                                                      d_prec[ig],
                                                                      d_rt[ig],
                                                                      d_result[ig]);
    }


    // Add result to results container (from **previous**)
    if (ip > 0) {
      int NBlocksUsed = 0;
      for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
        for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
          if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
            break;
          }
          int iss = ith + NThreadsPerBlock * NBlocksUsed;
          FluxContainer.AddToPoint(iss, h_result[ig][ith]);
        }
        NBlocksUsed += NBlocksThisGPU[ig];
      }
    }

    // Add copy back to streams
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d  = GPUsToUse[ig];
      hipSetDevice(d);
      hipMemcpyAsync(h_result[ig],  d_result[ig],  NTT * sizeof(double), hipMemcpyDeviceToHost);
      hipEventRecord(event_resultcopy[ig]);
    }

    // If it's not the last one, calculate a new trajectory
    if (ip < NParticlesReally - 1) {
      OSR.SetNewParticle();
      OSR.CalculateTrajectory();

      TOMATH::TSpline1D3<TParticleTrajectoryPoint> const& T = OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetSpline();
      int const NPointsThisTrajectory = T.GetNPoints();
      *h_nt = NPointsThisTrajectory;
      for (size_t it = 0; it < NPointsThisTrajectory; ++it) {
        TParticleTrajectoryPoint const& P  = T.GetY(it);
        TParticleTrajectoryPoint const& PP = T.GetYPP(it);

        h_t[it]   = T.GetX(it);

        h_x[it]   =  P.GetX().GetX();
        h_y[it]   =  P.GetX().GetY();
        h_z[it]   =  P.GetX().GetZ();
        h_xp[it]  = PP.GetX().GetX();
        h_yp[it]  = PP.GetX().GetY();
        h_zp[it]  = PP.GetX().GetZ();

        h_bx[it]  =  P.GetB().GetX();
        h_by[it]  =  P.GetB().GetY();
        h_bz[it]  =  P.GetB().GetZ();
        h_bxp[it] = PP.GetB().GetX();
        h_byp[it] = PP.GetB().GetY();
        h_bzp[it] = PP.GetB().GetZ();

        h_ax[it]  =  P.GetAoverC().GetX();
        h_ay[it]  =  P.GetAoverC().GetY();
        h_az[it]  =  P.GetAoverC().GetZ();
        h_axp[it] = PP.GetAoverC().GetX();
        h_ayp[it] = PP.GetAoverC().GetY();
        h_azp[it] = PP.GetAoverC().GetZ();
      }
    }
  }

  // Wait for last copy
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    hipEventSynchronize(event_resultcopy[ig]);
  }

  // Add result to results container (from **previous**)
  NBlocksUsed = 0;
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
      if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
        break;
      }
      int iss = ith + NThreadsPerBlock * NBlocksUsed;
      FluxContainer.AddToPoint(iss, h_result[ig][ith]);
    }
    NBlocksUsed += NBlocksThisGPU[ig];
  }

  // Weighting for multi-particle
  double const Weight = 1.0 / (double) NParticlesReally;
  FluxContainer.WeightAll(Weight);

  // Free host memory
  hipHostFree(h_nt);
  hipHostFree(h_t);
  hipHostFree(h_ns);
  hipHostFree(h_pol);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_xp);
  hipHostFree(h_yp);
  hipHostFree(h_zp);
  hipHostFree(h_bx);
  hipHostFree(h_by);
  hipHostFree(h_bz);
  hipHostFree(h_bxp);
  hipHostFree(h_byp);
  hipHostFree(h_bzp);
  hipHostFree(h_ax);
  hipHostFree(h_ay);
  hipHostFree(h_az);
  hipHostFree(h_axp);
  hipHostFree(h_ayp);
  hipHostFree(h_azp);
  hipHostFree(h_sx);
  hipHostFree(h_sy);
  hipHostFree(h_sz);
  hipHostFree(h_c0);
  hipHostFree(h_c2);
  hipHostFree(h_c);
  hipHostFree(h_omega);
  hipHostFree(h_ifirst);
  hipHostFree(h_ml);
  hipHostFree(h_prec);
  hipHostFree(h_rt);
  // Free host and GPU memory
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {

    hipHostFree(h_result[i]);

    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipFree(d_nt[i]);
    hipFree(d_t[i]);
    hipFree(d_ns[i]);
    hipFree(d_pol[i]);
    hipFree(d_x[i]);
    hipFree(d_y[i]);
    hipFree(d_z[i]);
    hipFree(d_xp[i]);
    hipFree(d_yp[i]);
    hipFree(d_zp[i]);
    hipFree(d_bx[i]);
    hipFree(d_by[i]);
    hipFree(d_bz[i]);
    hipFree(d_bxp[i]);
    hipFree(d_byp[i]);
    hipFree(d_bzp[i]);
    hipFree(d_ax[i]);
    hipFree(d_ay[i]);
    hipFree(d_az[i]);
    hipFree(d_axp[i]);
    hipFree(d_ayp[i]);
    hipFree(d_azp[i]);
    hipFree(d_sx[i]);
    hipFree(d_sy[i]);
    hipFree(d_sz[i]);
    hipFree(d_c0[i]);
    hipFree(d_c2[i]);
    hipFree(d_c[i]);
    hipFree(d_omega[i]);
    hipFree(d_ifirst[i]);
    hipFree(d_ml[i]);
    hipFree(d_prec[i]);
    hipFree(d_rt[i]);
    hipFree(d_result[i]);
  }
  hipFree(h_result);

  hipFree(d_nt);
  hipFree(d_t);
  hipFree(d_ns);
  hipFree(d_pol);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_xp);
  hipFree(d_yp);
  hipFree(d_zp);
  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);
  hipFree(d_bxp);
  hipFree(d_byp);
  hipFree(d_bzp);
  hipFree(d_ax);
  hipFree(d_ay);
  hipFree(d_az);
  hipFree(d_axp);
  hipFree(d_ayp);
  hipFree(d_azp);
  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);
  hipFree(d_c0);
  hipFree(d_c2);
  hipFree(d_c);
  hipFree(d_omega);
  hipFree(h_ifirst);
  hipFree(d_ml);
  hipFree(d_prec);
  hipFree(d_rt);
  hipFree(d_result);

  // Delete host gpu pointer arrays
  delete [] event_resultcopy;


  return;
}






















































__global__ void OSCARSSR_Cuda_SpectrumGPU (double          *t,                               // trj time
                                           double          *x,   double *y,   double *z,     // trj position
                                           double          *xp,  double *yp,  double *zp,    // trj pos deriv
                                           double          *bx,  double *by,  double *bz,    // trj beta
                                           double          *bxp, double *byp, double *bzp,   // trj beta deriv
                                           double          *ax,  double *ay,  double *az,    // trj a/c
                                           double          *axp, double *ayp, double *azp,   // trj a/c deriv
                                           double          *obs,                             // observation point [3]
                                           hipDoubleComplex *pol,                             // photon polarization vector [3]
                                           double          *tstart,           double *tstop, // start and stop time of trj
                                           int             *nt,                              // number of trj points
                                           double          *om,                              // omega (energy) array
                                           int             *no,                              // number of om elements
                                           double          *C0,  double *C2,  double *C,     // constants
                                           int             *ifirst,                          // first in om array for this block
                                           int             *ml,                              // max_level
                                           double          *prec,                            // precision requested
                                           int             *rt,                              // return quantity requested
                                           double          *result)                          // result array
{
  // Thread number and spectrum omega number
  int const ith = threadIdx.x + blockIdx.x * blockDim.x;
  int const io = ith + *ifirst;

  // Alternative returns
  double result_precision = -1;
  int    result_level = -1;

  // Energy (previously converted to omega)
  double const omega = io >= *no ? 0 : om[io];

  // Observation point
  double const ox = obs[0];
  double const oy = obs[1];
  double const oz = obs[2];

  // E-field components sum
  hipDoubleComplex SumEX = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEY = make_hipDoubleComplex(0, 0);
  hipDoubleComplex SumEZ = make_hipDoubleComplex(0, 0);


  bool use_pol = hipCabs(pol[0]) > 0.1 || hipCabs(pol[1]) > 0.1 || hipCabs(pol[2]) > 0.1 ? true : false;

  // Trajectory interpolated on fly and stored in shared memory for this block
  __shared__ double _t[NTHREADS_PER_BLOCK];
  __shared__ double _x[NTHREADS_PER_BLOCK];
  __shared__ double _y[NTHREADS_PER_BLOCK];
  __shared__ double _z[NTHREADS_PER_BLOCK];
  __shared__ double _bx[NTHREADS_PER_BLOCK];
  __shared__ double _by[NTHREADS_PER_BLOCK];
  __shared__ double _bz[NTHREADS_PER_BLOCK];
  __shared__ double _ax[NTHREADS_PER_BLOCK];
  __shared__ double _ay[NTHREADS_PER_BLOCK];
  __shared__ double _az[NTHREADS_PER_BLOCK];

  // Are all threads in this block done?
  __shared__ bool _all_done;

  // Is this thread a valid spectrum point?
  bool const in_spectrum = ((io < *no) ? true : false);

  // I will consider you "done" if you are not a valid spectrum point
  bool done = !in_spectrum;

  // Start off with the assumption that not all threads are done
  _all_done = false;


  // Number of trajectory points in current level
  int this_nt = 1;


  // Result up to this level and from last level for comparison
  double this_result = 0;
  double last_result = 1;

  // DeltaT for all levels up to this level
  double dt_total = 0;

  // Loop over all levels 
  for (int ilevel = 0; (ilevel <= *ml) && !_all_done; ++ilevel) {

    // DeltaT inclusive up to this level
    dt_total = (*tstop - *tstart) / pow(2., ilevel+1);//(*tstop - *tstart) / (2 * this_nt);

    // deltaT this level and Time start this level
    double const dt = (*tstop - *tstart) / pow(2., ilevel);//(*tstop - *tstart) / this_nt;
    double const ts = *tstart + (*tstop - *tstart) / pow(2., ilevel + 1);//*tstart + (*tstop - *tstart) / (2. * this_nt);

    // Number of trajectory "blocks" we will have to loop over for this ilevel
    int const NTrajectoryBlocks = this_nt / blockDim.x + (this_nt % blockDim.x == 0 ? 0 : 1);

    // Loop over all trajectory blocks
    for (int itb = 0; itb < NTrajectoryBlocks; ++itb) {

      // You better sync threads before you attempt to write to shared memory
      __syncthreads();

      // Calculate time of this thread in this trajectory block
      _t[threadIdx.x] = dt * (itb * blockDim.x + threadIdx.x) + ts;

      // If the time is outside we will not consider it
      if (_t[threadIdx.x] < *tstop) {

        // First get interpolating imin and imax for this time
        int imin, imax;
        GetInterpolatingIMinIMax(t, nt, &(_t[threadIdx.x]), &imin, &imax);

        // Fill trajectory with interpolated data
        _x[threadIdx.x]  = Interpolate(t,  x,  xp, &(_t[threadIdx.x]), &imin, &imax);
        _y[threadIdx.x]  = Interpolate(t,  y,  yp, &(_t[threadIdx.x]), &imin, &imax);
        _z[threadIdx.x]  = Interpolate(t,  z,  zp, &(_t[threadIdx.x]), &imin, &imax);
        _bx[threadIdx.x] = Interpolate(t, bx, bxp, &(_t[threadIdx.x]), &imin, &imax);
        _by[threadIdx.x] = Interpolate(t, by, byp, &(_t[threadIdx.x]), &imin, &imax);
        _bz[threadIdx.x] = Interpolate(t, bz, bzp, &(_t[threadIdx.x]), &imin, &imax);
        _ax[threadIdx.x] = Interpolate(t, ax, axp, &(_t[threadIdx.x]), &imin, &imax);
        _ay[threadIdx.x] = Interpolate(t, ay, ayp, &(_t[threadIdx.x]), &imin, &imax);
        _az[threadIdx.x] = Interpolate(t, az, azp, &(_t[threadIdx.x]), &imin, &imax);
      }

      // Must sync threads to make sure all trajectory data for this trajectory loop is written
      __syncthreads();

      // If this thread is not done then do some more calculations
      if (!done) {
        for (int i = 0; i < blockDim.x; ++i) {

          // Check if we are over the limit of trajectory points
          if (in_spectrum && (_t[i] < *tstop)) {

            // Distance to observer
            double const D = sqrt( pow( (ox) - _x[i], 2) + pow( (oy) - _y[i], 2) + pow((oz) - _z[i], 2) );

            // Normal in direction of observer
            double const NX = ((ox) - _x[i]) / D;
            double const NY = ((oy) - _y[i]) / D;
            double const NZ = ((oz) - _z[i]) / D;

            // Magnitude of Beta squared
            double const One_Minus_BMag2 = 1. -  (_bx[i] * _bx[i] + _by[i] * _by[i] + _bz[i] * _bz[i]);

            // N dot Beta
            double const NDotBeta = NX * _bx[i] + NY * _by[i] + NZ * _bz[i];

            // Field contibutions
            double const FarFieldDenominator =  D * (pow(1. - NDotBeta, 2));
            double const NearFieldDenominator = D * FarFieldDenominator;
            double const NearField_X = One_Minus_BMag2 * (NX - _bx[i]) / NearFieldDenominator;
            double const NearField_Y = One_Minus_BMag2 * (NY - _by[i]) / NearFieldDenominator;
            double const NearField_Z = One_Minus_BMag2 * (NZ - _bz[i]) / NearFieldDenominator;

            double const FFX = (NY - _by[i]) * _az[i] - (NZ - _bz[i]) * _ay[i];
            double const FFY = (NZ - _bz[i]) * _ax[i] - (NX - _bx[i]) * _az[i];
            double const FFZ = (NX - _bx[i]) * _ay[i] - (NY - _by[i]) * _ax[i];

            double const FarField_X = (NY * FFZ - NZ * FFY) / FarFieldDenominator;
            double const FarField_Y = (NZ * FFX - NX * FFZ) / FarFieldDenominator;
            double const FarField_Z = (NX * FFY - NY * FFX) / FarFieldDenominator;


            // Exponent for fourier transformed field
            hipDoubleComplex Exponent = make_hipDoubleComplex(0, -(omega) * (_t[i] + D / (*C)));

            hipDoubleComplex X1 = make_hipDoubleComplex(NearField_X + FarField_X, 0);
            hipDoubleComplex Y1 = make_hipDoubleComplex(NearField_Y + FarField_Y, 0);
            hipDoubleComplex Z1 = make_hipDoubleComplex(NearField_Z + FarField_Z, 0);

            hipDoubleComplex MyEXP = cuCexp(Exponent);

            hipDoubleComplex X2 = hipCmul(X1, MyEXP);
            hipDoubleComplex Y2 = hipCmul(Y1, MyEXP);
            hipDoubleComplex Z2 = hipCmul(Z1, MyEXP);

            // Add to current sums
            SumEX = hipCadd(SumEX, X2);
            SumEY = hipCadd(SumEY, Y2);
            SumEZ = hipCadd(SumEZ, Z2);

          }
        }
      }
    }

    // If this thread is a valid point and it is not "done" yet, calculate the result
    if (in_spectrum && !done) {
      hipDoubleComplex TSumEX = hipCmul(make_hipDoubleComplex((*C0) * (dt_total), 0), SumEX);
      hipDoubleComplex TSumEY = hipCmul(make_hipDoubleComplex((*C0) * (dt_total), 0), SumEY);
      hipDoubleComplex TSumEZ = hipCmul(make_hipDoubleComplex((*C0) * (dt_total), 0), SumEZ);

      if (use_pol) {
        hipDoubleComplex DotProduct = hipCadd(hipCadd(hipCmul(hipConj(TSumEX), pol[0]), hipCmul(hipConj(TSumEY), pol[1])), hipCmul(hipConj(TSumEZ), pol[2]));

        TSumEX = hipCmul(pol[0], DotProduct);
        TSumEY = hipCmul(pol[1], DotProduct);
        TSumEZ = hipCmul(pol[2], DotProduct);
      }

      double const EX = (TSumEX.x * TSumEX.x + TSumEX.y * TSumEX.y);
      double const EY = (TSumEY.x * TSumEY.x + TSumEY.y * TSumEY.y);
      double const EZ = (TSumEZ.x * TSumEZ.x + TSumEZ.y * TSumEZ.y);

      // Result up to this point
      this_result = fabs((*C2) * (EX + EY + EZ));

      result_precision = fabs((last_result - this_result) / last_result);

      // If below desired precision set as done
      if ( ilevel > 8 && result_precision < *prec ) {
        done = true;
        result_level = ilevel;
      }

      // Keep track of last result for precision test
      last_result = this_result;
    }

    // Check if any thread is not done
    _all_done = true;
    __syncthreads();
    if (!done) {
      _all_done = false;
    }
    __syncthreads();

    // Next level has *2 trajectory points
    this_nt *= 2;
  }

  // We're done, but if you're not a valid point just return
  if (!in_spectrum) {
    return;
  }

  // Set result and return
  switch (*rt) {
    case 1:
      result[ith] = result_precision;
      break;
    case 2:
      result[ith] = (double) result_level;
      break;
    default:
      result[ith] = this_result;
      break;
  }

  return;
}










extern "C" void OSCARSSR_Cuda_CalculateSpectrumGPU (OSCARSSR& OSR,
                                                    TVector3D const& ObservationPoint,
                                                    TSpectrumContainer& Spectrum,
                                                    std::string const& Polarization,
                                                    double const Angle,
                                                    TVector3D const& HorizontalDirection,
                                                    TVector3D const& PropogationDirection,
                                                    int const NParticles,
                                                    std::vector<int> const& GPUVector,
                                                    double const Precision,
                                                    int const MaxLevel,
                                                    int    const ReturnQuantity)
{
  // Calculate the spectrum for NParticles using the GPUs given in GPUVector.  Each particle's
  // trajectory will be sent to all GPUs for processing, meanwhile a new trajectory will
  // be calculated

  // Number of available GPUs
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Make sure that a gpu listed is within the range and not a duplicate
  std::vector<int> GPUsToUse;
  for (std::vector<int>::const_iterator it = GPUVector.begin(); it != GPUVector.end(); ++it) {
    if ( !(std::find(GPUsToUse.begin(), GPUsToUse.end(), *it) != GPUsToUse.end() && (*it < ngpu)) ) {
      GPUsToUse.push_back(*it);
    }
  }

  // Make sure we have at least one
  if (GPUsToUse.size() == 0) {
    throw std::invalid_argument("GPUs selected do not match hardware");
  }
  int const NGPUsToUse = (int) GPUsToUse.size();

  // Do we calculate for the current particle?
  bool const ThisParticleOnly = NParticles == 0 ? true : false;
  int  const NParticlesReally = ThisParticleOnly ? 1 : NParticles;


  // Calculate polarization vector to use (0, 0, 0) for 'all'
  TVector3D  const VerticalDirection = PropogationDirection.Cross(HorizontalDirection).UnitVector();
  TVector3DC const Positive = 1. / sqrt(2) * (TVector3DC(HorizontalDirection) + VerticalDirection * std::complex<double>(0, 1) );
  TVector3DC const Negative = 1. / sqrt(2) * (TVector3DC(HorizontalDirection) - VerticalDirection * std::complex<double>(0, 1) );
  TVector3DC PhotonPolarizationVector(0, 0, 0);
  if (Polarization == "all") {
    // Do nothing, it is already ALL
  } else if (Polarization == "linear-horizontal") {
    PhotonPolarizationVector = HorizontalDirection;
  } else if (Polarization == "linear-vertical") {
    PhotonPolarizationVector = VerticalDirection;
  } else if (Polarization == "linear") {
    TVector3D PolarizationAngle = HorizontalDirection;
    PolarizationAngle.RotateSelf(Angle, PropogationDirection);
    PhotonPolarizationVector = PolarizationAngle;
  } else if (Polarization == "circular-left") {
    PhotonPolarizationVector = Positive;
  } else if (Polarization == "circular-right") {
    PhotonPolarizationVector = Negative;
  } else {
    // Throw invalid argument if polarization is not recognized
    throw std::invalid_argument("Polarization requested not recognized");
  }


  // Observation point, number of spectrum points, and trajectory points
  double *h_obs;
  int    *h_no;
  int    *h_nt;
  hipDoubleComplex *h_pol;
  hipHostAlloc((void**) &h_obs, 3 * sizeof(double), hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_no,      sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nt,      sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_pol, 3 * sizeof(hipDoubleComplex), hipHostMallocWriteCombined | hipHostMallocMapped);

  // Set observation point values
  h_obs[0] = ObservationPoint.GetX();
  h_obs[1] = ObservationPoint.GetY();
  h_obs[2] = ObservationPoint.GetZ();

  // First one, set particle and trajectory
  if (!ThisParticleOnly) {
    OSR.SetNewParticle();
  }
  if (OSR.GetTrajectory().GetNPoints() == 0) {
    OSR.CalculateTrajectory();
  }

  // Needed number of points in the track and time step
  *h_nt     = (int) OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetNPoints();
  *h_no     = (int) Spectrum.GetNPoints();

  // Photon polarization
  h_pol[0] = make_hipDoubleComplex(PhotonPolarizationVector.GetX().real(), PhotonPolarizationVector.GetX().imag());
  h_pol[1] = make_hipDoubleComplex(PhotonPolarizationVector.GetY().real(), PhotonPolarizationVector.GetY().imag());
  h_pol[2] = make_hipDoubleComplex(PhotonPolarizationVector.GetZ().real(), PhotonPolarizationVector.GetZ().imag());

  int const NThreads = *h_no;
  int const NThreadsPerBlock = NTHREADS_PER_BLOCK;
  int const NThreadsRemainder = NThreads % NThreadsPerBlock;
  int const NBlocksTotal = (NThreads - 1) / NThreadsPerBlock + 1;
  int const NBlocksPerGPU = NBlocksTotal / NGPUsToUse;
  int const NRemainderBlocks = NBlocksTotal % NGPUsToUse;
  // UPDATE: To be modified
  int const NTT = NThreadsPerBlock * (NBlocksPerGPU + (NRemainderBlocks > 0 ? 1 : 0));

  std::vector<int> NBlocksThisGPU(NGPUsToUse, NBlocksPerGPU);
  for (int i = 0; i < NRemainderBlocks; ++i) {
    ++NBlocksThisGPU[i];
  }

  // Interpolating structure
  double  *h_t;
  double  *h_tstart, *h_tstop;
  double  *h_x,   *h_y,   *h_z;
  double  *h_xp,  *h_yp,  *h_zp;
  double  *h_bx,  *h_by,  *h_bz;
  double  *h_bxp, *h_byp, *h_bzp;
  double  *h_ax,  *h_ay,  *h_az;
  double  *h_axp, *h_ayp, *h_azp;

  // Spectrum points
  double  *h_om;

  // Constants and photon frequency
  double *h_c0,  *h_c2,  *h_c;

  // first point for each thread, max level
  int     *h_ifirst;
  int     *h_ml;

  // Precision desired
  double *h_prec;

  // Return quantity flag
  int    *h_rt;

  // Results
  double **h_result;

  // Allocate host memory
  hipHostAlloc((void**) &h_t,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_tstart,          sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_tstop,           sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_x,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_y,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_z,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_xp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_yp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_zp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bx,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_by,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bz,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bxp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_byp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bzp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ax,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ay,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_az,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_axp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ayp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_azp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_om,          *h_no * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_c0,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c2,                  sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_c,                   sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ifirst, NGPUsToUse * sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ml,                  sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_prec,                sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_rt,                  sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_result,   NGPUsToUse * sizeof(double*), hipHostMallocWriteCombined | hipHostMallocMapped);
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    hipHostAlloc((void**) &(h_result[i]), NTT * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  }

  // First surface point for each gpu
  int NBlocksUsed = 0;
  for (int i = 0; i < NGPUsToUse; ++i) {
    h_ifirst[i] = NBlocksUsed * NThreadsPerBlock;
    NBlocksUsed += NBlocksThisGPU[i];
  }

  *h_ml = MaxLevel; //UPDATE: max level should be an input

  // Precision
  *h_prec = Precision;

  // Return quantity
  *h_rt = ReturnQuantity;

  // Memor allocations for GPU
  int    **d_nt;
  double **d_tstart;
  double **d_tstop;
  int    **d_no;

  double **d_obs;
  hipDoubleComplex **d_pol;

  double **d_t;

  double **d_x;
  double **d_y;
  double **d_z;
  double **d_xp;
  double **d_yp;
  double **d_zp;

  double **d_bx;
  double **d_by;
  double **d_bz;
  double **d_bxp;
  double **d_byp;
  double **d_bzp;

  double **d_ax;
  double **d_ay;
  double **d_az;
  double **d_axp;
  double **d_ayp;
  double **d_azp;

  double **d_om;

  double **d_c0;
  double **d_c2;
  double **d_c;

  int    **d_ifirst;
  int    **d_ml;
  double **d_prec;
  int    **d_rt;
  double **d_result;

  hipHostAlloc((void **) &d_nt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_tstart, NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_tstop,  NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_no,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_obs,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_pol,    NGPUsToUse * sizeof(hipDoubleComplex*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_t,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_x,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_y,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_z,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_xp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_yp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_zp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_bx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_by,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bxp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_byp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bzp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_ax,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ay,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_az,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_axp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ayp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_azp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_om,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_c0,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c2,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_c,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_ifirst, NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ml,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_prec,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_rt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_result,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipMalloc((void **) &d_nt[i],                 sizeof(int));
    hipMalloc((void **) &d_tstart[i],             sizeof(double));
    hipMalloc((void **) &d_tstop[i],              sizeof(double));
    hipMalloc((void **) &d_no[i],                 sizeof(int));

    hipMalloc((void **) &d_obs[i],            3 * sizeof(double));
    hipMalloc((void **) &d_pol[i],            3 * sizeof(hipDoubleComplex));

    hipMalloc((void **) &d_t[i],          *h_nt * sizeof(double));

    hipMalloc((void **) &d_x[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_y[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_z[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_xp[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_yp[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_zp[i],         *h_nt * sizeof(double));

    hipMalloc((void **) &d_bx[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_by[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_bz[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_bxp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_byp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_bzp[i],        *h_nt * sizeof(double));

    hipMalloc((void **) &d_ax[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_ay[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_az[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_axp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_ayp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_azp[i],        *h_nt * sizeof(double));

    hipMalloc((void **) &d_om[i],         *h_no * sizeof(double));

    hipMalloc((void **) &d_c0[i],                 sizeof(double));
    hipMalloc((void **) &d_c2[i],                 sizeof(double));
    hipMalloc((void **) &d_c[i],                  sizeof(double));

    hipMalloc((void **) &d_ifirst[i],             sizeof(int));
    hipMalloc((void **) &d_ml[i],                 sizeof(int));
    hipMalloc((void **) &d_prec[i],               sizeof(double));
    hipMalloc((void **) &d_rt[i],                 sizeof(int));
    hipMalloc((void **) &d_result[i],       NTT * sizeof(double));

    // Copy device number to device
    hipMemcpyAsync(d_ifirst[i], &(h_ifirst[i]), sizeof(int), hipMemcpyHostToDevice);
  }

  // Compute known host values
  *h_c0    = OSR.GetCurrentParticle().GetQ() / (TOSCARSSR::FourPi() * TOSCARSSR::C() * TOSCARSSR::Epsilon0() * TOSCARSSR::Sqrt2Pi());
  *h_c2    = TOSCARSSR::FourPi() * OSR.GetCurrentParticle().GetCurrent() / (TOSCARSSR::H() * fabs(OSR.GetCurrentParticle().GetQ()) * TOSCARSSR::Mu0() * TOSCARSSR::C()) * 1e-6 * 0.001;
  *h_c     = TOSCARSSR::C();
  for (size_t i = 0; i < *h_no; ++i) {
    h_om[i] = Spectrum.GetAngularFrequency(i);
  }

  // Copy constants to first device (async)
  int const d0 = GPUsToUse[0];
  hipSetDevice(d0);
  hipMemcpyAsync(d_no[0],    h_no,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_obs[0],   h_obs,     3 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_pol[0],   h_pol,     3 * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c0[0],    h_c0,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c2[0],    h_c2,          sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_c[0],     h_c,           sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_om[0],    h_om,  *h_no * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_ml[0],    h_ml,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_prec[0],  h_prec,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_rt[0],    h_rt,          sizeof(int),    hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_no[i+1],     d1, d_no[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_obs[i+1],    d1, d_obs[i],    d, 3 * sizeof(double));
    hipMemcpyPeerAsync( d_pol[i+1],    d1, d_pol[i],    d, 3 * sizeof(hipDoubleComplex));
    hipMemcpyPeerAsync( d_c0[i+1],     d1, d_c0[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c2[i+1],     d1, d_c2[i],     d, sizeof(double));
    hipMemcpyPeerAsync( d_c[i+1],      d1, d_c[i],      d, sizeof(double));
    hipMemcpyPeerAsync( d_om[i+1],     d1, d_om[i],     d, *h_no * sizeof(double));
    hipMemcpyPeerAsync( d_ml[i+1],     d1, d_ml[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_prec[i+1],   d1, d_prec[i],   d, sizeof(double));
    hipMemcpyPeerAsync( d_rt[i+1],     d1, d_rt[i],     d, sizeof(int));
  }

  // Set first trajectory
  TOMATH::TSpline1D3<TParticleTrajectoryPoint> const& T = OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetSpline();
  int const NPointsThisTrajectory = T.GetNPoints();
  *h_nt = NPointsThisTrajectory;
  *h_tstart = T.GetXStart();
  *h_tstop  = T.GetXStop();
  for (size_t i = 0; i < NPointsThisTrajectory; ++i) {
    TParticleTrajectoryPoint const& P  = T.GetY(i);
    TParticleTrajectoryPoint const& PP = T.GetYPP(i);

    h_t[i]   = T.GetX(i);

    h_x[i]   =  P.GetX().GetX();
    h_y[i]   =  P.GetX().GetY();
    h_z[i]   =  P.GetX().GetZ();
    h_xp[i]  = PP.GetX().GetX();
    h_yp[i]  = PP.GetX().GetY();
    h_zp[i]  = PP.GetX().GetZ();

    h_bx[i]  =  P.GetB().GetX();
    h_by[i]  =  P.GetB().GetY();
    h_bz[i]  =  P.GetB().GetZ();
    h_bxp[i] = PP.GetB().GetX();
    h_byp[i] = PP.GetB().GetY();
    h_bzp[i] = PP.GetB().GetZ();

    h_ax[i]  =  P.GetAoverC().GetX();
    h_ay[i]  =  P.GetAoverC().GetY();
    h_az[i]  =  P.GetAoverC().GetZ();
    h_axp[i] = PP.GetAoverC().GetX();
    h_ayp[i] = PP.GetAoverC().GetY();
    h_azp[i] = PP.GetAoverC().GetZ();
  }


  // Set the surface points
  // GPU events
  hipEvent_t *event_resultcopy = new hipEvent_t[NGPUsToUse];
  for (int ig = 0; ig < NGPUsToUse; ++ig) {
    int const d = GPUsToUse[ig];
    hipSetDevice(d);
    hipEventCreate(&(event_resultcopy[ig]));
  }

  // Enable peer (direct gpu-gpu) writes
  for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
    // Device number
    int const d  = GPUsToUse[ig];
    int const d1 = GPUsToUse[ig+1];
    int access;
    hipDeviceCanAccessPeer(&access, d, d1);
    if (access == 1) {
      hipSetDevice(d);
      hipDeviceEnablePeerAccess(d1, 0);
    }
  }

  // Loop over number of particles
  for (int ip = 0; ip < NParticlesReally; ++ip) {

    // Copy trajectory to first GPU, then internal async transfers (where possible)
    hipSetDevice(d0);
    hipMemcpyAsync(d_nt[0],  h_nt,          sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpyAsync(d_t[0],   h_t,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_tstart[0], h_tstart,   sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_tstop[0],  h_tstop,    sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_x[0],   h_x,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_y[0],   h_y,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_z[0],   h_z,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_xp[0],  h_xp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_yp[0],  h_yp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_zp[0],  h_zp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bx[0],  h_bx,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_by[0],  h_by,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bz[0],  h_bz,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bxp[0], h_bxp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_byp[0], h_byp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bzp[0], h_bzp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ax[0],  h_ax,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ay[0],  h_ay,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_az[0],  h_az,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_axp[0], h_axp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ayp[0], h_ayp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_azp[0], h_azp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
      // Device number
      int const d  = GPUsToUse[ig];
      int const d1 = GPUsToUse[ig+1];
      hipSetDevice(d);
      hipMemcpyPeerAsync(d_nt[ig+1],  d1, d_nt[ig],  d,         sizeof(int));
      hipMemcpyPeerAsync(d_t[ig+1],   d1, d_t[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_tstart[ig+1], d1, d_tstart[ig], d,   sizeof(double));
      hipMemcpyPeerAsync(d_tstop[ig+1],  d1, d_tstop[ig],  d,    sizeof(double));
      hipMemcpyPeerAsync(d_x[ig+1],   d1, d_x[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_y[ig+1],   d1, d_y[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_z[ig+1],   d1, d_z[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_xp[ig+1],  d1, d_xp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_yp[ig+1],  d1, d_yp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_zp[ig+1],  d1, d_zp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bx[ig+1],  d1, d_bx[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_by[ig+1],  d1, d_by[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bz[ig+1],  d1, d_bz[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bxp[ig+1], d1, d_bxp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_byp[ig+1], d1, d_byp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bzp[ig+1], d1, d_bzp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ax[ig+1],  d1, d_ax[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ay[ig+1],  d1, d_ay[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_az[ig+1],  d1, d_az[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_axp[ig+1], d1, d_axp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ayp[ig+1], d1, d_ayp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_azp[ig+1], d1, d_azp[ig], d, *h_nt * sizeof(double));

    }

    // Wait for previous copy, start next one
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d = GPUsToUse[ig];
      hipSetDevice(d);
      hipEventSynchronize(event_resultcopy[ig]);
      OSCARSSR_Cuda_SpectrumGPU <<<NBlocksThisGPU[ig], NThreadsPerBlock>>>(
                                                                           d_t[ig],
                                                                           d_x[ig],   d_y[ig],   d_z[ig],
                                                                           d_xp[ig],  d_yp[ig],  d_zp[ig],
                                                                           d_bx[ig],  d_by[ig],  d_bz[ig],
                                                                           d_bxp[ig], d_byp[ig], d_bzp[ig],
                                                                           d_ax[ig],  d_ay[ig],  d_az[ig],
                                                                           d_axp[ig], d_ayp[ig], d_azp[ig],
                                                                           d_obs[ig],
                                                                           d_pol[ig],
                                                                           d_tstart[ig], d_tstop[ig],
                                                                           d_nt[ig],
                                                                           d_om[ig],
                                                                           d_no[ig],
                                                                           d_c0[ig], d_c2[ig], d_c[ig],
                                                                           d_ifirst[ig],
                                                                           d_ml[ig],
                                                                           d_prec[ig],
                                                                           d_rt[ig],
                                                                           d_result[ig]);
    }


    // Add result to results container (from **previous**)
    if (ip > 0) {
      int NBlocksUsed = 0;
      for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
        for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
          if (ith + NThreadsPerBlock * NBlocksUsed >= *h_no) {
            break;
          }
          int iss = ith + NThreadsPerBlock * NBlocksUsed;
          Spectrum.AddToFlux(iss, h_result[ig][ith]);
        }
        NBlocksUsed += NBlocksThisGPU[ig];
      }
    }

    // Add copy back to streams
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d  = GPUsToUse[ig];
      hipSetDevice(d);
      hipMemcpyAsync(h_result[ig],  d_result[ig],  NTT * sizeof(double), hipMemcpyDeviceToHost);
      hipEventRecord(event_resultcopy[ig]);
    }

    // If it's not the last one, calculate a new trajectory
    if (ip < NParticlesReally - 1) {
      OSR.SetNewParticle();
      OSR.CalculateTrajectory();

      TOMATH::TSpline1D3<TParticleTrajectoryPoint> const& T = OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetSpline();
      int const NPointsThisTrajectory = T.GetNPoints();
      *h_nt = NPointsThisTrajectory;
      for (size_t it = 0; it < NPointsThisTrajectory; ++it) {
        TParticleTrajectoryPoint const& P  = T.GetY(it);
        TParticleTrajectoryPoint const& PP = T.GetYPP(it);

        h_t[it]   = T.GetX(it);

        h_x[it]   =  P.GetX().GetX();
        h_y[it]   =  P.GetX().GetY();
        h_z[it]   =  P.GetX().GetZ();
        h_xp[it]  = PP.GetX().GetX();
        h_yp[it]  = PP.GetX().GetY();
        h_zp[it]  = PP.GetX().GetZ();

        h_bx[it]  =  P.GetB().GetX();
        h_by[it]  =  P.GetB().GetY();
        h_bz[it]  =  P.GetB().GetZ();
        h_bxp[it] = PP.GetB().GetX();
        h_byp[it] = PP.GetB().GetY();
        h_bzp[it] = PP.GetB().GetZ();

        h_ax[it]  =  P.GetAoverC().GetX();
        h_ay[it]  =  P.GetAoverC().GetY();
        h_az[it]  =  P.GetAoverC().GetZ();
        h_axp[it] = PP.GetAoverC().GetX();
        h_ayp[it] = PP.GetAoverC().GetY();
        h_azp[it] = PP.GetAoverC().GetZ();
      }
    }
  }

  // Wait for last copy
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    hipEventSynchronize(event_resultcopy[ig]);
  }

  // Add result to results container (from **previous**)
  NBlocksUsed = 0;
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
      if (ith + NThreadsPerBlock * NBlocksUsed >= *h_no) {
        break;
      }
      int iss = ith + NThreadsPerBlock * NBlocksUsed;
      Spectrum.AddToFlux(iss, h_result[ig][ith]);
    }
    NBlocksUsed += NBlocksThisGPU[ig];
  }

  // Weighting for multi-particle
  double const Weight = 1.0 / (double) NParticlesReally;
  Spectrum.Scale(Weight);

  // Free host memory
  hipHostFree(h_nt);
  hipHostFree(h_t);
  hipHostFree(h_no);
  hipHostFree(h_obs);
  hipHostFree(h_pol);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_xp);
  hipHostFree(h_yp);
  hipHostFree(h_zp);
  hipHostFree(h_bx);
  hipHostFree(h_by);
  hipHostFree(h_bz);
  hipHostFree(h_bxp);
  hipHostFree(h_byp);
  hipHostFree(h_bzp);
  hipHostFree(h_ax);
  hipHostFree(h_ay);
  hipHostFree(h_az);
  hipHostFree(h_axp);
  hipHostFree(h_ayp);
  hipHostFree(h_azp);
  hipHostFree(h_om);
  hipHostFree(h_c0);
  hipHostFree(h_c2);
  hipHostFree(h_c);
  hipHostFree(h_ifirst);
  hipHostFree(h_ml);
  hipHostFree(h_prec);
  hipHostFree(h_rt);
  // Free host and GPU memory
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {

    hipHostFree(h_result[i]);

    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipFree(d_nt[i]);
    hipFree(d_t[i]);
    hipFree(d_no[i]);
    hipFree(d_obs[i]);
    hipFree(d_pol[i]);
    hipFree(d_x[i]);
    hipFree(d_y[i]);
    hipFree(d_z[i]);
    hipFree(d_xp[i]);
    hipFree(d_yp[i]);
    hipFree(d_zp[i]);
    hipFree(d_bx[i]);
    hipFree(d_by[i]);
    hipFree(d_bz[i]);
    hipFree(d_bxp[i]);
    hipFree(d_byp[i]);
    hipFree(d_bzp[i]);
    hipFree(d_ax[i]);
    hipFree(d_ay[i]);
    hipFree(d_az[i]);
    hipFree(d_axp[i]);
    hipFree(d_ayp[i]);
    hipFree(d_azp[i]);
    hipFree(d_om[i]);
    hipFree(d_c0[i]);
    hipFree(d_c2[i]);
    hipFree(d_c[i]);
    hipFree(d_ifirst[i]);
    hipFree(d_ml[i]);
    hipFree(d_prec[i]);
    hipFree(d_rt[i]);
    hipFree(d_result[i]);
  }
  hipFree(h_result);

  hipFree(d_nt);
  hipFree(d_t);
  hipFree(d_no);
  hipFree(d_obs);
  hipFree(d_pol);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_xp);
  hipFree(d_yp);
  hipFree(d_zp);
  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);
  hipFree(d_bxp);
  hipFree(d_byp);
  hipFree(d_bzp);
  hipFree(d_ax);
  hipFree(d_ay);
  hipFree(d_az);
  hipFree(d_axp);
  hipFree(d_ayp);
  hipFree(d_azp);
  hipFree(d_om);
  hipFree(d_c0);
  hipFree(d_c2);
  hipFree(d_c);
  hipFree(h_ifirst);
  hipFree(d_ml);
  hipFree(d_prec);
  hipFree(d_rt);
  hipFree(d_result);

  // Delete host gpu pointer arrays
  delete [] event_resultcopy;


  return;
}

















__global__ void OSCARSSR_Cuda_PowerDensityGPU (double  *t,
                                               double  *x,   double *y,   double *z,
                                               double  *xp,  double *yp,  double *zp,
                                               double  *bx,  double *by,  double *bz,
                                               double  *bxp, double *byp, double *bzp,
                                               double  *ax,  double *ay,  double *az,
                                               double  *axp, double *ayp, double *azp,
                                               double  *sx,  double *sy,  double *sz,
                                               double  *nx,  double *ny,  double *nz,
                                               double  *cons,
                                               double  *tstart, double *tstop,
                                               int *nt,
                                               int *ns,
                                               int *shn,
                                               int *ifirst,
                                               int *ml,
                                               double *prec,
                                               int    *rt,
                                               double *result)
{
  // Thread number and surface number
  int const ith = threadIdx.x + blockIdx.x * blockDim.x;
  int const is = ith + *ifirst;

  // Alternative returns
  double result_precision = -1;
  int    result_level = -1;

  // Observation point
  double const ox = is >= *ns ? 0 : sx[is];
  double const oy = is >= *ns ? 0 : sy[is];
  double const oz = is >= *ns ? 0 : sz[is];


  // Trajectory interpolated on fly and stored in shared memory for this block
  __shared__ double _t[NTHREADS_PER_BLOCK];
  __shared__ double _x[NTHREADS_PER_BLOCK];
  __shared__ double _y[NTHREADS_PER_BLOCK];
  __shared__ double _z[NTHREADS_PER_BLOCK];
  __shared__ double _bx[NTHREADS_PER_BLOCK];
  __shared__ double _by[NTHREADS_PER_BLOCK];
  __shared__ double _bz[NTHREADS_PER_BLOCK];
  __shared__ double _ax[NTHREADS_PER_BLOCK];
  __shared__ double _ay[NTHREADS_PER_BLOCK];
  __shared__ double _az[NTHREADS_PER_BLOCK];

  // Are all threads in this block done?
  __shared__ bool _all_done;

  // Is this thread a valid surface point?
  bool const in_surface = ((is < *ns) ? true : false);

  // I will consider you "done" if you are not a valid surface
  bool done = !in_surface;

  // Start off with the assumption that not all threads are done
  _all_done = false;


  // Number of trajectory points in current level
  int this_nt = 1;


  // Normal vector from input
  double const NormalX = nx[is];
  double const NormalY = ny[is];
  double const NormalZ = nz[is];

  // Summed result
  double Sum = 0;

  // Result up to this level and from last level for comparison
  double this_result = 0;
  double last_result = 1;

  // DeltaT for all levels up to this level
  double dt_total = 0;

  // Loop over all levels 
  for (int ilevel = 0; (ilevel <= *ml) && !_all_done; ++ilevel) {

    // DeltaT inclusive up to this level
    dt_total = (*tstop - *tstart) / pow(2., ilevel+1);//(*tstop - *tstart) / (2 * this_nt);

    // deltaT this level and Time start this level
    double const dt = (*tstop - *tstart) / pow(2., ilevel);//(*tstop - *tstart) / this_nt;
    double const ts = *tstart + (*tstop - *tstart) / pow(2., ilevel + 1);//*tstart + (*tstop - *tstart) / (2. * this_nt);

    // Number of trajectory "blocks" we will have to loop over for this ilevel
    int const NTrajectoryBlocks = this_nt / blockDim.x + (this_nt % blockDim.x == 0 ? 0 : 1);

    // Loop over all trajectory blocks
    for (int itb = 0; itb < NTrajectoryBlocks; ++itb) {

      // You better sync threads before you attempt to write to shared memory
      __syncthreads();

      // Calculate time of this thread in this trajectory block
      _t[threadIdx.x] = dt * (itb * blockDim.x + threadIdx.x) + ts;

      // If the time is outside we will not consider it
      if (_t[threadIdx.x] < *tstop) {

        // First get interpolating imin and imax for this time
        int imin, imax;
        GetInterpolatingIMinIMax(t, nt, &(_t[threadIdx.x]), &imin, &imax);

        // Fill trajectory with interpolated data
        _x[threadIdx.x]  = Interpolate(t,  x,  xp, &(_t[threadIdx.x]), &imin, &imax);
        _y[threadIdx.x]  = Interpolate(t,  y,  yp, &(_t[threadIdx.x]), &imin, &imax);
        _z[threadIdx.x]  = Interpolate(t,  z,  zp, &(_t[threadIdx.x]), &imin, &imax);
        _bx[threadIdx.x] = Interpolate(t, bx, bxp, &(_t[threadIdx.x]), &imin, &imax);
        _by[threadIdx.x] = Interpolate(t, by, byp, &(_t[threadIdx.x]), &imin, &imax);
        _bz[threadIdx.x] = Interpolate(t, bz, bzp, &(_t[threadIdx.x]), &imin, &imax);
        _ax[threadIdx.x] = Interpolate(t, ax, axp, &(_t[threadIdx.x]), &imin, &imax);
        _ay[threadIdx.x] = Interpolate(t, ay, ayp, &(_t[threadIdx.x]), &imin, &imax);
        _az[threadIdx.x] = Interpolate(t, az, azp, &(_t[threadIdx.x]), &imin, &imax);
      }

      // Must sync threads to make sure all trajectory data for this trajectory loop is written
      __syncthreads();

      // If this thread is not done then do some more calculations
      if (!done) {
        for (int i = 0; i < blockDim.x; ++i) {

          // Check if we are over the limit of trajectory points
          if (in_surface && (_t[i] < *tstop)) {

            // Normal vector in direction of observation point
            double const R1 = sqrt( pow(ox - _x[i], 2) + pow(oy - _y[i], 2) + pow(oz - _z[i], 2) );
            double const N1X = (ox - _x[i]) / R1;
            double const N1Y = (oy - _y[i]) / R1;
            double const N1Z = (oz - _z[i]) / R1;

            // Surface normal dot with vector normal
            double const N1DotNormal = *shn == 1 ? N1X * NormalX + N1Y * NormalY + N1Z * NormalZ : 1;

            // Orthogonal vector 2 & 3
            double N2X;
            double N2Y;
            double N2Z;

            double const xx = N1X < 0.0 ? -N1X : N1X;
            double const yy = N1Y < 0.0 ? -N1Y : N1Y;
            double const zz = N1Z < 0.0 ? -N1Z : N1Z;
            if (xx < yy) {
              if (xx < zz) {
                N2X = 0;
                N2Y = N1Z;
                N2Z = -N1Y;
              } else {
                N2X = N1Y;
                N2Y = -N1X;
                N2Z = 0;
              }
            } else {
              if (yy < zz) {
                N2X = -N1Z;
                N2Y = 0;
                N2Z = N1X;
              } else {
                N2X = N1Y;
                N2Y = -N1X;
                N2Z = 0;
              }
            }
            double const R2 = sqrt(N2X * N2X + N2Y * N2Y + N2Z * N2Z);
            N2X /= R2;
            N2Y /= R2;
            N2Z /= R2;

            // Ortohgonal vector N3
            double const N3X = N1Y * N2Z - N1Z * N2Y;
            double const N3Y = N1Z * N2X - N1X * N2Z;
            double const N3Z = N1X * N2Y - N1Y * N2X;

            double const x1 = N1X - _bx[i];
            double const y1 = N1Y - _by[i];
            double const z1 = N1Z - _bz[i];

            double const x2 = y1 * _az[i] - z1 * _ay[i];
            double const y2 = z1 * _ax[i] - x1 * _az[i];
            double const z2 = x1 * _ay[i] - y1 * _ax[i];

            // Numerator = N1.Cross( ( (N1 - B).Cross((AoverC)) ) );
            double const x3 = N1Y * z2 - N1Z * y2;
            double const y3 = N1Z * x2 - N1X * z2;
            double const z3 = N1X * y2 - N1Y * x2;

            double const BdotN1 = _bx[i] * N1X + _by[i] * N1Y + _bz[i] * N1Z;
            double const Denominator = pow(1. - BdotN1, 5);

            Sum += pow( x3 * N2X + y3 * N2Y + z3 * N2Z, 2) / Denominator / (R1 * R1) * N1DotNormal;
            Sum += pow( x3 * N3X + y3 * N3Y + z3 * N3Z, 2) / Denominator / (R1 * R1) * N1DotNormal;


          }
        }
      }
    }

    // If this thread is a valid surface point and it is not "done" yet, calculate the result
    if (in_surface && !done) {

      // Result up to this point
      this_result = Sum * dt_total;

      result_precision = fabs((last_result - this_result) / last_result);

      // If below desired precision set as done
      if ( ilevel > 8 && result_precision < *prec ) {
        done = true;
        result_level = ilevel;
      }

      // Keep track of last result for precision test
      last_result = this_result;
    }

    // Check if any thread is not done
    _all_done = true;
    __syncthreads();
    if (!done) {
      _all_done = false;
    }
    __syncthreads();

    // Next level has *2 trajectory points
    this_nt *= 2;
  }

  // We're done, but if you're not a surface point just return
  if (!in_surface) {
    return;
  }

  // Set result and return
  switch (*rt) {
    case 1:
      result[ith] = result_precision;
      break;
    case 2:
      result[ith] = (double) result_level;
      break;
    default:
      result[ith] = this_result * *cons;
      break;
  }

  return;
}


































extern "C" void OSCARSSR_Cuda_CalculatePowerDensityGPU (OSCARSSR& OSR,
                                                        TSurfacePoints const& Surface,
                                                        T3DScalarContainer& PowerDensityContainer,
                                                        int const NParticles,
                                                        std::vector<int> const& GPUVector,
                                                        double const Precision,
                                                        int const MaxLevel,
                                                        int const ReturnQuantity)
{
  // Calculate the power density for NParticles using the GPUs given in GPUVector.  Each particle's
  // trajectory will be sent to all GPUs for processing, meanwhile a new trajectory will
  // be calculated

  // Number of available GPUs
  int ngpu = 0;
  hipGetDeviceCount(&ngpu);
  if (ngpu == 0) {
    throw std::invalid_argument("No GPU found");
  }

  // Make sure that a gpu listed is within the range and not a duplicate
  std::vector<int> GPUsToUse;
  for (std::vector<int>::const_iterator it = GPUVector.begin(); it != GPUVector.end(); ++it) {
    if ( !(std::find(GPUsToUse.begin(), GPUsToUse.end(), *it) != GPUsToUse.end() && (*it < ngpu)) ) {
      GPUsToUse.push_back(*it);
    }
  }

  // Make sure we have at least one
  if (GPUsToUse.size() == 0) {
    throw std::invalid_argument("GPUs selected do not match hardware");
  }
  int const NGPUsToUse = (int) GPUsToUse.size();

  // Do we calculate for the current particle?
  bool const ThisParticleOnly = NParticles == 0 ? true : false;
  int  const NParticlesReally = ThisParticleOnly ? 1 : NParticles;


  // Number of surface points and trajectory points
  int *h_ns;
  int *h_nt;
  hipHostAlloc((void**) &h_ns,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nt,     sizeof(int),    hipHostMallocWriteCombined | hipHostMallocMapped);

  // First one, set particle and trajectory
  if (!ThisParticleOnly) {
    OSR.SetNewParticle();
  }
  if (OSR.GetTrajectory().GetNPoints() == 0) {
    OSR.CalculateTrajectory();
  }

  // Needed number of points in the track and time step
  *h_nt     = (int) OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetNPoints();
  *h_ns     = (int) Surface.GetNPoints();


  int const NThreads = *h_ns;
  int const NThreadsPerBlock = NTHREADS_PER_BLOCK;
  int const NThreadsRemainder = NThreads % NThreadsPerBlock;
  int const NBlocksTotal = (NThreads - 1) / NThreadsPerBlock + 1;
  int const NBlocksPerGPU = NBlocksTotal / NGPUsToUse;
  int const NRemainderBlocks = NBlocksTotal % NGPUsToUse;
  // UPDATE: To be modified
  int const NTT = NThreadsPerBlock * (NBlocksPerGPU + (NRemainderBlocks > 0 ? 1 : 0));

  std::vector<int> NBlocksThisGPU(NGPUsToUse, NBlocksPerGPU);
  for (int i = 0; i < NRemainderBlocks; ++i) {
    ++NBlocksThisGPU[i];
  }

  // Interpolating structure
  double  *h_t;
  double  *h_tstart, *h_tstop;
  double  *h_x,   *h_y,   *h_z;
  double  *h_xp,  *h_yp,  *h_zp;
  double  *h_bx,  *h_by,  *h_bz;
  double  *h_bxp, *h_byp, *h_bzp;
  double  *h_ax,  *h_ay,  *h_az;
  double  *h_axp, *h_ayp, *h_azp;

  // Surface points and normal vectors
  double  *h_sx,  *h_sy,  *h_sz;
  double  *h_nx,  *h_ny,  *h_nz;

  // Invert or ignore normal
  int     *h_shn;

  // Constant to multiply result by
  double *h_const;

  // first point for each thread, max level
  int     *h_ifirst;
  int     *h_ml;

  // Precision desired
  double *h_prec;

  // Return quantity flag
  int    *h_rt;

  // Results
  double **h_result;

  // Allocate host memory
  hipHostAlloc((void**) &h_t,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_tstart,          sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_tstop,           sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_x,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_y,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_z,       *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_xp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_yp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_zp,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bx,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_by,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bz,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bxp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_byp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_bzp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ax,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ay,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_az,      *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_axp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ayp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_azp,     *h_nt * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_sx,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sy,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_sz,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nx,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ny,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_nz,          *h_ns * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_shn,                 sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_const,               sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_ifirst, NGPUsToUse * sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_ml,                  sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_prec,                sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void**) &h_rt,                  sizeof(int),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void**) &h_result,   NGPUsToUse * sizeof(double*), hipHostMallocWriteCombined | hipHostMallocMapped);
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    hipHostAlloc((void**) &(h_result[i]), NTT * sizeof(double),  hipHostMallocWriteCombined | hipHostMallocMapped);
  }

  // First surface point for each gpu
  int NBlocksUsed = 0;
  for (int i = 0; i < NGPUsToUse; ++i) {
    h_ifirst[i] = NBlocksUsed * NThreadsPerBlock;
    NBlocksUsed += NBlocksThisGPU[i];
  }

  *h_shn = 1;
  *h_ml = MaxLevel;

  // Precision
  *h_prec = Precision;

  // Return quantity
  *h_rt = ReturnQuantity;

  // Memor allocations for GPU
  int    **d_nt;
  double **d_tstart;
  double **d_tstop;
  int    **d_ns;

  double **d_t;

  double **d_x;
  double **d_y;
  double **d_z;
  double **d_xp;
  double **d_yp;
  double **d_zp;

  double **d_bx;
  double **d_by;
  double **d_bz;
  double **d_bxp;
  double **d_byp;
  double **d_bzp;

  double **d_ax;
  double **d_ay;
  double **d_az;
  double **d_axp;
  double **d_ayp;
  double **d_azp;

  double **d_sx;
  double **d_sy;
  double **d_sz;
  double **d_nx;
  double **d_ny;
  double **d_nz;
  int    **d_shn;

  double **d_const;

  int    **d_ifirst;
  int    **d_ml;
  double **d_prec;
  int    **d_rt;
  double **d_result;

  hipHostAlloc((void **) &d_nt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_tstart, NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_tstop,  NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ns,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_t,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_x,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_y,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_z,      NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_xp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_yp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_zp,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_bx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_by,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bxp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_byp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_bzp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_ax,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ay,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_az,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_axp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ayp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_azp,    NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_sx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sy,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_sz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_nx,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ny,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_nz,     NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_shn,    NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_const,  NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  hipHostAlloc((void **) &d_ifirst, NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_ml,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_prec,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_rt,     NGPUsToUse * sizeof(int*),     hipHostMallocWriteCombined | hipHostMallocMapped);
  hipHostAlloc((void **) &d_result,   NGPUsToUse * sizeof(double*),  hipHostMallocWriteCombined | hipHostMallocMapped);

  for (size_t i = 0; i < GPUsToUse.size(); ++i) {
    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipMalloc((void **) &d_nt[i],                 sizeof(int));
    hipMalloc((void **) &d_tstart[i],             sizeof(double));
    hipMalloc((void **) &d_tstop[i],              sizeof(double));
    hipMalloc((void **) &d_ns[i],                 sizeof(int));

    hipMalloc((void **) &d_t[i],          *h_nt * sizeof(double));

    hipMalloc((void **) &d_x[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_y[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_z[i],          *h_nt * sizeof(double));
    hipMalloc((void **) &d_xp[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_yp[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_zp[i],         *h_nt * sizeof(double));

    hipMalloc((void **) &d_bx[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_by[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_bz[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_bxp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_byp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_bzp[i],        *h_nt * sizeof(double));

    hipMalloc((void **) &d_ax[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_ay[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_az[i],         *h_nt * sizeof(double));
    hipMalloc((void **) &d_axp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_ayp[i],        *h_nt * sizeof(double));
    hipMalloc((void **) &d_azp[i],        *h_nt * sizeof(double));

    hipMalloc((void **) &d_sx[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sy[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_sz[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_nx[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_ny[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_nz[i],         *h_ns * sizeof(double));
    hipMalloc((void **) &d_shn[i],                sizeof(int));
    hipMalloc((void **) &d_const[i],              sizeof(double));

    hipMalloc((void **) &d_ifirst[i],             sizeof(int));
    hipMalloc((void **) &d_ml[i],                 sizeof(int));
    hipMalloc((void **) &d_prec[i],               sizeof(double));
    hipMalloc((void **) &d_rt[i],                 sizeof(int));
    hipMalloc((void **) &d_result[i],       NTT * sizeof(double));

    // Copy device number to device
    hipMemcpyAsync(d_ifirst[i], &(h_ifirst[i]), sizeof(int), hipMemcpyHostToDevice);
  }

  for (size_t i = 0; i < *h_ns; ++i) {
    h_sx[i] = Surface.GetPoint(i).GetX();
    h_sy[i] = Surface.GetPoint(i).GetY();
    h_sz[i] = Surface.GetPoint(i).GetZ();
    h_nx[i] = Surface.GetPoint(i).GetNormalX();
    h_ny[i] = Surface.GetPoint(i).GetNormalY();
    h_nz[i] = Surface.GetPoint(i).GetNormalZ();
  }

  // Copy constants to first device (async)
  int const d0 = GPUsToUse[0];
  hipSetDevice(d0);
  hipMemcpyAsync(d_ns[0],    h_ns,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sx[0],    h_sx,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sy[0],    h_sy,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_sz[0],    h_sz,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_nx[0],    h_nx,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_ny[0],    h_ny,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_nz[0],    h_nz,  *h_ns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_shn[0],   h_shn,         sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_ml[0],    h_ml,          sizeof(int),    hipMemcpyHostToDevice);
  hipMemcpyAsync(d_prec[0],  h_prec,        sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_rt[0],    h_rt,          sizeof(int),    hipMemcpyHostToDevice);
  for (size_t i = 0; i < GPUsToUse.size() - 1; ++i) {
    // Device number
    int const d  = GPUsToUse[i];
    int const d1 = GPUsToUse[i+1];
    hipSetDevice(d);
    hipMemcpyPeerAsync( d_ns[i+1],     d1, d_ns[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_sx[i+1],     d1, d_sx[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sy[i+1],     d1, d_sy[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_sz[i+1],     d1, d_sz[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_nx[i+1],     d1, d_nx[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_ny[i+1],     d1, d_ny[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_nz[i+1],     d1, d_nz[i],     d, *h_ns * sizeof(double));
    hipMemcpyPeerAsync( d_shn[i+1],    d1, d_shn[i],    d, sizeof(int));
    hipMemcpyPeerAsync( d_ml[i+1],     d1, d_ml[i],     d, sizeof(int));
    hipMemcpyPeerAsync( d_prec[i+1],   d1, d_prec[i],   d, sizeof(double));
    hipMemcpyPeerAsync( d_rt[i+1],     d1, d_rt[i],     d, sizeof(int));
  }

  // Set first trajectory
  TOMATH::TSpline1D3<TParticleTrajectoryPoint> const& T = OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetSpline();
  int const NPointsThisTrajectory = T.GetNPoints();
  *h_nt = NPointsThisTrajectory;
  *h_tstart = T.GetXStart();
  *h_tstop  = T.GetXStop();
  for (size_t i = 0; i < NPointsThisTrajectory; ++i) {
    TParticleTrajectoryPoint const& P  = T.GetY(i);
    TParticleTrajectoryPoint const& PP = T.GetYPP(i);

    h_t[i]   = T.GetX(i);

    h_x[i]   =  P.GetX().GetX();
    h_y[i]   =  P.GetX().GetY();
    h_z[i]   =  P.GetX().GetZ();
    h_xp[i]  = PP.GetX().GetX();
    h_yp[i]  = PP.GetX().GetY();
    h_zp[i]  = PP.GetX().GetZ();

    h_bx[i]  =  P.GetB().GetX();
    h_by[i]  =  P.GetB().GetY();
    h_bz[i]  =  P.GetB().GetZ();
    h_bxp[i] = PP.GetB().GetX();
    h_byp[i] = PP.GetB().GetY();
    h_bzp[i] = PP.GetB().GetZ();

    h_ax[i]  =  P.GetAoverC().GetX();
    h_ay[i]  =  P.GetAoverC().GetY();
    h_az[i]  =  P.GetAoverC().GetZ();
    h_axp[i] = PP.GetAoverC().GetX();
    h_ayp[i] = PP.GetAoverC().GetY();
    h_azp[i] = PP.GetAoverC().GetZ();
  }


  // Set the surface points
  // GPU events
  hipEvent_t *event_resultcopy = new hipEvent_t[NGPUsToUse];
  for (int ig = 0; ig < NGPUsToUse; ++ig) {
    int const d = GPUsToUse[ig];
    hipSetDevice(d);
    hipEventCreate(&(event_resultcopy[ig]));
  }

  // Enable peer (direct gpu-gpu) writes
  for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
    // Device number
    int const d  = GPUsToUse[ig];
    int const d1 = GPUsToUse[ig+1];
    int access;
    hipDeviceCanAccessPeer(&access, d, d1);
    if (access == 1) {
      hipSetDevice(d);
      hipDeviceEnablePeerAccess(d1, 0);
    }
  }

  // Loop over number of particles
  for (int ip = 0; ip < NParticlesReally; ++ip) {
    // Set constant for this particle
    *h_const = fabs(OSR.GetCurrentParticle().GetQ() * OSR.GetCurrentParticle().GetCurrent()) / (16 * TOSCARSSR::Pi2() * TOSCARSSR::Epsilon0() * TOSCARSSR::C()) / 1e6;

    // Copy trajectory to first GPU, then internal async transfers (where possible)
    hipSetDevice(d0);
    hipMemcpyAsync(d_const[0], h_const,     sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_nt[0],  h_nt,          sizeof(int),    hipMemcpyHostToDevice);
    hipMemcpyAsync(d_t[0],   h_t,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_tstart[0], h_tstart,   sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_tstop[0],  h_tstop,    sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_x[0],   h_x,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_y[0],   h_y,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_z[0],   h_z,   *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_xp[0],  h_xp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_yp[0],  h_yp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_zp[0],  h_zp,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bx[0],  h_bx,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_by[0],  h_by,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bz[0],  h_bz,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bxp[0], h_bxp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_byp[0], h_byp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_bzp[0], h_bzp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ax[0],  h_ax,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ay[0],  h_ay,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_az[0],  h_az,  *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_axp[0], h_axp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_ayp[0], h_ayp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_azp[0], h_azp, *h_nt * sizeof(double), hipMemcpyHostToDevice);
    for (size_t ig = 0; ig < GPUsToUse.size() - 1; ++ig) {
      // Device number
      int const d  = GPUsToUse[ig];
      int const d1 = GPUsToUse[ig+1];
      hipSetDevice(d);
      hipMemcpyPeerAsync(d_const[ig+1],  d1, d_const[ig], d,    sizeof(double));
      hipMemcpyPeerAsync(d_nt[ig+1],  d1, d_nt[ig],  d,         sizeof(int));
      hipMemcpyPeerAsync(d_t[ig+1],   d1, d_t[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_tstart[ig+1], d1, d_tstart[ig], d,   sizeof(double));
      hipMemcpyPeerAsync(d_tstop[ig+1],  d1, d_tstop[ig],  d,    sizeof(double));
      hipMemcpyPeerAsync(d_x[ig+1],   d1, d_x[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_y[ig+1],   d1, d_y[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_z[ig+1],   d1, d_z[ig],   d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_xp[ig+1],  d1, d_xp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_yp[ig+1],  d1, d_yp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_zp[ig+1],  d1, d_zp[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bx[ig+1],  d1, d_bx[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_by[ig+1],  d1, d_by[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bz[ig+1],  d1, d_bz[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bxp[ig+1], d1, d_bxp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_byp[ig+1], d1, d_byp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_bzp[ig+1], d1, d_bzp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ax[ig+1],  d1, d_ax[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ay[ig+1],  d1, d_ay[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_az[ig+1],  d1, d_az[ig],  d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_axp[ig+1], d1, d_axp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_ayp[ig+1], d1, d_ayp[ig], d, *h_nt * sizeof(double));
      hipMemcpyPeerAsync(d_azp[ig+1], d1, d_azp[ig], d, *h_nt * sizeof(double));

    }

    // Wait for previous copy, start next one
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d = GPUsToUse[ig];
      hipSetDevice(d);
      hipEventSynchronize(event_resultcopy[ig]);
      OSCARSSR_Cuda_PowerDensityGPU<<<NBlocksThisGPU[ig], NThreadsPerBlock>>>(
                                                                              d_t[ig],
                                                                              d_x[ig],   d_y[ig],   d_z[ig],
                                                                              d_xp[ig],  d_yp[ig],  d_zp[ig],
                                                                              d_bx[ig],  d_by[ig],  d_bz[ig],
                                                                              d_bxp[ig], d_byp[ig], d_bzp[ig],
                                                                              d_ax[ig],  d_ay[ig],  d_az[ig],
                                                                              d_axp[ig], d_ayp[ig], d_azp[ig],
                                                                              d_sx[ig],  d_sy[ig],  d_sz[ig],
                                                                              d_nx[ig],  d_ny[ig],  d_nz[ig],
                                                                              d_const[ig],
                                                                              d_tstart[ig], d_tstop[ig],
                                                                              d_nt[ig],
                                                                              d_ns[ig],
                                                                              d_shn[ig],
                                                                              d_ifirst[ig],
                                                                              d_ml[ig],
                                                                              d_prec[ig],
                                                                              d_rt[ig],
                                                                              d_result[ig]);
    }


    // Add result to result container (from **previous**)
    if (ip > 0) {
      int NBlocksUsed = 0;
      for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
        for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
          if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
            break;
          }
          int iss = ith + NThreadsPerBlock * NBlocksUsed;
          PowerDensityContainer.AddToPoint(iss, h_result[ig][ith]);
        }
        NBlocksUsed += NBlocksThisGPU[ig];
      }
    }

    // Add copy back to streams
    for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
      int const d  = GPUsToUse[ig];
      hipSetDevice(d);
      hipMemcpyAsync(h_result[ig],  d_result[ig],  NTT * sizeof(double), hipMemcpyDeviceToHost);
      hipEventRecord(event_resultcopy[ig]);
    }

    // If it's not the last one, calculate a new trajectory
    if (ip < NParticlesReally - 1) {
      OSR.SetNewParticle();
      OSR.CalculateTrajectory();

      TOMATH::TSpline1D3<TParticleTrajectoryPoint> const& T = OSR.GetCurrentParticle().GetTrajectoryInterpolated().GetSpline();
      int const NPointsThisTrajectory = T.GetNPoints();
      *h_nt = NPointsThisTrajectory;
      for (size_t it = 0; it < NPointsThisTrajectory; ++it) {
        TParticleTrajectoryPoint const& P  = T.GetY(it);
        TParticleTrajectoryPoint const& PP = T.GetYPP(it);

        h_t[it]   = T.GetX(it);

        h_x[it]   =  P.GetX().GetX();
        h_y[it]   =  P.GetX().GetY();
        h_z[it]   =  P.GetX().GetZ();
        h_xp[it]  = PP.GetX().GetX();
        h_yp[it]  = PP.GetX().GetY();
        h_zp[it]  = PP.GetX().GetZ();

        h_bx[it]  =  P.GetB().GetX();
        h_by[it]  =  P.GetB().GetY();
        h_bz[it]  =  P.GetB().GetZ();
        h_bxp[it] = PP.GetB().GetX();
        h_byp[it] = PP.GetB().GetY();
        h_bzp[it] = PP.GetB().GetZ();

        h_ax[it]  =  P.GetAoverC().GetX();
        h_ay[it]  =  P.GetAoverC().GetY();
        h_az[it]  =  P.GetAoverC().GetZ();
        h_axp[it] = PP.GetAoverC().GetX();
        h_ayp[it] = PP.GetAoverC().GetY();
        h_azp[it] = PP.GetAoverC().GetZ();
      }
    }
  }

  // Wait for last copy
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    hipEventSynchronize(event_resultcopy[ig]);
  }

  NBlocksUsed = 0;
  for (size_t ig = 0; ig < GPUsToUse.size(); ++ig) {
    for (size_t ith = 0; ith < NBlocksThisGPU[ig] * NThreadsPerBlock; ++ith) {
      if (ith + NThreadsPerBlock * NBlocksUsed >= *h_ns) {
        break;
      }
      int iss = ith + NThreadsPerBlock * NBlocksUsed;
      PowerDensityContainer.AddToPoint(iss, h_result[ig][ith]);
    }
    NBlocksUsed += NBlocksThisGPU[ig];
  }

  // Weighting for multi-particle
  double const Weight = 1.0 / (double) NParticlesReally;
  PowerDensityContainer.WeightAll(Weight);

  // Free host memory
  hipHostFree(h_nt);
  hipHostFree(h_t);
  hipHostFree(h_ns);
  hipHostFree(h_x);
  hipHostFree(h_y);
  hipHostFree(h_z);
  hipHostFree(h_xp);
  hipHostFree(h_yp);
  hipHostFree(h_zp);
  hipHostFree(h_bx);
  hipHostFree(h_by);
  hipHostFree(h_bz);
  hipHostFree(h_bxp);
  hipHostFree(h_byp);
  hipHostFree(h_bzp);
  hipHostFree(h_ax);
  hipHostFree(h_ay);
  hipHostFree(h_az);
  hipHostFree(h_axp);
  hipHostFree(h_ayp);
  hipHostFree(h_azp);
  hipHostFree(h_sx);
  hipHostFree(h_sy);
  hipHostFree(h_sz);
  hipHostFree(h_nx);
  hipHostFree(h_ny);
  hipHostFree(h_nz);
  hipHostFree(h_shn);
  hipHostFree(h_const);
  hipHostFree(h_ifirst);
  hipHostFree(h_ml);
  hipHostFree(h_prec);
  hipHostFree(h_rt);
  // Free host and GPU memory
  for (size_t i = 0; i < GPUsToUse.size(); ++i) {

    hipHostFree(h_result[i]);

    // Device number
    int const d = GPUsToUse[i];

    hipSetDevice(d);
    hipFree(d_nt[i]);
    hipFree(d_t[i]);
    hipFree(d_ns[i]);
    hipFree(d_x[i]);
    hipFree(d_y[i]);
    hipFree(d_z[i]);
    hipFree(d_xp[i]);
    hipFree(d_yp[i]);
    hipFree(d_zp[i]);
    hipFree(d_bx[i]);
    hipFree(d_by[i]);
    hipFree(d_bz[i]);
    hipFree(d_bxp[i]);
    hipFree(d_byp[i]);
    hipFree(d_bzp[i]);
    hipFree(d_ax[i]);
    hipFree(d_ay[i]);
    hipFree(d_az[i]);
    hipFree(d_axp[i]);
    hipFree(d_ayp[i]);
    hipFree(d_azp[i]);
    hipFree(d_sx[i]);
    hipFree(d_sy[i]);
    hipFree(d_sz[i]);
    hipFree(d_nx[i]);
    hipFree(d_ny[i]);
    hipFree(d_nz[i]);
    hipFree(d_shn[i]);
    hipFree(d_const[i]);
    hipFree(d_ifirst[i]);
    hipFree(d_ml[i]);
    hipFree(d_prec[i]);
    hipFree(d_rt[i]);
    hipFree(d_result[i]);
  }
  hipFree(h_result);

  hipFree(d_nt);
  hipFree(d_t);
  hipFree(d_ns);
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  hipFree(d_xp);
  hipFree(d_yp);
  hipFree(d_zp);
  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_bz);
  hipFree(d_bxp);
  hipFree(d_byp);
  hipFree(d_bzp);
  hipFree(d_ax);
  hipFree(d_ay);
  hipFree(d_az);
  hipFree(d_axp);
  hipFree(d_ayp);
  hipFree(d_azp);
  hipFree(d_sx);
  hipFree(d_sy);
  hipFree(d_sz);
  hipFree(d_nx);
  hipFree(d_ny);
  hipFree(d_nz);
  hipFree(h_ifirst);
  hipFree(d_shn);
  hipFree(d_const);
  hipFree(d_prec);
  hipFree(d_rt);
  hipFree(d_result);

  // Delete host gpu pointer arrays
  delete [] event_resultcopy;


  return;
}






























